#include <stdafx.h>

#include <iostream>
#include <thread>

#include "hip/hip_runtime.h"

#include "DictionaryScan.cuh"

#include "consts.h"
#include "AdaptiveBase.h"


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "Helper.h"

bool  DispatchDictionaryScan(ConfigClass* Config, data_class* Data, stride_class* Stride) {

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS];
	uint64_t trunkInitEntropy[2];
	uint8_t reqChecksum;

	//TODO: fill host_EntropyAbsolutePrefix64 and host_EntropyBatchNext24
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] = 0;
	host_EntropyNextPrefix2[PTR_AVOIDER] = 0;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[0]) << 53;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[1]) << 42;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[2]) << 31;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[3]) << 20;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[4]) << 9;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[5]) >> 2;
	host_EntropyNextPrefix2[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[5]) << 62; //two bits from main 6 words

	if (SyncWorldWideJobVariables() == false)
		return false;



	AdaptiveDigitsToEntropy(
		host_AdaptiveBaseCurrentBatchInitialDigits
		, host_AdaptiveBaseDigitCarryTrigger
		, host_AdaptiveBaseDigitSet
		, host_EntropyAbsolutePrefix64
		, host_EntropyNextPrefix2
		, host_AdaptiveBaseCurrentBatchInitialDigits
		, trunkInitEntropy,
		&reqChecksum);

	if (trunkInitEntropy[0] == host_EntropyAbsolutePrefix64[0]) {
		std::cout << "Init Entropy Sucessfully initialized by higher bits " << trunkInitEntropy[0] << std::endl;
		if (host_EntropyNextPrefix2[0] == trunkInitEntropy[1]) {
			std::cout << "Init Entropy Sucessfully tested for lower bits " << trunkInitEntropy[1] << std::endl;

		}
	}

	host_EntropyNextPrefix2[0] &= 0xFFFFFF0000000000ULL; //test done, revert nack to only 24 msbs


	size_t copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS * MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION;
	hipError_t cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitSet), host_AdaptiveBaseDigitSet, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "dev_AdaptiveBaseCurrentBatchInitialDigits copying " << copySize << " bytes to dev_AdaptiveBaseDigitSet failed!: " << cudaResult << std::endl;
		return false;
	}


	//Initial zeroing
	//host_nProcessedFromBatch[0] = 0;
	//host_nProcessedMoreThanBatch[0] = 0;




	int nPlannedTrunks = 1;//host_AdaptiveBaseDigitCarryTrigger[0] * host_AdaptiveBaseDigitCarryTrigger[1];

	uint64_t nPrevBatchProcessed = 0;

	uint64_t nPlanned44BitCombos = host_AdaptiveBaseDigitCarryTrigger[2]
		* host_AdaptiveBaseDigitCarryTrigger[3]
		* host_AdaptiveBaseDigitCarryTrigger[4]
		* host_AdaptiveBaseDigitCarryTrigger[5];

	uint64_t nUniversalMax =
		host_AdaptiveBaseDigitCarryTrigger[0]
		* host_AdaptiveBaseDigitCarryTrigger[1]
		* host_AdaptiveBaseDigitCarryTrigger[2]
		* host_AdaptiveBaseDigitCarryTrigger[3]
		* host_AdaptiveBaseDigitCarryTrigger[4]
		* host_AdaptiveBaseDigitCarryTrigger[5];//nPlanned24BitTrunks* nPlanned44BitCombos;

	uint64_t nUniversalProcessed = 0;

	uint64_t nThreadsInBatch = Config->cuda_block * Config->cuda_grid;

	uint64_t nBatchMax = nUniversalMax / nThreadsInBatch;

	if (nBatchMax * nThreadsInBatch < nUniversalMax)
		nBatchMax++;

	int nBatch = 0;


	std::cout << ">> (" << __DATE__ << "@" << __TIME__ << ") ->" << "Planing to check total " << nUniversalMax << " combinations structured in maximum " << nBatchMax << " batches "
		" of " << nThreadsInBatch << " threads each" << std::endl;


	uint64_t nTrunk = 0;
	do
	{
		if (nTrunk >= nPlannedTrunks) {
			break;
		}

		std::cout << "> Starting Dictionary SCAN -- " << std::endl;

		if (SyncWorldWideJobVariables() == false)
			return false;


		nBatch = 0;

		int16_t batchDigits[MAX_ADAPTIVE_BASE_POSITIONS];
		if (IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger
			, host_AdaptiveBaseCurrentBatchInitialDigits
			, 0 //kinda copy
			, batchDigits)) {
			//printf("Batch digits initialized for the first time.\r\n");
		}

		//std::cout << "ALL VARIANTS:" << std::endl;

		//uint64_t batchMnemo[2];
		//batchMnemo[0] = host_EntropyAbsolutePrefix64[0];
		//batchMnemo[1] = host_EntropyNextPrefix2[0] & 0xB0000000; //scrutinize;

		//for (int i = 0; i < 4; i++) {
		//	PrintNextMnemo(batchMnemo, i, host_AdaptiveBaseDigitCarryTrigger , host_AdaptiveBaseCurrentBatchInitialDigits, host_AdaptiveBaseDigitSet);
		//}

		//for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++) {
		//	std::cout << host_AdaptiveBaseCurrentBatchInitialDigits[i] << "=" << batchDigits[i] << std::endl;
		//}

		size_t copySize;
		hipError_t cudaResult;

		copySize = sizeof(uint64_t);
		cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyAbsolutePrefix64), host_EntropyAbsolutePrefix64, copySize, 0, hipMemcpyHostToDevice);
		if (cudaResult != hipSuccess)
		{
			std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyAbsolutePrefix64 failed!: " << cudaResult << std::endl;
			return false;
		}

		copySize = sizeof(host_AdaptiveBaseDigitCarryTrigger[0]) * MAX_ADAPTIVE_BASE_POSITIONS;
		cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitCarryTrigger), host_AdaptiveBaseDigitCarryTrigger, copySize, 0, hipMemcpyHostToDevice);
		if (cudaResult != hipSuccess)
		{
			std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseDigitCarryTrigger failed!: " << cudaResult << std::endl;
			return false;
		}

		do { //batch


			//TODO: increment entropy here accordingto grid , processed and extra

			const int elemSize = sizeof(int16_t);
			copySize = elemSize * MAX_ADAPTIVE_BASE_POSITIONS;

			cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), batchDigits, copySize, 0, hipMemcpyHostToDevice);
			if (cudaResult != hipSuccess)
			{
				std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
				return false;
			}



			std::cout << "BATCH #"
				<< nBatch << " of " << nBatchMax << std::endl;

			*Data->host.host_nProcessedFromBatch = 0;
			*Data->host.host_nProcessedMoreThanBatch = 0;

			if (hipSuccess != hipMemcpy(Data->dev.dev_nProcessedFromBatch, Data->host.host_nProcessedFromBatch, 8, hipMemcpyHostToDevice)) {
				std::cout << "Error-Line--" << __LINE__ << std::endl;
			}

			if (hipSuccess != hipMemcpy(Data->dev.dev_nProcessedMoreThanBatch, Data->host.host_nProcessedMoreThanBatch, 8, hipMemcpyHostToDevice)) {
				std::cout << "Error-Line--" << __LINE__ << std::endl;
			}

			tools::start_time();

			if (Stride->startDictionaryAttack(Config->cuda_grid, Config->cuda_block) != 0) {
				std::cerr << "Error START!!" << std::endl;
				return false;
			}

			//TODO: Handled by dictionary attack with index
			std::cout << "Waiting for batch " << nBatch << " to finish." << std::endl;
			//if (save_thread.joinable()) save_thread.join();

			if (Stride->endDictionaryAttack() != 0) {
				std::cerr << "Error END!!" << std::endl;
				return false;
			}

			//if (bCfgSaveResultsIntoFile) {
			//	save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config->generate_path);
			//}

			if (hipSuccess != hipMemcpy(Data->host.host_nProcessedFromBatch, Data->dev.dev_nProcessedFromBatch, 8, hipMemcpyDeviceToHost)) {
				std::cout << "Error-Line--" << __LINE__ << std::endl;
			}

			if (hipSuccess != hipMemcpy(Data->host.host_nProcessedMoreThanBatch, Data->dev.dev_nProcessedMoreThanBatch, 8, hipMemcpyDeviceToHost)) {
				std::cout << "Error-Line--" << __LINE__ << std::endl;
			}

			uint64_t nTotalThisBatch = 0;
			uint64_t v1 = *Data->host.host_nProcessedFromBatch;
			uint64_t v2 = *Data->host.host_nProcessedMoreThanBatch;
			if (nBatch != nBatchMax && v1 != nThreadsInBatch) {
				printf("This batch appears to be the last one!\r\n");
			}
			nTotalThisBatch = v1 + v2;

			printf("checking results of %ul + %ul = %ul checkups\r\n", v1, v2, nTotalThisBatch);
			tools::checkResult(Data->host.ret);

			float delay;
			tools::stop_time_and_calc_sec(&delay);
			std::cout << std::endl << "PROCESSED: at " << tools::formatPrefix((double)nTotalThisBatch / delay) << " COMBO/Sec" << std::endl;
			//std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
			//	<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
			//	<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
			//	<< " | ROUND: " << nTrunk;

			//nPrevBatchProcessed = Data->host.host_nProcessedFromBatch[PTR_AVOIDER]
			//	+ Data->host.host_nProcessedMoreThanBatch[PTR_AVOIDER];
			//std::cout << ">>>This batch (#" << nBatch << ") completed processing " << nPrevBatchProcessed << " combos." << std::endl;
//				nCumulativeCombosProcessedInTrunk += nPrevBatchProcessed;

			if (IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger
				, host_AdaptiveBaseCurrentBatchInitialDigits
				, nTotalThisBatch
				, batchDigits) == false) {
				printf("Nothing more to traverse\r\n");
			}
			//memcpy(&host_AdaptiveBaseCurrentBatchInitialDigits[0], &batchDigits[0], sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS)
			for (int x = 0; x < MAX_ADAPTIVE_BASE_POSITIONS; x++) {
				host_AdaptiveBaseCurrentBatchInitialDigits[x] = batchDigits[x];
			}

			nUniversalProcessed += nTotalThisBatch;

			nBatch++;
		} while (nUniversalProcessed < nUniversalMax); //batch

		std::cout << ">>This Trunk (#" << nTrunk << ") completed processing " << nUniversalProcessed << "/" << nUniversalMax << "  combinations" << std::endl;
		//nCumulativeCombosProcessedInTrunk = 0;

	} while (false);//trunk
	return true;
}


bool SyncWorldWideJobVariables()
{
	AdaptiveUpdateMnemonicLow64(host_EntropyNextPrefix2
		, host_AdaptiveBaseDigitSet
		, host_AdaptiveBaseCurrentBatchInitialDigits);

	host_EntropyNextPrefix2[0] &= 0xFFFFFF00000000;
	size_t copySize = sizeof(uint64_t);
	hipError_t cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyNextPrefix2), host_EntropyNextPrefix2, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyBatchNext24 failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), host_AdaptiveBaseCurrentBatchInitialDigits, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
		return false;
	}

	return true;
}
