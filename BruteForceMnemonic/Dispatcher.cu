﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		28-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "AdaptiveBase.h"

#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"






static std::thread save_thread;

int Generate_Mnemonic(void)
{


	std::cout << "Compile on Date **** : " << __DATE__ << ", Time:" << __TIME__ << std::endl;
	//{//TODO make all NULL
	//	dev_nProcessedFromBatch = NULL;
	//	host_nProcessedFromBatch = NULL;
	//	dev_nProcessedMoreThanBatch = NULL;
	//	host_nProcessedMoreThanBatch = NULL;
	//}


	hipError_t cudaStatus = hipSuccess;
	int err;
	ConfigClass Config;
	try {

		for (int x = 0; x < MAX_ADAPTIVE_BASE_POSITIONS; x++) {
			host_AdaptiveBaseCurrentBatchInitialDigits[x] = 0;
		}

		parse_config(&Config, "config.cfg");

		std::vector<std::string> startFrom = tools::SplitWords(Config.static_words_starting_point);


		int nLastKnownPos = -1;
		std::vector<int> validIndexListPerPos[NUM_WORDS_MNEMONIC];

		for (int nemoIter = 0; nemoIter < NUM_WORDS_MNEMONIC; nemoIter++) {
			int16_t thisPosBipStarting;
			std::string thisPosStartFromWord = startFrom[nemoIter];
			tools::GetSingleWordIndex(thisPosStartFromWord, &thisPosBipStarting);
			//int16_t thisPosDicStarting = -1;



			std::vector<std::string> thisMnemoPosDictionaryLine = tools::SplitWords(Config.dynamic_words[nemoIter]);
			int thisPosLineWordCount = thisMnemoPosDictionaryLine.size();

			if (thisPosLineWordCount > MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION) {
				std::cout << "ERROR: Maximum Allowed word count per line is " << MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION <<std::endl;
				goto Error;

			}

			int64_t adaptivePositionIdx = nemoIter - MAX_ADAPTIVE_BASE_POSITIONS;

			if (thisPosLineWordCount == 1) { //find consequtive count of single-word dictionaries
				int prev = nemoIter - 1;
				if (prev == nLastKnownPos)
					nLastKnownPos = nemoIter;
			}


			for (int16_t thisPosDictTraverseIdx = 0; thisPosDictTraverseIdx < thisPosLineWordCount; thisPosDictTraverseIdx++) {
				
				std::string thisWord = thisMnemoPosDictionaryLine[thisPosDictTraverseIdx];


				//Fill the digit-space for each adaptive base position (last 6 in our case)
				int16_t thisBipIdx;
				tools::GetSingleWordIndex(thisWord, &thisBipIdx);

				if (thisPosDictTraverseIdx == 0) {//leave old algorithm working for now with separated positions					
					Config.words_indicies_mnemonic[nemoIter] = thisBipIdx; //or even -1 when ? 
				}

				if (adaptivePositionIdx < 0)
					break;

				//FROM now on, we are on the second 6 words

				host_AdaptiveBaseDigitSet[adaptivePositionIdx][thisPosDictTraverseIdx] = thisBipIdx; //scrutinize what we do with -1 instances
				host_AdaptiveBaseDigitCarryTrigger[adaptivePositionIdx] = thisPosLineWordCount; //TODO: scrutinize (minus one needed?)

				//Check if we are going to start from this word, make adjustments and print info messages
				bool bStartsFromThisWord = (0 == strcmp(thisWord.c_str(), thisPosStartFromWord.c_str()));
				if (!bStartsFromThisWord)
					continue;

				//FROM now on, we start from this word

				std::ostringstream isAdaptiveStr;

				isAdaptiveStr.str("");

				host_AdaptiveBaseCurrentBatchInitialDigits[adaptivePositionIdx] = thisPosDictTraverseIdx;

				//std::cout << "SETTING " << adaptivePositionIdx << " @" << thisPosDictTraverseIdx << std::endl;

				if (adaptivePositionIdx >= 0) {
					isAdaptiveStr << "[Dynamic:" << thisPosLineWordCount << "]";
				}
				else if (thisPosLineWordCount == 1) {
					isAdaptiveStr.str("[STATIC]");
				}

				std::cout << "Postition " << nemoIter << isAdaptiveStr.str() << " starts from word: " << thisWord << " at PosDictionary: " << thisPosDictTraverseIdx << " BIP: "  << thisBipIdx << " and carries at:" << host_AdaptiveBaseDigitCarryTrigger[nemoIter] << std::endl;

			}//single dictionary in each position
		} //nemo positions

		if (nLastKnownPos >= 0)
			std::cout << "Words up to position " << nLastKnownPos << " (out of 0 to 11) are known" << std::endl;
		else
			std::cout << "All words are dynamic" << std::endl;

		for (int pp = 0; pp < MAX_ADAPTIVE_BASE_POSITIONS; pp++) {
			std::cout << "The position:" << pp + MAX_ADAPTIVE_BASE_POSITIONS << " Carries at:" << host_AdaptiveBaseDigitCarryTrigger[pp] << " and starts from " << host_AdaptiveBaseCurrentBatchInitialDigits[pp] << std::endl;
		}


		uint64_t number_of_generated_mnemonics = (Config.number_of_generated_mnemonics / (Config.cuda_block * Config.cuda_grid)) * (Config.cuda_block * Config.cuda_grid);
		if ((Config.number_of_generated_mnemonics % (Config.cuda_block * Config.cuda_grid)) != 0) number_of_generated_mnemonics += Config.cuda_block * Config.cuda_grid;
		Config.number_of_generated_mnemonics = number_of_generated_mnemonics;	
	}//try
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}//catch


	int nDevCount = devicesInfo();



	uint32_t num_device = 0;
	if (nDevCount != 1) { //select only cuda device automatically
#ifndef TEST_MODE
		std::cout << "\n\nEnter number of device: ";
		std::cin >> num_device;
#endif //TEST_MODE
	}
	else {
		std::cout << " The only CUDA capable device selected automatically." << std::endl;
	}
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = Config.cuda_grid * Config.cuda_block;
	if (num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE)
	{
		std::cerr << "Error num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE!" << std::endl;
		return -1;
	}
	uint32_t num_bytes = 0;
	if (Config.chech_equal_bytes_in_adresses == "yes")
	{
#ifdef TEST_MODE
		num_bytes = 5;
#else
		num_bytes = 8;
#endif //TEST_MODE
	}

	std::cout << "\nNUM WALLETS IN PACKET GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
	data_class* Data = new data_class();
	stride_class* Stride = new stride_class(Data);
	size_t num_addresses_in_tables = 0;

	std::cout << "READ TABLES! WAIT..." << std::endl;
	tools::clearFiles();
	if((Config.generate_path[0] != 0) || (Config.generate_path[1] != 0) || (Config.generate_path[2] != 0) || (Config.generate_path[3] != 0) || (Config.generate_path[4] != 0)
		|| (Config.generate_path[5] != 0))
	{
		std::cout << "READ TABLES LEGACY(BIP32, BIP44)..." << std::endl;
	err = tools::readAllTables(Data->host.tables_legacy, Config.folder_tables_legacy, "", &num_addresses_in_tables);
	if (err == -1) {
		std::cerr << "Error readAllTables legacy!" << std::endl;
		goto Error;
	}
	}

	bool bCfgSaveResultsIntoFile = (Config.save_generation_result_in_file == "yes")?true:false;
	bool bCfgUseOldMethod = (Config.use_old_random_method == "yes")?true:false;


	//if (bCfgUseOldMethod) {
		if ((Config.generate_path[6] != 0) || (Config.generate_path[7] != 0))
		{
			std::cout << "READ TABLES SEGWIT(BIP49)..." << std::endl;
			err = tools::readAllTables(Data->host.tables_segwit, Config.folder_tables_segwit, "", &num_addresses_in_tables);
			if (err == -1) {
				std::cerr << "Error readAllTables segwit!" << std::endl;
				goto Error;
			}
		}
		if ((Config.generate_path[8] != 0) || (Config.generate_path[9] != 0))
		{
			std::cout << "READ TABLES NATIVE SEGWIT(BIP84)..." << std::endl;
			err = tools::readAllTables(Data->host.tables_native_segwit, Config.folder_tables_native_segwit, "", &num_addresses_in_tables);
			if (err == -1) {
				std::cerr << "Error readAllTables native segwit!" << std::endl;
				goto Error;
			}
		}
		std::cout << std::endl << std::endl;

		if (num_addresses_in_tables == 0) {
			std::cerr << "ERROR READ TABLES!! NO ADDRESSES IN FILES!!" << std::endl;
			goto Error;
		}
//	}

	if (Data->malloc(Config.cuda_grid, Config.cuda_block, Config.num_paths, Config.num_child_addresses, bCfgSaveResultsIntoFile) != 0) {
		std::cerr << "Error Data->malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		std::cerr << "Error INIT!!" << std::endl;
		goto Error;
	}

	Data->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "PATH: " << std::endl;

	//if (bCfgUseOldMethod) {
		if (Config.generate_path[0] != 0) std::cout << "m/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[1] != 0) std::cout << "m/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[2] != 0) std::cout << "m/0/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[3] != 0) std::cout << "m/0/1/0.." << (Config.num_child_addresses - 1) << std::endl;
//	}
	if (Config.generate_path[4] != 0) std::cout << "m/44'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;

//	if (bCfgUseOldMethod) {
		if (Config.generate_path[5] != 0) std::cout << "m/44'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[6] != 0) std::cout << "m/49'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[7] != 0) std::cout << "m/49'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[8] != 0) std::cout << "m/84'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[9] != 0) std::cout << "m/84'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
//	}
	std::cout << "\nGENERATE " << tools::formatWithCommas(Config.number_of_generated_mnemonics) << " MNEMONICS. " << tools::formatWithCommas(Config.number_of_generated_mnemonics * Data->num_all_childs) << " ADDRESSES. MNEMONICS IN ROUNDS " << tools::formatWithCommas(Data->wallets_in_round_gpu) << ". WAIT...\n\n";

	//TODO: Here we should create incremental task: /or here
	tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to num_bytes_find failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_generate_path), &Config.generate_path, sizeof(Config.generate_path), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_generate_path failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_childs), &Config.num_child_addresses, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_child failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_paths), &Config.num_paths, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_paths failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_static_words_indices), &Config.words_indicies_mnemonic, 12*2, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_gen_words_indices failed!" << std::endl;
		goto Error;
	}

	if (bCfgUseOldMethod == false) {

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS];
	uint64_t trunkInitEntropy[2];
	uint8_t reqChecksum;

	//TODO: fill host_EntropyAbsolutePrefix64 and host_EntropyBatchNext24
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] = 0;
	host_EntropyNextPrefix2[PTR_AVOIDER] = 0;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[0]) << 53;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[1]) << 42;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[2]) << 31;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[3]) << 20;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[4]) << 9;
								 
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[5]) >> 2;
	host_EntropyNextPrefix2[PTR_AVOIDER]      |= (uint64_t)(Config.words_indicies_mnemonic[5]) << 62; //two bits from main 6 words

	if (NewTrunkPrefix() == false)
		goto Error;
	


	AdaptiveDigitsToEntropy(
		  host_AdaptiveBaseCurrentBatchInitialDigits
		, host_AdaptiveBaseDigitCarryTrigger
		, host_AdaptiveBaseDigitSet
		, host_EntropyAbsolutePrefix64
		, host_EntropyNextPrefix2
		, host_AdaptiveBaseCurrentBatchInitialDigits
		, trunkInitEntropy, 
		  &reqChecksum);

	if (trunkInitEntropy[0] == host_EntropyAbsolutePrefix64[0]) {
		std::cout << "Init Entropy Sucessfully initialized by higher bits "<< trunkInitEntropy[0] << std::endl;
		if (host_EntropyNextPrefix2[0] == trunkInitEntropy[1]) {
			std::cout << "Init Entropy Sucessfully tested for lower bits " << trunkInitEntropy[1] << std::endl;

		}
	}

	host_EntropyNextPrefix2[0] &= 0xFFFFFF0000000000ULL; //test done, revert nack to only 24 msbs


	size_t copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS * MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION;
	hipError_t cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitSet), host_AdaptiveBaseDigitSet, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "dev_AdaptiveBaseCurrentBatchInitialDigits copying " << copySize << " bytes to dev_AdaptiveBaseDigitSet failed!: " << cudaResult << std::endl;
		goto Error;
	}


	//Initial zeroing
	//host_nProcessedFromBatch[0] = 0;
	//host_nProcessedMoreThanBatch[0] = 0;




	int nPlannedTrunks = 1;//host_AdaptiveBaseDigitCarryTrigger[0] * host_AdaptiveBaseDigitCarryTrigger[1];

		uint64_t nPrevBatchProcessed = 0;

		uint64_t nPlanned44BitCombos = host_AdaptiveBaseDigitCarryTrigger[2]
			* host_AdaptiveBaseDigitCarryTrigger[3]
			* host_AdaptiveBaseDigitCarryTrigger[4]
			* host_AdaptiveBaseDigitCarryTrigger[5];

		uint64_t nUniversalMax = 
			  host_AdaptiveBaseDigitCarryTrigger[0]
			* host_AdaptiveBaseDigitCarryTrigger[1]
			* host_AdaptiveBaseDigitCarryTrigger[2]
			* host_AdaptiveBaseDigitCarryTrigger[3]
			* host_AdaptiveBaseDigitCarryTrigger[4]
			* host_AdaptiveBaseDigitCarryTrigger[5];//nPlanned24BitTrunks* nPlanned44BitCombos;

		uint64_t nUniversalProcessed = 0;

		uint64_t nThreadsInBatch = Config.cuda_block * Config.cuda_grid;
		
		uint64_t nBatchMax = nUniversalMax / nThreadsInBatch;
		
		if (nBatchMax * nThreadsInBatch < nUniversalMax)
			nBatchMax++;

		//uint64_t nCumulativeCombosProcessedInTrunk = 0;
		int nBatch = 0;


		std::cout << ">> (" << __DATE__ << "@" << __TIME__ << ") ->" << "Planing to check total "<< nUniversalMax <<" combinations structured in maximum " << nBatchMax << " batches "
			" of " << nThreadsInBatch << " threads each" << std::endl;


		//Config.number_of_generated_mnemonics / (Data->wallets_in_round_gpu)
		uint64_t nTrunk = 0;
		do 
		{
			if (nTrunk >= nPlannedTrunks) {
				break;
			}

			std::cout << "> NEW TRUNK -- " << "No:" << nTrunk << "/" << nPlannedTrunks - 1 << std::endl;

			if (NewTrunkPrefix() == false)
				goto Error;


			nBatch = 0;

			int16_t batchDigits[MAX_ADAPTIVE_BASE_POSITIONS];
			if (IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger
				, host_AdaptiveBaseCurrentBatchInitialDigits
				, 0 //kinda copy
				, batchDigits)) {
				printf("Batch digits initialized for the first time.\r\n");
			}

			//std::cout << "ALL VARIANTS:" << std::endl;

			uint64_t batchMnemo[2];
			batchMnemo[0] = host_EntropyAbsolutePrefix64[0];
			batchMnemo[1] = host_EntropyNextPrefix2[0] & 0xB0000000; //scrutinize;

			for (int i = 0; i < 4; i++) {
				PrintNextMnemo(batchMnemo, i, host_AdaptiveBaseDigitCarryTrigger , host_AdaptiveBaseCurrentBatchInitialDigits, host_AdaptiveBaseDigitSet);
			}

			//for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++) {
			//	std::cout << host_AdaptiveBaseCurrentBatchInitialDigits[i] << "=" << batchDigits[i] << std::endl;
			//}
			do  { //batch


				//TODO: increment entropy here accordingto grid , processed and extra

				size_t copySize;
				hipError_t cudaResult;

				copySize = sizeof(uint64_t);
				cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyAbsolutePrefix64), host_EntropyAbsolutePrefix64, copySize, 0, hipMemcpyHostToDevice);
				if (cudaResult != hipSuccess)
				{
					std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyAbsolutePrefix64 failed!: " << cudaResult << std::endl;
					goto Error;
				}


				const int elemSize = sizeof(int16_t);
				copySize = elemSize * MAX_ADAPTIVE_BASE_POSITIONS;

				cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), batchDigits, copySize, 0, hipMemcpyHostToDevice);
				if (cudaResult != hipSuccess)
				{
					std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
					goto Error;
				}
				copySize = sizeof(host_AdaptiveBaseDigitCarryTrigger[0]) * MAX_ADAPTIVE_BASE_POSITIONS;
				cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitCarryTrigger), host_AdaptiveBaseDigitCarryTrigger, copySize, 0, hipMemcpyHostToDevice);
				if (cudaResult != hipSuccess)
				{
					std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseDigitCarryTrigger failed!: " << cudaResult << std::endl;
					goto Error;
				}

				
	
				std::cout << ">> NEW BATCH -- "
					<< "No:" << nBatch << "/" << nBatchMax << std::endl;

				*Data->host.host_nProcessedFromBatch = 0;
				*Data->host.host_nProcessedMoreThanBatch = 0;
				
				if (hipSuccess != hipMemcpy(Data->dev.dev_nProcessedFromBatch, Data->host.host_nProcessedFromBatch, 8, hipMemcpyHostToDevice)) {
					std::cout << "Error-Line--" << __LINE__ << std::endl;
				}

				if (hipSuccess != hipMemcpy(Data->dev.dev_nProcessedMoreThanBatch, Data->host.host_nProcessedMoreThanBatch, 8, hipMemcpyHostToDevice)) {
					std::cout << "Error-Line--" << __LINE__ << std::endl;
				}

				tools::start_time();

				if (Stride->startDictionaryAttack(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}

				//TODO: Handled by dictionary attack with index
				//tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));
				std::cout << "Waiting for batch " << nBatch << " to finish." << std::endl;
				if (save_thread.joinable()) save_thread.join();

				if (Stride->endDictionaryAttack() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}

				if (bCfgSaveResultsIntoFile) {
					save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
					//tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
				}

				if (hipSuccess != hipMemcpy( Data->host.host_nProcessedFromBatch, Data->dev.dev_nProcessedFromBatch, 8, hipMemcpyDeviceToHost)) {
					std::cout << "Error-Line--" << __LINE__ << std::endl;
				}

				if (hipSuccess != hipMemcpy(Data->host.host_nProcessedMoreThanBatch, Data->dev.dev_nProcessedMoreThanBatch, 8, hipMemcpyDeviceToHost)) {
					std::cout << "Error-Line--" << __LINE__ << std::endl;
				}

				uint64_t nTotalThisBatch = 0;
				uint64_t v1 = *Data->host.host_nProcessedFromBatch;
				uint64_t v2 = *Data->host.host_nProcessedMoreThanBatch;
				if (nBatch != nBatchMax && v1 != nThreadsInBatch) {
					printf("This batch appears to be the last one!\r\n");
				}
				nTotalThisBatch = v1 + v2;

				printf("checking results of %ul + %ul = %ul checkups\r\n", v1, v2, nTotalThisBatch);
				tools::checkResult(Data->host.ret);

				float delay;
				tools::stop_time_and_calc_sec(&delay);
				//std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
				//	<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
				//	<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
				//	<< " | ROUND: " << nTrunk;

				//nPrevBatchProcessed = Data->host.host_nProcessedFromBatch[PTR_AVOIDER]
				//	+ Data->host.host_nProcessedMoreThanBatch[PTR_AVOIDER];
				//std::cout << ">>>This batch (#" << nBatch << ") completed processing " << nPrevBatchProcessed << " combos." << std::endl;
//				nCumulativeCombosProcessedInTrunk += nPrevBatchProcessed;

				if (IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger
					, host_AdaptiveBaseCurrentBatchInitialDigits
					, nTotalThisBatch
					, batchDigits) == false) {
					printf("Nothing more to traverse\r\n");
				}
				//memcpy(&host_AdaptiveBaseCurrentBatchInitialDigits[0], &batchDigits[0], sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS)
				for (int x = 0; x < MAX_ADAPTIVE_BASE_POSITIONS; x++) {
					host_AdaptiveBaseCurrentBatchInitialDigits[x] = batchDigits[x];
				}

				nUniversalProcessed += nTotalThisBatch;

				nBatch++;
			} while (nUniversalProcessed < nUniversalMax); //batch

			std::cout << ">>This Trunk (#" << nTrunk << ") completed processing " << nUniversalProcessed <<"/"<< nUniversalMax <<"  combinations" << std::endl;
			//nCumulativeCombosProcessedInTrunk = 0;

		}while (false);//trunk
	}//NEW METHOD
	else {
		for (uint64_t step = 0; step < Config.number_of_generated_mnemonics / (Data->wallets_in_round_gpu); step++)
		{
			tools::start_time();

			if (bCfgSaveResultsIntoFile) {
				if (Stride->start_for_save(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}
			}
			else
			{
				if (Stride->start(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}
			}

			//TODO: Here we should create incremental task : /here
			tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

			if (save_thread.joinable()) save_thread.join();

			if (bCfgSaveResultsIntoFile) {
				if (Stride->end_for_save() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}
			}
			else
			{
				if (Stride->end() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}
			}
			

			if (bCfgSaveResultsIntoFile) {
				save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
				//tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
			}

			tools::checkResult(Data->host.ret);

			float delay;
			tools::stop_time_and_calc_sec(&delay);
			std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
				<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
				<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
				<< " | ROUND: " << step;

		}//for (step)

	}

	std::cout << "\n\nEND!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.


	{//TODO: Free Memory
		//hipFree(dev_nProcessedFromBatch);
		//hipFree(dev_nProcessedMoreThanBatch);
		//hipHostFree(host_nProcessedFromBatch);
		//hipHostFree(host_nProcessedMoreThanBatch);
	}



	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}


	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;


}
__host__ __device__
void PrintNextMnemo(uint64_t batchMnemo[2] , uint64_t nHowMuch, int16_t carry [MAX_ADAPTIVE_BASE_POSITIONS]
	, int16_t initDigits[MAX_ADAPTIVE_BASE_POSITIONS]
	, int16_t digitSet [MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
)
{
	int16_t  batchDigits[6];
	//uint64_t batchMnemo[2];
	//batchMnemo[0] = host_EntropyAbsolutePrefix64[0];
	//batchMnemo[1] = host_EntropyBatchNext24[0] & 0xB0000000; //scrutinize;
	printf("before->after::[%ul] == \n", nHowMuch  );

	if (IncrementAdaptiveDigits(carry, initDigits, nHowMuch, batchDigits) == false) {
		printf("Not able to add %ul\r\n", nHowMuch);
	}

	for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++)
		printf("[ %d,  %d ] - ", initDigits[i] ,batchDigits[i]);

	AdaptiveUpdateMnemonicLow64(&batchMnemo[1]
		, digitSet
		, batchDigits);


	int16_t tmp2[12] = {
		(batchMnemo[0] >> 53) & 2047,
		(batchMnemo[0] >> 42) & 2047,
		(batchMnemo[0] >> 31) & 2047,
		(batchMnemo[0] >> 20) & 2047,
		(batchMnemo[0] >> 9) & 2047,
		((batchMnemo[0] & ((1 << 9) - 1)) << 2) | ((batchMnemo[1] >> 62) & 3),
		(batchMnemo[1] >> 51) & 2047,
		(batchMnemo[1] >> 40) & 2047,
		(batchMnemo[1] >> 29) & 2047,
		(batchMnemo[1] >> 18) & 2047,
		(batchMnemo[1] >> 7) & 2047,
		((batchMnemo[1] & ((1 << 7) - 1)) << 4)

	};

	int16_t temArr[6] = {
		digitSet[0][batchDigits[0]]
		,	digitSet[1][batchDigits[1]]
		,	digitSet[2][batchDigits[2]]
		,	digitSet[3][batchDigits[3]]
		,	digitSet[4][batchDigits[4]]
		,	digitSet[5][batchDigits[5]] };
	printf ("Stars from 2nd half [%ul] --> %s\r\n", nHowMuch , tools::GetMnemoString(temArr, 6).c_str() );
	printf ("Fully last checksum: [%ul] --> %s\r\n" ,nHowMuch, tools::GetMnemoString(tmp2, 12).c_str());
}

bool NewTrunkPrefix()
{
	AdaptiveUpdateMnemonicLow64(host_EntropyNextPrefix2
		, host_AdaptiveBaseDigitSet
		, host_AdaptiveBaseCurrentBatchInitialDigits);

	host_EntropyNextPrefix2[0] &= 0xFFFFFF00000000;
	size_t copySize = sizeof(uint64_t);
	hipError_t cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyNextPrefix2), host_EntropyNextPrefix2, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyBatchNext24 failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), host_AdaptiveBaseCurrentBatchInitialDigits, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
		return false;
	}

	return true;
}







