﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		28-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "AdaptiveBase.h"

#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"

#include "DispatchDictionaryScan.cuh"




static std::thread save_thread;

int Generate_Mnemonic(void)
{
	std::cout << "Compiled on Date *** : " << __DATE__ << ", Time:" << __TIME__ << std::endl;

	ConfigClass Config;
	hipError_t cudaStatus = hipSuccess;
	int err;

	if (ApplyConfig(Config) == false) {
		goto Error;
	}

	uint64_t number_of_generated_mnemonics = (Config.number_of_generated_mnemonics / (Config.cuda_block * Config.cuda_grid)) * (Config.cuda_block * Config.cuda_grid);
	if ((Config.number_of_generated_mnemonics % (Config.cuda_block * Config.cuda_grid)) != 0) number_of_generated_mnemonics += Config.cuda_block * Config.cuda_grid;
	Config.number_of_generated_mnemonics = number_of_generated_mnemonics;


	int nDevCount = devicesInfo();



	uint32_t num_device = 0;
	if (nDevCount != 1) { //select only cuda device automatically
#ifndef TEST_MODE
		std::cout << "\n\nEnter number of device: ";
		std::cin >> num_device;
#endif //TEST_MODE
	}
	else {
		std::cout << " The only CUDA capable device selected automatically." << std::endl;
	}
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = Config.cuda_grid * Config.cuda_block;
	if (num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE)
	{
		std::cerr << "Error num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE!" << std::endl;
		return -1;
	}
	uint32_t num_bytes = 0;
	if (Config.chech_equal_bytes_in_adresses == "yes")
	{
#ifdef TEST_MODE
		num_bytes = 5;
#else
		num_bytes = 8;
#endif //TEST_MODE
	}

	std::cout << "\nNUM WALLETS IN PACKET GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
	data_class* Data = new data_class();
	stride_class* Stride = new stride_class(Data);
	size_t num_addresses_in_tables = 0;

	std::cout << "READ TABLES! WAIT..." << std::endl;
	tools::clearFiles();
	if((Config.generate_path[0] != 0) || (Config.generate_path[1] != 0) || (Config.generate_path[2] != 0) || (Config.generate_path[3] != 0) || (Config.generate_path[4] != 0)
		|| (Config.generate_path[5] != 0))
	{
		std::cout << "READ TABLES LEGACY(BIP32, BIP44)..." << std::endl;
	err = tools::readAllTables(Data->host.tables_legacy, Config.folder_tables_legacy, "", &num_addresses_in_tables);
	if (err == -1) {
		std::cerr << "Error readAllTables legacy!" << std::endl;
		goto Error;
	}
	}

	bool bCfgSaveResultsIntoFile = (Config.save_generation_result_in_file == "yes")?true:false;
	bool bCfgUseOldMethod = (Config.use_old_random_method == "yes")?true:false;


	//if (bCfgUseOldMethod) {
		if ((Config.generate_path[6] != 0) || (Config.generate_path[7] != 0))
		{
			std::cout << "READ TABLES SEGWIT(BIP49)..." << std::endl;
			err = tools::readAllTables(Data->host.tables_segwit, Config.folder_tables_segwit, "", &num_addresses_in_tables);
			if (err == -1) {
				std::cerr << "Error readAllTables segwit!" << std::endl;
				goto Error;
			}
		}
		if ((Config.generate_path[8] != 0) || (Config.generate_path[9] != 0))
		{
			std::cout << "READ TABLES NATIVE SEGWIT(BIP84)..." << std::endl;
			err = tools::readAllTables(Data->host.tables_native_segwit, Config.folder_tables_native_segwit, "", &num_addresses_in_tables);
			if (err == -1) {
				std::cerr << "Error readAllTables native segwit!" << std::endl;
				goto Error;
			}
		}
		std::cout << std::endl << std::endl;

		if (num_addresses_in_tables == 0) {
			std::cerr << "ERROR READ TABLES!! NO ADDRESSES IN FILES!!" << std::endl;
			goto Error;
		}
//	}

	if (Data->malloc(Config.cuda_grid, Config.cuda_block, Config.num_paths, Config.num_child_addresses, bCfgSaveResultsIntoFile) != 0) {
		std::cerr << "Error Data->malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		std::cerr << "Error INIT!!" << std::endl;
		goto Error;
	}

	Data->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "PATH: " << std::endl;

	//if (bCfgUseOldMethod) {
		if (Config.generate_path[0] != 0) std::cout << "m/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[1] != 0) std::cout << "m/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[2] != 0) std::cout << "m/0/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[3] != 0) std::cout << "m/0/1/0.." << (Config.num_child_addresses - 1) << std::endl;
//	}
	if (Config.generate_path[4] != 0) std::cout << "m/44'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;

//	if (bCfgUseOldMethod) {
		if (Config.generate_path[5] != 0) std::cout << "m/44'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[6] != 0) std::cout << "m/49'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[7] != 0) std::cout << "m/49'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[8] != 0) std::cout << "m/84'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[9] != 0) std::cout << "m/84'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
//	}
	std::cout << "\nGENERATE " << tools::formatWithCommas(Config.number_of_generated_mnemonics) << " MNEMONICS. " << tools::formatWithCommas(Config.number_of_generated_mnemonics * Data->num_all_childs) << " ADDRESSES. MNEMONICS IN ROUNDS " << tools::formatWithCommas(Data->wallets_in_round_gpu) << ". WAIT...\n\n";

	//TODO: Here we should create incremental task: /or here
	tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to num_bytes_find failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_generate_path), &Config.generate_path, sizeof(Config.generate_path), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_generate_path failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_childs), &Config.num_child_addresses, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_child failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_paths), &Config.num_paths, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_paths failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_static_words_indices), &Config.words_indicies_mnemonic, 12*2, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_gen_words_indices failed!" << std::endl;
		goto Error;
	}

	if (bCfgUseOldMethod == false) {
		if (!DispatchDictionaryScan(&Config, Data, Stride)) {
			goto Error;
		}
	}//NEW METHOD
	else {
		for (uint64_t step = 0; step < Config.number_of_generated_mnemonics / (Data->wallets_in_round_gpu); step++)
		{
			tools::start_time();

			if (bCfgSaveResultsIntoFile) {
				if (Stride->start_for_save(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}
			}
			else
			{
				if (Stride->start(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}
			}

			//TODO: Here we should create incremental task : /here
			tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

			if (save_thread.joinable()) save_thread.join();

			if (bCfgSaveResultsIntoFile) {
				if (Stride->end_for_save() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}
			}
			else
			{
				if (Stride->end() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}
			}
			

			if (bCfgSaveResultsIntoFile) {
				save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
				//tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
			}

			tools::checkResult(Data->host.ret);

			float delay;
			tools::stop_time_and_calc_sec(&delay);
			std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
				<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
				<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
				<< " | ROUND: " << step;

		}//for (step)

	}

	std::cout << "\n\nEND!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.


	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}


	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;


}
bool ApplyConfig(ConfigClass& Config)
{
	try {

		for (int x = 0; x < MAX_ADAPTIVE_BASE_POSITIONS; x++) {
			host_AdaptiveBaseCurrentBatchInitialDigits[x] = 0;
		}

		parse_config(&Config, "config.cfg");

		std::vector<std::string> startFrom = tools::SplitWords(Config.static_words_starting_point);


		int nLastKnownPos = -1;
		std::vector<int> validIndexListPerPos[NUM_WORDS_MNEMONIC];

		for (int nemoIter = 0; nemoIter < NUM_WORDS_MNEMONIC; nemoIter++) {
			int16_t thisPosBipStarting;
			std::string thisPosStartFromWord = startFrom[nemoIter];
			tools::GetSingleWordIndex(thisPosStartFromWord, &thisPosBipStarting);
			//int16_t thisPosDicStarting = -1;



			std::vector<std::string> thisMnemoPosDictionaryLine = tools::SplitWords(Config.dynamic_words[nemoIter]);
			int thisPosLineWordCount = thisMnemoPosDictionaryLine.size();

			if (thisPosLineWordCount > MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION) {
				std::cout << "ERROR: Maximum Allowed word count per line is " << MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION << std::endl;
				//goto Error;
				return false;

			}

			int64_t adaptivePositionIdx = nemoIter - MAX_ADAPTIVE_BASE_POSITIONS;

			if (thisPosLineWordCount == 1) { //find consequtive count of single-word dictionaries
				int prev = nemoIter - 1;
				if (prev == nLastKnownPos)
					nLastKnownPos = nemoIter;
			}


			for (int16_t thisPosDictTraverseIdx = 0; thisPosDictTraverseIdx < thisPosLineWordCount; thisPosDictTraverseIdx++) {

				std::string thisWord = thisMnemoPosDictionaryLine[thisPosDictTraverseIdx];


				//Fill the digit-space for each adaptive base position (last 6 in our case)
				int16_t thisBipIdx;
				tools::GetSingleWordIndex(thisWord, &thisBipIdx);

				if (thisPosDictTraverseIdx == 0) {//leave old algorithm working for now with separated positions					
					Config.words_indicies_mnemonic[nemoIter] = thisBipIdx; //or even -1 when ? 
				}

				if (adaptivePositionIdx < 0)
					break;

				//FROM now on, we are on the second 6 words

				host_AdaptiveBaseDigitSet[adaptivePositionIdx][thisPosDictTraverseIdx] = thisBipIdx; //scrutinize what we do with -1 instances
				host_AdaptiveBaseDigitCarryTrigger[adaptivePositionIdx] = thisPosLineWordCount; //TODO: scrutinize (minus one needed?)

				//Check if we are going to start from this word, make adjustments and print info messages
				bool bStartsFromThisWord = (0 == strcmp(thisWord.c_str(), thisPosStartFromWord.c_str()));
				if (!bStartsFromThisWord)
					continue;

				//FROM now on, we start from this word

				std::ostringstream isAdaptiveStr;

				isAdaptiveStr.str("");

				host_AdaptiveBaseCurrentBatchInitialDigits[adaptivePositionIdx] = thisPosDictTraverseIdx;

				//std::cout << "SETTING " << adaptivePositionIdx << " @" << thisPosDictTraverseIdx << std::endl;

				if (adaptivePositionIdx >= 0) {
					isAdaptiveStr << "[Dynamic:" << thisPosLineWordCount << "]";
				}
				else if (thisPosLineWordCount == 1) {
					isAdaptiveStr.str("[STATIC]");
				}

				std::cout << "Postition " << nemoIter << isAdaptiveStr.str() << " starts from word: " << thisWord << " at PosDictionary: " << thisPosDictTraverseIdx << " BIP: " << thisBipIdx << " and carries at:" << host_AdaptiveBaseDigitCarryTrigger[nemoIter] << std::endl;

			}//single dictionary in each position
		} //nemo positions

		if (nLastKnownPos >= 0)
			std::cout << "Words up to position " << nLastKnownPos << " (out of 0 to 11) are known" << std::endl;
		else
			std::cout << "All words are dynamic" << std::endl;

		for (int pp = 0; pp < MAX_ADAPTIVE_BASE_POSITIONS; pp++) {
			std::cout << "The position:" << pp + MAX_ADAPTIVE_BASE_POSITIONS << " Carries at:" << host_AdaptiveBaseDigitCarryTrigger[pp] << " and starts from " << host_AdaptiveBaseCurrentBatchInitialDigits[pp] << std::endl;
		}


	}//try
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}//catch
	return true;
}
__host__ __device__
void PrintNextMnemo(uint64_t batchMnemo[2] , uint64_t nHowMuch, int16_t carry [MAX_ADAPTIVE_BASE_POSITIONS]
	, int16_t initDigits[MAX_ADAPTIVE_BASE_POSITIONS]
	, int16_t digitSet [MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
)
{
	int16_t  batchDigits[6];
	//uint64_t batchMnemo[2];
	//batchMnemo[0] = host_EntropyAbsolutePrefix64[0];
	//batchMnemo[1] = host_EntropyBatchNext24[0] & 0xB0000000; //scrutinize;
	printf("before->after::[%llu] == \n", nHowMuch  );

	if (IncrementAdaptiveDigits(carry, initDigits, nHowMuch, batchDigits) == false) {
		printf("Not able to add %llu\r\n", nHowMuch);
	}

	for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++)
		printf("[ %d,  %d ] - ", initDigits[i] ,batchDigits[i]);

	AdaptiveUpdateMnemonicLow64(&batchMnemo[1]
		, digitSet
		, batchDigits);


	int16_t tmp2[12] = {
		(batchMnemo[0] >> 53) & 2047,
		(batchMnemo[0] >> 42) & 2047,
		(batchMnemo[0] >> 31) & 2047,
		(batchMnemo[0] >> 20) & 2047,
		(batchMnemo[0] >> 9) & 2047,
		((batchMnemo[0] & ((1 << 9) - 1)) << 2) | ((batchMnemo[1] >> 62) & 3),
		(batchMnemo[1] >> 51) & 2047,
		(batchMnemo[1] >> 40) & 2047,
		(batchMnemo[1] >> 29) & 2047,
		(batchMnemo[1] >> 18) & 2047,
		(batchMnemo[1] >> 7) & 2047,
		((batchMnemo[1] & ((1 << 7) - 1)) << 4)

	};

	int16_t temArr[6] = {
		digitSet[0][batchDigits[0]]
		,	digitSet[1][batchDigits[1]]
		,	digitSet[2][batchDigits[2]]
		,	digitSet[3][batchDigits[3]]
		,	digitSet[4][batchDigits[4]]
		,	digitSet[5][batchDigits[5]] };
	printf ("Stars from 2nd half [%llu] --> %s\r\n", nHowMuch , tools::GetMnemoString(temArr, 6).c_str() );
	printf ("Fully last checksum: [%llu] --> %s\r\n" ,nHowMuch, tools::GetMnemoString(tmp2, 12).c_str());
}








