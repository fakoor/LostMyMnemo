﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		28-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "AdaptiveBase.h"

#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"






static std::thread save_thread;

int Generate_Mnemonic(void)
{


	std::cout << "Compile on Date **** : " << __DATE__ << ", Time:" << __TIME__ << std::endl;
	//{//TODO make all NULL
	//	dev_nProcessedFromBatch = NULL;
	//	host_nProcessedFromBatch = NULL;
	//	dev_nProcessedMoreThanBatch = NULL;
	//	host_nProcessedMoreThanBatch = NULL;
	//}


	hipError_t cudaStatus = hipSuccess;
	int err;
	ConfigClass Config;
	try {

		for (int x = 0; x < MAX_ADAPTIVE_BASE_POSITIONS; x++) {
			host_AdaptiveBaseCurrentBatchInitialDigits[x] = 0;
			std::cout << "ZERO:" << x << std::endl;
		}

		parse_config(&Config, "config.cfg");

		std::vector<std::string> startFrom = tools::SplitWords(Config.static_words_starting_point);


		int nLastKnownPos = -1;
		std::vector<int> validIndexListPerPos[NUM_WORDS_MNEMONIC];

		for (int nemoIter = 0; nemoIter < NUM_WORDS_MNEMONIC; nemoIter++) {
			int16_t thisPosBipStarting;
			std::string thisPosStartFromWord = startFrom[nemoIter];
			tools::GetSingleWordIndex(thisPosStartFromWord, &thisPosBipStarting);
			int16_t thisPosDicStarting = -1;



			std::vector<std::string> thisPos = tools::SplitWords(Config.dynamic_words[nemoIter]);
			int thisPosDictCount = thisPos.size();

			if (thisPosDictCount > MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION) {
				std::cout << "ERROR: Maximum Allowed word count per line is " << MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION <<std::endl;
				goto Error;

			}

			int64_t adaptivePortionIdx = nemoIter - MAX_ADAPTIVE_BASE_POSITIONS;

			if (thisPosDictCount == 1) { //find consequtive count of single-word dictionaries
				int prev = nemoIter - 1;
				if (prev == nLastKnownPos)
					nLastKnownPos = nemoIter;
			}


			for (int16_t thisDicIdx = 0; thisDicIdx < thisPosDictCount; thisDicIdx++) {
				
				std::string thisWord = thisPos[thisDicIdx];


				//Fill the digit-space for each adaptive base position (last 6 in our case)
				int16_t thisBipIdx;
				tools::GetSingleWordIndex(thisWord, &thisBipIdx);

				if (thisDicIdx == 0) {//leave old algorithm working for now with separated positions					
					Config.words_indicies_mnemonic[nemoIter] = thisBipIdx;
				}

				if (adaptivePortionIdx < 0)
					break;

				//FROM now on, we are on the second 6 words

				host_AdaptiveBaseDigitSet[adaptivePortionIdx][thisDicIdx] = thisBipIdx;
				host_AdaptiveBaseDigitCarryTrigger[adaptivePortionIdx] = thisPosDictCount; //TODO: scrutinize (minus one needed?)

				//Check if we are going to start from this word, make adjustments and print info messages
				bool bStartsFromThisWord = (0 == strcmp(thisWord.c_str(), thisPosStartFromWord.c_str()));
				if (!bStartsFromThisWord)
					continue;

				//FROM now on, we start from this word

				std::ostringstream isAdaptiveStr;

				isAdaptiveStr.str("");

				host_AdaptiveBaseCurrentBatchInitialDigits[adaptivePortionIdx] = thisDicIdx;

				std::cout << "SETTING " << adaptivePortionIdx << " @" << thisDicIdx << std::endl;

				if (adaptivePortionIdx >= 0) {
					isAdaptiveStr << "[Dynamic:" << thisPosDictCount << "]";
				}
				else if (thisPosDictCount == 1) {
					isAdaptiveStr.str("[STATIC]");
				}

				std::cout << "Postition " << nemoIter << isAdaptiveStr.str() << " starts from word: " << thisWord << " at PosDictionary: " << thisDicIdx << " BIP: " << thisBipIdx << std::endl;

			}//single dictionary in each position
		} //nemo positions

		if (nLastKnownPos >= 0)
			std::cout << "Words up to position " << nLastKnownPos << " are known" << std::endl;
		else
			std::cout << "All words are dynamic" << std::endl;

		for (int pp = 0; pp < MAX_ADAPTIVE_BASE_POSITIONS; pp++) {
			std::cout << "The position:" << pp + MAX_ADAPTIVE_BASE_POSITIONS << " Carries at:" << host_AdaptiveBaseDigitCarryTrigger[pp] << " and starts from " << host_AdaptiveBaseCurrentBatchInitialDigits[pp] << std::endl;
		}


		uint64_t number_of_generated_mnemonics = (Config.number_of_generated_mnemonics / (Config.cuda_block * Config.cuda_grid)) * (Config.cuda_block * Config.cuda_grid);
		if ((Config.number_of_generated_mnemonics % (Config.cuda_block * Config.cuda_grid)) != 0) number_of_generated_mnemonics += Config.cuda_block * Config.cuda_grid;
		Config.number_of_generated_mnemonics = number_of_generated_mnemonics;	
	}//try
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}//catch


	int nDevCount = devicesInfo();



	uint32_t num_device = 0;
	if (nDevCount != 1) { //select only cuda device automatically
#ifndef TEST_MODE
		std::cout << "\n\nEnter number of device: ";
		std::cin >> num_device;
#endif //TEST_MODE
	}
	else {
		std::cout << " The only CUDA capable device selected automatically." << std::endl;
	}
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = Config.cuda_grid * Config.cuda_block;
	if (num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE)
	{
		std::cerr << "Error num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE!" << std::endl;
		return -1;
	}
	uint32_t num_bytes = 0;
	if (Config.chech_equal_bytes_in_adresses == "yes")
	{
#ifdef TEST_MODE
		num_bytes = 5;
#else
		num_bytes = 8;
#endif //TEST_MODE
	}

	std::cout << "\nNUM WALLETS IN PACKET GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
	data_class* Data = new data_class();
	stride_class* Stride = new stride_class(Data);
	size_t num_addresses_in_tables = 0;

	{//TODO: allocate batch memory
		//if (hipMalloc((uint8_t**)&dev_nProcessedFromBatch, sizeof(uint64_t)) != hipSuccess) return -1;
		//if (hipMalloc((uint8_t**)&dev_nProcessedMoreThanBatch, sizeof(uint64_t)) != hipSuccess) return -1;

		//if (hipHostMalloc((void**)&host_nProcessedFromBatch, sizeof(uint64_t)) != hipSuccess) return -1;
		//if (hipHostMalloc((void**)&host_nProcessedMoreThanBatch, sizeof(uint64_t)) != hipSuccess) return -1;
		//std::cout << "Batch memory initied to " << dev_nProcessedFromBatch <<"," << dev_nProcessedMoreThanBatch<<"," << host_nProcessedFromBatch<<"," << host_nProcessedMoreThanBatch<<"." << std::endl;
		//*host_nProcessedFromBatch = 0;
		//*host_nProcessedMoreThanBatch = 0;
	}


	std::cout << "READ TABLES! WAIT..." << std::endl;
	tools::clearFiles();
	if((Config.generate_path[0] != 0) || (Config.generate_path[1] != 0) || (Config.generate_path[2] != 0) || (Config.generate_path[3] != 0) || (Config.generate_path[4] != 0)
		|| (Config.generate_path[5] != 0))
	{
		std::cout << "READ TABLES LEGACY(BIP32, BIP44)..." << std::endl;
	err = tools::readAllTables(Data->host.tables_legacy, Config.folder_tables_legacy, "", &num_addresses_in_tables);
	if (err == -1) {
		std::cerr << "Error readAllTables legacy!" << std::endl;
		goto Error;
	}
	}

	bool bCfgSaveResultsIntoFile = (Config.save_generation_result_in_file == "yes")?true:false;
	bool bCfgUseOldMethod = (Config.use_old_random_method == "yes")?true:false;


	if (bCfgUseOldMethod) {
		if ((Config.generate_path[6] != 0) || (Config.generate_path[7] != 0))
		{
			std::cout << "READ TABLES SEGWIT(BIP49)..." << std::endl;
			err = tools::readAllTables(Data->host.tables_segwit, Config.folder_tables_segwit, "", &num_addresses_in_tables);
			if (err == -1) {
				std::cerr << "Error readAllTables segwit!" << std::endl;
				goto Error;
			}
		}
		if ((Config.generate_path[8] != 0) || (Config.generate_path[9] != 0))
		{
			std::cout << "READ TABLES NATIVE SEGWIT(BIP84)..." << std::endl;
			err = tools::readAllTables(Data->host.tables_native_segwit, Config.folder_tables_native_segwit, "", &num_addresses_in_tables);
			if (err == -1) {
				std::cerr << "Error readAllTables native segwit!" << std::endl;
				goto Error;
			}
		}
		std::cout << std::endl << std::endl;

		if (num_addresses_in_tables == 0) {
			std::cerr << "ERROR READ TABLES!! NO ADDRESSES IN FILES!!" << std::endl;
			goto Error;
		}
	}

	if (Data->malloc(Config.cuda_grid, Config.cuda_block, Config.num_paths, Config.num_child_addresses, bCfgSaveResultsIntoFile) != 0) {
		std::cerr << "Error Data->malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		std::cerr << "Error INIT!!" << std::endl;
		goto Error;
	}

	Data->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "PATH: " << std::endl;

	if (bCfgUseOldMethod) {
		if (Config.generate_path[0] != 0) std::cout << "m/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[1] != 0) std::cout << "m/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[2] != 0) std::cout << "m/0/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[3] != 0) std::cout << "m/0/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	}
	if (Config.generate_path[4] != 0) std::cout << "m/44'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;

	if (bCfgUseOldMethod) {
		if (Config.generate_path[5] != 0) std::cout << "m/44'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[6] != 0) std::cout << "m/49'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[7] != 0) std::cout << "m/49'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[8] != 0) std::cout << "m/84'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[9] != 0) std::cout << "m/84'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	}
	std::cout << "\nGENERATE " << tools::formatWithCommas(Config.number_of_generated_mnemonics) << " MNEMONICS. " << tools::formatWithCommas(Config.number_of_generated_mnemonics * Data->num_all_childs) << " ADDRESSES. MNEMONICS IN ROUNDS " << tools::formatWithCommas(Data->wallets_in_round_gpu) << ". WAIT...\n\n";

	//TODO: Here we should create incremental task: /or here
	tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to num_bytes_find failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_generate_path), &Config.generate_path, sizeof(Config.generate_path), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_generate_path failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_childs), &Config.num_child_addresses, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_child failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_paths), &Config.num_paths, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_paths failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_static_words_indices), &Config.words_indicies_mnemonic, 12*2, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_gen_words_indices failed!" << std::endl;
		goto Error;
	}


	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS];
	uint64_t trunkInitEntropy[2];
	uint8_t reqChecksum;

	//TODO: fill host_EntropyAbsolutePrefix64 and host_EntropyBatchNext24
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] = 0;
	host_EntropyBatchNext24[PTR_AVOIDER] = 0;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[0]) << 53;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[1]) << 42;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[2]) << 31;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[3]) << 20;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[4]) << 9;
								 
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config.words_indicies_mnemonic[5]) >> 2;
	host_EntropyBatchNext24[PTR_AVOIDER]      |= (uint64_t)(Config.words_indicies_mnemonic[5]) << 62; //two bits from main 6 words

	if (NewTrunkPrefix() == false)
		goto Error;
	


	AdaptiveDigitsToEntropy(
		  host_AdaptiveBaseCurrentBatchInitialDigits
		, host_AdaptiveBaseDigitCarryTrigger
		, host_AdaptiveBaseDigitSet
		, host_EntropyAbsolutePrefix64
		, host_EntropyBatchNext24
		, host_AdaptiveBaseCurrentBatchInitialDigits
		, trunkInitEntropy, 
		  &reqChecksum);

	if (trunkInitEntropy[0] == host_EntropyAbsolutePrefix64[0]) {
		std::cout << "Init Entropy Sucessfully initialized by higher bits "<< trunkInitEntropy[0] << std::endl;
		if (host_EntropyBatchNext24[0] == trunkInitEntropy[1]) {
			std::cout << "Init Entropy Sucessfully tested for lower bits " << trunkInitEntropy[1] << std::endl;

		}
	}

	host_EntropyBatchNext24[0] &= 0xFFFFFF0000000000ULL; //test done, revert nack to only 24 msbs

	size_t copySize;
	hipError_t cudaResult;

	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyAbsolutePrefix64), host_EntropyAbsolutePrefix64, copySize, 0, hipMemcpyHostToDevice);
	if ( cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying "<< copySize <<" bytes to dev_EntropyAbsolutePrefix64 failed!: " << cudaResult << std::endl;
		goto Error;
	}


	const int elemSize = sizeof(host_AdaptiveBaseCurrentBatchInitialDigits[0]);
	copySize = elemSize * MAX_ADAPTIVE_BASE_POSITIONS;

	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), host_AdaptiveBaseCurrentBatchInitialDigits, copySize , 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
		goto Error;
	}
	copySize = sizeof(host_AdaptiveBaseDigitCarryTrigger[0]) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitCarryTrigger), host_AdaptiveBaseDigitCarryTrigger, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseDigitCarryTrigger failed!: " << cudaResult << std::endl;
		goto Error;
	}

	copySize = sizeof(host_AdaptiveBaseDigitSet[0][0]) * MAX_ADAPTIVE_BASE_POSITIONS * MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitSet), host_AdaptiveBaseDigitSet, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "dev_AdaptiveBaseCurrentBatchInitialDigits copying " << copySize << " bytes to dev_AdaptiveBaseDigitSet failed!: " << cudaResult << std::endl;
		goto Error;
	}


	//Initial zeroing
	//host_nProcessedFromBatch[0] = 0;
	//host_nProcessedMoreThanBatch[0] = 0;




	if (bCfgUseOldMethod == false){
		int nPlanned24BitTrunks = host_AdaptiveBaseDigitCarryTrigger[0] * host_AdaptiveBaseDigitCarryTrigger[1];

		uint64_t nPrevBatchProcessed = 0;

		uint64_t nPlanned44BitCombos = host_AdaptiveBaseDigitCarryTrigger[2]
			* host_AdaptiveBaseDigitCarryTrigger[3]
			* host_AdaptiveBaseDigitCarryTrigger[4]
			* host_AdaptiveBaseDigitCarryTrigger[5];

		uint64_t nUniversalMax = nPlanned24BitTrunks * nPlanned44BitCombos;
		uint64_t nUniversalProcessed = 0;
		uint64_t nThreadsInBatch = Config.cuda_block * Config.cuda_grid;
		uint64_t nBatchMax = nPlanned44BitCombos / nThreadsInBatch;
		
		if (nBatchMax * nThreadsInBatch < nPlanned44BitCombos)
			nBatchMax++;

		uint64_t nCumulativeCombosProcessedInTrunk = 0;
		int nBatch = 0;

		std::cout << "Planing to check total "<< nUniversalMax <<" combinations structured in " << nPlanned24BitTrunks << " Trunks X " << nPlanned44BitCombos << " Subordinates via " << nBatchMax << " batches "
			" of " << nThreadsInBatch << " threads each" << std::endl;


		//Config.number_of_generated_mnemonics / (Data->wallets_in_round_gpu)
		for (uint64_t nTrunk = 0; nTrunk < nPlanned24BitTrunks; nTrunk++)
		{
			std::cout << "> NEW TRUNK -- " << "No:" << nTrunk << "/" << nPlanned24BitTrunks - 1 << std::endl;

			if (NewTrunkPrefix() == false)
				goto Error;

			int16_t batchDigits[MAX_ADAPTIVE_BASE_POSITIONS];
			uint64_t batchMnemo[2];

			nBatch = 0;

			batchMnemo[0] = host_EntropyAbsolutePrefix64[0];
			batchMnemo[1] = host_EntropyBatchNext24[0] & 0xB;
			IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger, host_AdaptiveBaseCurrentBatchInitialDigits, 0, batchDigits);

			AdaptiveUpdateMnemonicLow64(&batchMnemo[1]
				, host_AdaptiveBaseDigitSet
				, batchDigits);


			int16_t tmp2[12] = {
(batchMnemo[0] >> 53) & 2047,
(batchMnemo[0] >> 42) & 2047,
(batchMnemo[0] >> 31) & 2047,
(batchMnemo[0] >> 20) & 2047,
(batchMnemo[0] >> 9) & 2047,
((batchMnemo[0] & ((1 << 9) - 1)) << 2) | ((batchMnemo[1] >> 62) & 3),
(batchMnemo[1] >> 51) & 2047,
(batchMnemo[1] >> 40) & 2047,
(batchMnemo[1] >> 29) & 2047,
(batchMnemo[1] >> 18) & 2047,
(batchMnemo[1] >> 7) & 2047,
((batchMnemo[1] & ((1 << 7) - 1)) << 4)

			};

			int16_t temArr[6] = {
		host_AdaptiveBaseDigitSet[0][batchDigits[0]]
	,	host_AdaptiveBaseDigitSet[1][batchDigits[1]]
	,	host_AdaptiveBaseDigitSet[2][batchDigits[2]]
	,	host_AdaptiveBaseDigitSet[3][batchDigits[3]]
	,	host_AdaptiveBaseDigitSet[4][batchDigits[4]]
	,	host_AdaptiveBaseDigitSet[5][batchDigits[5]] };

			for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++) {
				std::cout << host_AdaptiveBaseCurrentBatchInitialDigits[i] << "=" << batchDigits[i] << std::endl;
			}
			do  {


				//TODO: increment entropy here accordingto grid , processed and extra


				
				//tools::entropyTo12Words(batchMnemo, 
	
	
				std::cout << ">> NEW BATCH -- "
					<< "No:" << nBatch << "/" << nBatchMax << std::endl;
				std::cout << "Stars from 2nd half:" << tools::GetMnemoString(temArr, 6) << std::endl;
					std::cout <<"Fully:"<< tools::GetMnemoString(tmp2, 12) << std::endl;
				*Data->host.host_nProcessedFromBatch = 0;
				*Data->host.host_nProcessedMoreThanBatch = 0;


				tools::start_time();

				if (Stride->startDictionaryAttack(Config.cuda_grid, Config.cuda_block, Data->host.host_nProcessedFromBatch, Data->host.host_nProcessedMoreThanBatch) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}

				//TODO: Handled by dictionary attack with index
				//tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));
				std::cout << "Waiting for batch " << nBatch << " to finish." << std::endl;
				if (save_thread.joinable()) save_thread.join();

				if (Stride->endDictionaryAttack() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}

				if (bCfgSaveResultsIntoFile) {
					save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
					//tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
				}

				tools::checkResult(Data->host.ret);

				float delay;
				tools::stop_time_and_calc_sec(&delay);
				//std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
				//	<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
				//	<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
				//	<< " | ROUND: " << nTrunk;

				nPrevBatchProcessed = Data->host.host_nProcessedFromBatch[PTR_AVOIDER]
					+ Data->host.host_nProcessedMoreThanBatch[PTR_AVOIDER];
				std::cout << ">>>This batch (#" << nBatch << ") completed processing " << nPrevBatchProcessed << " combos." << std::endl;
				nCumulativeCombosProcessedInTrunk += nPrevBatchProcessed;

				IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger, host_AdaptiveBaseCurrentBatchInitialDigits, nPrevBatchProcessed, batchDigits);
				//memcpy(&host_AdaptiveBaseCurrentBatchInitialDigits[0], &batchDigits[0], sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS)
				for (int x = 0; x < MAX_ADAPTIVE_BASE_POSITIONS; x++) {
					host_AdaptiveBaseCurrentBatchInitialDigits[x] = batchDigits[x];
				}

				nBatch++;
			} while (nCumulativeCombosProcessedInTrunk < nPlanned44BitCombos); //batch
			nUniversalProcessed += nCumulativeCombosProcessedInTrunk;

			std::cout << ">>This Trunk (#" << nTrunk << ") completed processing " << nCumulativeCombosProcessedInTrunk<<"/"<< nUniversalProcessed <<" current combinations" << std::endl;
			nCumulativeCombosProcessedInTrunk = 0;

		}//trunk
	}
	else {
		for (uint64_t step = 0; step < Config.number_of_generated_mnemonics / (Data->wallets_in_round_gpu); step++)
		{
			tools::start_time();

			if (bCfgSaveResultsIntoFile) {
				if (Stride->start_for_save(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}
			}
			else
			{
				if (Stride->start(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}
			}

			//TODO: Here we should create incremental task : /here
			tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

			if (save_thread.joinable()) save_thread.join();

			if (bCfgSaveResultsIntoFile) {
				if (Stride->end_for_save() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}
			}
			else
			{
				if (Stride->end() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}
			}
			

			if (bCfgSaveResultsIntoFile) {
				save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
				//tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
			}

			tools::checkResult(Data->host.ret);

			float delay;
			tools::stop_time_and_calc_sec(&delay);
			std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
				<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
				<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
				<< " | ROUND: " << step;

		}//for (step)

	}

	std::cout << "\n\nEND!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.


	{//TODO: Free Memory
		//hipFree(dev_nProcessedFromBatch);
		//hipFree(dev_nProcessedMoreThanBatch);
		//hipHostFree(host_nProcessedFromBatch);
		//hipHostFree(host_nProcessedMoreThanBatch);
	}



	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}


	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;


}

bool NewTrunkPrefix()
{
	AdaptiveUpdateMnemonicLow64(host_EntropyBatchNext24
		, host_AdaptiveBaseDigitSet
		, host_AdaptiveBaseCurrentBatchInitialDigits);

	host_EntropyBatchNext24[0] &= 0xFFFFFF00000000;
	size_t copySize = sizeof(uint64_t);
	hipError_t cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyBatchNext24), host_EntropyBatchNext24, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyBatchNext24 failed!: " << cudaResult << std::endl;
		return false;
	}
	return true;
}

__host__ __device__
void AdaptiveUpdateMnemonicLow64(uint64_t* low64
	, int16_t digitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
	, int16_t curDigits [MAX_ADAPTIVE_BASE_POSITIONS]
)

{
	uint64_t tmpHigh = *low64;
	
	*low64 = tmpHigh >> 62;
	*low64 = *low64 << 2;

	for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS-1; i++) {
		*low64 = *low64 << 11;
		*low64 & 0xFFFFFFFFFFFFF7F;
		*low64 |= (uint64_t)(digitSet[i][curDigits[i]]);
	}
	*low64 = *low64 << 7;
	*low64 |= ((uint64_t)(digitSet[MAX_ADAPTIVE_BASE_POSITIONS - 1][curDigits[MAX_ADAPTIVE_BASE_POSITIONS - 1]]) >> 4);
}







