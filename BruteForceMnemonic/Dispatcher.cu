﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		28-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"




static std::thread save_thread;

int Generate_Mnemonic(void)
{
	std::cout << "Compile on Date: " << __DATE__ << ", Time:" << __TIME__ << std::endl;

	hipError_t cudaStatus = hipSuccess;
	int err;
	ConfigClass Config;
	try {
		parse_config(&Config, "config.cfg");

		std::vector<std::string> startFrom = tools::SplitWords(Config.static_words_starting_point);


		int nLastKnownPos = -1;
		std::vector<int> validIndexListPerPos[NUM_WORDS_MNEMONIC];

		for (int i = 0; i < NUM_WORDS_MNEMONIC; i++) {
			int16_t thisPosBipStarting;
			std::string thisPosStartFromWord = startFrom[i];
			tools::GetSingleWordIndex(thisPosStartFromWord, &thisPosBipStarting);
			int16_t thisPosDicStarting = -1;



			std::vector<std::string> thisPos = tools::SplitWords(Config.dynamic_words[i]);
			int thisPosDictCount = thisPos.size();

			for (int thisDicIdx = 0; thisDicIdx < thisPosDictCount; thisDicIdx++) {
				
				std::string thisWord = thisPos[thisDicIdx];


				//Fill the digit-space for each adaptive base position (last 6 in our case)
				int16_t thisBipIdx;
				tools::GetSingleWordIndex(thisWord, &thisBipIdx);

				int64_t last6Index = i - MAX_ADAPTIVE_BASE_POSITIONS;
				if (last6Index >= 0) {
					dev_AdaptiveBaseDigitSet[last6Index][thisDicIdx] = thisBipIdx;
				}

				//leave old algorithm working for now
				Config.words_indicies_mnemonic[i] = thisBipIdx;

				//Check if we are going to start from this word, make adjustments and print info messages
				bool bStartsFromThisWord = (0 == strcmp(thisWord.c_str(), thisPosStartFromWord.c_str()));
				if (!bStartsFromThisWord)
					continue;

				

				std::ostringstream isAdaptiveStr;

				isAdaptiveStr.str("");

				if (last6Index >= 0) {
					dev_AdaptiveBaseCurrentBatchInitialDigits[last6Index] = thisDicIdx;
					isAdaptiveStr << "[Dynamic:" << thisPosDictCount << "]";
				}
				else if (thisPosDictCount == 1) {
					isAdaptiveStr.str("[STATIC]");
				}

				std::cout << "Postition " << i << isAdaptiveStr.str() << " starts from word: " << thisWord << " at PosDictionary: " << thisDicIdx << " BIP: " << thisBipIdx << std::endl;

				if (thisPosDictCount == 1) { //match in a single-word dictionary
					int prev = i - 1;
					if (prev == nLastKnownPos && thisBipIdx >= 0)
						nLastKnownPos = i;
				}			
			}
		}

		if (nLastKnownPos >= 0)
			std::cout << "Words up to position " << nLastKnownPos << " are known" << std::endl;
		else
			std::cout << "All words are dynamic" << std::endl;



		uint64_t number_of_generated_mnemonics = (Config.number_of_generated_mnemonics / (Config.cuda_block * Config.cuda_grid)) * (Config.cuda_block * Config.cuda_grid);
		if ((Config.number_of_generated_mnemonics % (Config.cuda_block * Config.cuda_grid)) != 0) number_of_generated_mnemonics += Config.cuda_block * Config.cuda_grid;
		Config.number_of_generated_mnemonics = number_of_generated_mnemonics;	
	}//try
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}//catch


	devicesInfo();



	uint32_t num_device = 0;
#ifndef TEST_MODE
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //TEST_MODE
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = Config.cuda_grid * Config.cuda_block;
	if (num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE)
	{
		std::cerr << "Error num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE!" << std::endl;
		return -1;
	}
	uint32_t num_bytes = 0;
	if (Config.chech_equal_bytes_in_adresses == "yes")
	{
#ifdef TEST_MODE
		num_bytes = 5;
#else
		num_bytes = 8;
#endif //TEST_MODE
	}

	std::cout << "\nNUM WALLETS IN PACKET GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
	data_class* Data = new data_class();
	stride_class* Stride = new stride_class(Data);
	size_t num_addresses_in_tables = 0;


	std::cout << "READ TABLES! WAIT..." << std::endl;
	tools::clearFiles();
	if((Config.generate_path[0] != 0) || (Config.generate_path[1] != 0) || (Config.generate_path[2] != 0) || (Config.generate_path[3] != 0) || (Config.generate_path[4] != 0)
		|| (Config.generate_path[5] != 0))
	{
		std::cout << "READ TABLES LEGACY(BIP32, BIP44)..." << std::endl;
	err = tools::readAllTables(Data->host.tables_legacy, Config.folder_tables_legacy, "", &num_addresses_in_tables);
	if (err == -1) {
		std::cerr << "Error readAllTables legacy!" << std::endl;
		goto Error;
	}
	}

	bool bCfgSaveResultsIntoFile = (Config.save_generation_result_in_file == "yes")?true:false;
	bool bCfgUseOldMethod = (Config.use_old_random_method == "yes")?true:false;


	if (bCfgUseOldMethod) {
		if ((Config.generate_path[6] != 0) || (Config.generate_path[7] != 0))
		{
			std::cout << "READ TABLES SEGWIT(BIP49)..." << std::endl;
			err = tools::readAllTables(Data->host.tables_segwit, Config.folder_tables_segwit, "", &num_addresses_in_tables);
			if (err == -1) {
				std::cerr << "Error readAllTables segwit!" << std::endl;
				goto Error;
			}
		}
		if ((Config.generate_path[8] != 0) || (Config.generate_path[9] != 0))
		{
			std::cout << "READ TABLES NATIVE SEGWIT(BIP84)..." << std::endl;
			err = tools::readAllTables(Data->host.tables_native_segwit, Config.folder_tables_native_segwit, "", &num_addresses_in_tables);
			if (err == -1) {
				std::cerr << "Error readAllTables native segwit!" << std::endl;
				goto Error;
			}
		}
		std::cout << std::endl << std::endl;

		if (num_addresses_in_tables == 0) {
			std::cerr << "ERROR READ TABLES!! NO ADDRESSES IN FILES!!" << std::endl;
			goto Error;
		}
	}

	if (Data->malloc(Config.cuda_grid, Config.cuda_block, Config.num_paths, Config.num_child_addresses, bCfgSaveResultsIntoFile) != 0) {
		std::cerr << "Error Data->malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		std::cerr << "Error INIT!!" << std::endl;
		goto Error;
	}

	Data->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "PATH: " << std::endl;

	if (bCfgUseOldMethod) {
		if (Config.generate_path[0] != 0) std::cout << "m/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[1] != 0) std::cout << "m/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[2] != 0) std::cout << "m/0/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[3] != 0) std::cout << "m/0/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	}
	if (Config.generate_path[4] != 0) std::cout << "m/44'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;

	if (bCfgUseOldMethod) {
		if (Config.generate_path[5] != 0) std::cout << "m/44'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[6] != 0) std::cout << "m/49'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[7] != 0) std::cout << "m/49'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[8] != 0) std::cout << "m/84'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
		if (Config.generate_path[9] != 0) std::cout << "m/84'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	}
	std::cout << "\nGENERATE " << tools::formatWithCommas(Config.number_of_generated_mnemonics) << " MNEMONICS. " << tools::formatWithCommas(Config.number_of_generated_mnemonics * Data->num_all_childs) << " ADDRESSES. MNEMONICS IN ROUNDS " << tools::formatWithCommas(Data->wallets_in_round_gpu) << ". WAIT...\n\n";

	//TODO: Here we should create incremental task: /or here
	tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to num_bytes_find failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_generate_path), &Config.generate_path, sizeof(Config.generate_path), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_generate_path failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_childs), &Config.num_child_addresses, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_child failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_paths), &Config.num_paths, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_paths failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_static_words_indices), &Config.words_indicies_mnemonic, 12*2, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_gen_words_indices failed!" << std::endl;
		goto Error;
	}


	if (bCfgUseOldMethod == false){
		std::cout << "Using NEW method (bCfgUseOldMethod=" << bCfgUseOldMethod<< ")." << std::endl;

		for (uint64_t step = 0; step < Config.number_of_generated_mnemonics / (Data->wallets_in_round_gpu); step++)
		{
			tools::start_time();

			if (Stride->startDictionaryAttack(Config.cuda_grid, Config.cuda_block) != 0) {
				std::cerr << "Error START!!" << std::endl;
				goto Error;
			}

			//TODO: Here we should create incremental task : /here
			tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

			if (save_thread.joinable()) save_thread.join();

			if (Stride->endDictionaryAttack() != 0) {
				std::cerr << "Error END!!" << std::endl;
				goto Error;
			}

			if (bCfgSaveResultsIntoFile) {
				save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
				//tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
			}

			tools::checkResult(Data->host.ret);

			float delay;
			tools::stop_time_and_calc_sec(&delay);
			std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
				<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
				<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
				<< " | ROUND: " << step;

		}//for (step)
	}
	else {
		for (uint64_t step = 0; step < Config.number_of_generated_mnemonics / (Data->wallets_in_round_gpu); step++)
		{
			tools::start_time();

			if (bCfgSaveResultsIntoFile) {
				if (Stride->start_for_save(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}
			}
			else
			{
				if (Stride->start(Config.cuda_grid, Config.cuda_block) != 0) {
					std::cerr << "Error START!!" << std::endl;
					goto Error;
				}
			}

			//TODO: Here we should create incremental task : /here
			tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

			if (save_thread.joinable()) save_thread.join();

			if (bCfgSaveResultsIntoFile) {
				if (Stride->end_for_save() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}
			}
			else
			{
				if (Stride->end() != 0) {
					std::cerr << "Error END!!" << std::endl;
					goto Error;
				}
			}
			

			if (bCfgSaveResultsIntoFile) {
				save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
				//tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
			}

			tools::checkResult(Data->host.ret);

			float delay;
			tools::stop_time_and_calc_sec(&delay);
			std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
				<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
				<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
				<< " | ROUND: " << step;

		}//for (step)

	}

	std::cout << "\n\nEND!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}







