﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		28-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
//#include <synchapi.h>

#include <Dispatcher.h>
#include <thread>
#include "EntropyTools.cuh"

int main()
{
    printf("Compiled on Date ** : %s , time: %s \r\n", __DATE__, __TIME__);
    std::this_thread::sleep_for(std::chrono::seconds(1));

    int ret = Generate_Mnemonic();


    if (ret !=0) {
        printf("Some errors ocurre during program execution, see hints in messages above, fix and re-run the program.\r\n");

        std::this_thread::sleep_for(std::chrono::seconds(2));
    }
    else {

        printf("Program Completed Successfully.\r\n");
    }
    return 0;
}

