#include <stdafx.h>

#include <iostream>
#include <thread>

#include "hip/hip_runtime.h"

#include "DispatchDictionaryScan.cuh"
#include "DictionaryScanner.cuh"

#include "consts.h"
#include "AdaptiveBase.h"


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "Helper.h"
#include "EntropyTools.cuh"

bool  DispatchDictionaryScan(ConfigClass* Config, data_class* Data, stride_class* Stride) {

	if (InitalSync(Config) == false)
		return false;



	uint64_t nProblemPower =
		(uint64_t)host_AdaptiveBaseDigitCarryTrigger[0]
		* host_AdaptiveBaseDigitCarryTrigger[1]
		* host_AdaptiveBaseDigitCarryTrigger[2]
		* host_AdaptiveBaseDigitCarryTrigger[3]
		* host_AdaptiveBaseDigitCarryTrigger[4]
		* host_AdaptiveBaseDigitCarryTrigger[5];


	uint64_t nSolverThreads = Config->cuda_block * Config->cuda_grid;
	uint64_t nIterationPower = nSolverThreads * host_AdaptiveBaseDigitCarryTrigger[5];
	uint64_t nIterationsNeeded = nProblemPower / nIterationPower;

	if (nIterationsNeeded * nIterationPower < nProblemPower)
		nIterationsNeeded++;



	std::cout << "-- Starting Dictionary SCAN -- " << std::endl;

	std::cout << " Going to dispatch " << nProblemPower << " total COMBOs"
		<< " via " << nIterationsNeeded << " iterations "
		" (each able to process " << nIterationPower << " instances)." << std::endl;

	uint64_t nUniversalProcessed = 0;


	uint64_t nBatchMax = 1;

	int nBatch = 0;


	
	size_t copySize;
	hipError_t cudaResult;

	//uint64_t nMasterIteration = 0;
	*Data->host.nProcessedInstances = 0;
	*Data->host.nProcessedIterations = 0;

	if (hipSuccess != hipMemcpy(Data->dev.nProcessedInstances, Data->host.nProcessedInstances, 8, hipMemcpyHostToDevice)) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}


	do
	{
		//Set Master Iteration
		if (hipSuccess != hipMemcpy(Data->dev.nProcessedIterations, Data->host.nProcessedIterations, 8, hipMemcpyHostToDevice)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		//Zero Previous Count
		*Data->host.nProcessedInstances = 0;
		if (hipSuccess != hipMemcpy( Data->dev.nProcessedInstances, Data->host.nProcessedInstances, 8, hipMemcpyHostToDevice)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}


		printf("Iteration: %llu started.\r\n", *Data->host.nProcessedIterations);



		tools::start_time();

		if (Stride->startDictionaryAttack(Config->cuda_grid, Config->cuda_block) != 0) {
			std::cerr << "Error START!!" << std::endl;
			return false;
		}


		if (Stride->endDictionaryAttack() != 0) {
			std::cerr << "Error END!!" << std::endl;
			return false;
		}

		//if (bCfgSaveResultsIntoFile) {
		//	save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config->generate_path);
		//}



		if (hipSuccess != hipMemcpy(Data->host.nProcessedInstances, Data->dev.nProcessedInstances, 8, hipMemcpyDeviceToHost)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		printf("Checking results of %llu checkups.\r\n", *Data->host.nProcessedInstances);
		tools::checkResult(Data->host.ret);

		float delay;
		tools::stop_time_and_calc_sec(&delay);
		//std::cout << std::endl << "PROCESSED: at " << tools::formatPrefix((double)*Data->host.nProcessedInstances / delay) << " COMBO/Sec" << std::endl;

		std::cout << "Iteration " << *Data->host.nProcessedIterations
			<< " completed we have processed  " << *Data->host.nProcessedInstances << " COMBOs  at " << tools::formatPrefix((double)*Data->host.nProcessedInstances / delay) << " COMBO/Sec" << std::endl;
		++*Data->host.nProcessedIterations;
	} while (*Data->host.nProcessedIterations < nIterationsNeeded);//trunk
	return true;
}


bool InitalSync(ConfigClass* Config)
{
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] = 0;
	host_EntropyNextPrefix2[PTR_AVOIDER] = 0;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[0]) << 53;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[1]) << 42;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[2]) << 31;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[3]) << 20;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[4]) << 9;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[5]) >> 2;
	host_EntropyNextPrefix2[PTR_AVOIDER] = (uint64_t)(Config->words_indicies_mnemonic[5]) << 62; //two bits from main 6 words


	size_t copySize;
	hipError_t cudaResult;

	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyAbsolutePrefix64), host_EntropyAbsolutePrefix64, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyAbsolutePrefix64 failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyNextPrefix2), host_EntropyNextPrefix2, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyBatchNext24 failed!: " << cudaResult << std::endl;
		return false;
	}


	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS * MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitSet), host_AdaptiveBaseDigitSet, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "dev_AdaptiveBaseCurrentBatchInitialDigits copying " << copySize << " bytes to dev_AdaptiveBaseDigitSet failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(host_AdaptiveBaseDigitCarryTrigger[0]) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitCarryTrigger), host_AdaptiveBaseDigitCarryTrigger, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseDigitCarryTrigger failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), host_AdaptiveBaseCurrentBatchInitialDigits, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
		return false;
	}

	return true;
}
