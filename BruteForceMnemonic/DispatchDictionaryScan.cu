#include "hip/hip_runtime.h"
#include <stdafx.h>

#include <iostream>
#include <thread>

#include "hip/hip_runtime.h"

#include "DispatchDictionaryScan.cuh"
#include "DictionaryScanner.cuh"

#include "consts.h"
#include "AdaptiveBase.h"


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "Helper.h"

#include <windows.h> //some beeping fancey
#include <mmsystem.h>

#define _USE_MATH_DEFINES
#include <cmath>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#include "EntropyTools.cuh"









//// A simple 1-second sine wave sound (440 Hz)
//const unsigned char soundDataSimple[] = {
//	0x52, 0x49, 0x46, 0x46, 0x24, 0x08, 0x00, 0x00, // RIFF header
//	0x57, 0x41, 0x56, 0x45, 0x66, 0x6d, 0x74, 0x20, // WAVE header
//	0x10, 0x00, 0x00, 0x00, 0x01, 0x00, 0x01, 0x00, // Format chunk
//	0x44, 0xac, 0x00, 0x00, 0x10, 0x00, 0x00, 0x00, // 44.1kHz, 16-bit
//	0x64, 0x61, 0x74, 0x61, 0x00, 0x08, 0x00, 0x00, // Data chunk header
//	// Actual audio data (440 Hz sine wave)
//	0x00, 0x00, 0x1d, 0x00, 0x38, 0x00, 0x4f, 0x00,
//	0x5e, 0x00, 0x64, 0x00, 0x68, 0x00, 0x6a, 0x00,
//	0x68, 0x00, 0x64, 0x00, 0x5e, 0x00, 0x4f, 0x00,
//	0x38, 0x00, 0x1d, 0x00, 0x00, 0x00, 0xe3, 0xff,
//	0xc8, 0xff, 0xa0, 0xff, 0x8f, 0xff, 0x7c, 0xff,
//	0x6f, 0xff, 0x68, 0xff, 0x68, 0xff, 0x70, 0xff,
//	0x7c, 0xff, 0x8f, 0xff, 0xa0, 0xff, 0xc8, 0xff,
//	0xe3, 0xff, 0x00, 0x00, 0x1d, 0x00, 0x38, 0x00,
//};

//void playWavFromMemory(const unsigned char* data, size_t size) {
//	HWAVEOUT hWaveOut;
//	WAVEFORMATEX wfx;
//
//	// Set up the WAVEFORMATEX structure
//	wfx.wFormatTag = WAVE_FORMAT_PCM;
//	wfx.nChannels = 1; // Mono
//	wfx.nSamplesPerSec = 44100; // Sample rate
//	wfx.wBitsPerSample = 16; // Bits per sample
//	wfx.nBlockAlign = (wfx.nChannels * wfx.wBitsPerSample) / 8;
//	wfx.nAvgBytesPerSec = wfx.nSamplesPerSec * wfx.nBlockAlign;
//
//	// Open the wave output device
//	waveOutOpen(&hWaveOut, WAVE_MAPPER, &wfx, 0, 0, CALLBACK_NULL);
//
//	// Prepare the wave header
//	WAVEHDR whdr;
//	whdr.lpData = (LPSTR)data; // Pointer to the data
//	whdr.dwBufferLength = (DWORD)size; // Size of the data
//	whdr.dwFlags = 0;
//
//	// Prepare and write the header
//	waveOutPrepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
//	waveOutWrite(hWaveOut, &whdr, sizeof(WAVEHDR));
//
//	// Wait for the sound to finish playing
//	while (!(whdr.dwFlags & WHDR_DONE)) {
//		Sleep(100);
//	}
//
//	// Clean up
//	waveOutUnprepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
//	waveOutClose(hWaveOut);
//}



const int SAMPLE_RATE = 44100;
const int DURATION = 1; // 1 second
const int FREQUENCY = 440; // Frequency of the sine wave (A4 note)

const int NUM_SAMPLES = SAMPLE_RATE * DURATION;
const int BYTE_RATE = SAMPLE_RATE * 2; // 16 bits = 2 bytes per sample

// Generating stereo sine wave data
unsigned char soundDataSine[NUM_SAMPLES * 4]; // 2 channels (stereo), 2 bytes per sample


void generateSineWave() {
	for (int i = 0; i < NUM_SAMPLES; i++) {
		// Calculate the sample value
		int16_t sample = static_cast<int16_t>(32767 * sin((2.0 * M_PI * FREQUENCY * i) / SAMPLE_RATE));

		// Fill left channel
		soundDataSine[i * 4] = (sample & 0xFF);          // Low byte
		soundDataSine[i * 4 + 1] = (sample >> 8) & 0xFF; // High byte

		// Fill right channel (same value for stereo effect)
		soundDataSine[i * 4 + 2] = (sample & 0xFF);      // Low byte
		soundDataSine[i * 4 + 3] = (sample >> 8) & 0xFF; // High byte
	}
}

void playSineWavFromMemory(const unsigned char* data, size_t size) {
	HWAVEOUT hWaveOut;
	WAVEFORMATEX wfx;

	// Set up the WAVEFORMATEX structure
	wfx.wFormatTag = WAVE_FORMAT_PCM;
	wfx.nChannels = 2; // Stereo
	wfx.nSamplesPerSec = SAMPLE_RATE; // Sample rate
	wfx.wBitsPerSample = 16; // Bits per sample
	wfx.nBlockAlign = (wfx.nChannels * wfx.wBitsPerSample) / 8;
	wfx.nAvgBytesPerSec = wfx.nSamplesPerSec * wfx.nBlockAlign;

	// Open the wave output device
	waveOutOpen(&hWaveOut, WAVE_MAPPER, &wfx, 0, 0, CALLBACK_NULL);

	// Prepare the wave header
	WAVEHDR whdr;
	whdr.lpData = (LPSTR)data; // Pointer to the data
	whdr.dwBufferLength = (DWORD)size; // Size of the data
	whdr.dwFlags = 0;

	// Prepare and write the header
	waveOutPrepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
	waveOutWrite(hWaveOut, &whdr, sizeof(WAVEHDR));

	// Wait for the sound to finish playing
	while (!(whdr.dwFlags & WHDR_DONE)) {
		Sleep(100);
	}

	// Clean up
	waveOutUnprepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
	waveOutClose(hWaveOut);
}


void playAlert() {
	//printf("Playing sound...\r\n");
	//Beep(2000, 1000);

	generateSineWave(); // Fill soundData with sine wave
	playSineWavFromMemory(soundDataSine, sizeof(soundDataSine));

	//playWavFromMemory(soundDataSimple, sizeof(soundDataSimple));
	//int frequencies[] = { 800, 1000, 1200, 1000 }; // Frequencies in Hz
	//int durations[] = { 300, 300, 300, 400 }; // Durations in milliseconds

	//for (int i = 0; i < sizeof(frequencies) / sizeof(frequencies[0]); ++i) {
	//	Beep(frequencies[i], durations[i]);
	//	Sleep(50);
	//}
}

bool  DispatchDictionaryScan(ConfigClass* Config, data_class* Data, stride_class* Stride) {

	if (InitalSync(Config) == false)
		return false;



	uint64_t nProblemPower =
		(uint64_t)host_AdaptiveBaseDigitCarryTrigger[0]
		* host_AdaptiveBaseDigitCarryTrigger[1]
		* host_AdaptiveBaseDigitCarryTrigger[2]
		* host_AdaptiveBaseDigitCarryTrigger[3]
		* host_AdaptiveBaseDigitCarryTrigger[4]
		* host_AdaptiveBaseDigitCarryTrigger[5];


	uint64_t nSolverThreads = Config->cuda_block * Config->cuda_grid;
	uint64_t nIterationPower = nSolverThreads * host_AdaptiveBaseDigitCarryTrigger[5];
	uint64_t nIterationsNeeded = nProblemPower / nIterationPower;

	if (nIterationsNeeded * nIterationPower < nProblemPower)
		nIterationsNeeded++;



	std::cout << "-- Starting Dictionary SCAN -- " << std::endl;

	std::cout << " Going to dispatch " << nProblemPower << " total COMBOs"
		<< " via " << nIterationsNeeded << " iterations "
		" (each able to process " << nIterationPower << " instances)." << std::endl;



	uint64_t nBatchMax = 1;

	int nBatch = 0;


	
	size_t copySize;
	hipError_t cudaResult;

	//uint64_t nMasterIteration = 0;
	*Data->host.nProcessedInstances = 0;
	*Data->host.nProcessedIterations = 0;

	host_retEntropy[0] = 0ui64;
	host_retEntropy[1] = 0ui64;


	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dev_retEntropy), host_retEntropy, 16)) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}





	if (hipSuccess != hipMemcpy(Data->dev.nProcessedInstances, Data->host.nProcessedInstances, 8, hipMemcpyHostToDevice)) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}
	const int nMnemoShowLen = MAX_ADAPTIVE_BASE_POSITIONS * 9 + MAX_ADAPTIVE_BASE_POSITIONS;
	char strMnemoShow[nMnemoShowLen] = {0};
	int16_t digitShow[MAX_ADAPTIVE_BASE_POSITIONS];
	uint64_t nUniversalProcessed = 0;

	do
	{
		//Set Master Iteration
		if (hipSuccess != hipMemcpy(Data->dev.nProcessedIterations, Data->host.nProcessedIterations, 8, hipMemcpyHostToDevice)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		//Zero Previous Count
		*Data->host.nProcessedInstances = 0;
		if (hipSuccess != hipMemcpy( Data->dev.nProcessedInstances, Data->host.nProcessedInstances, 8, hipMemcpyHostToDevice)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		printf("Iteration: %llu started.\r\n", *Data->host.nProcessedIterations + 1);
		IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger, host_AdaptiveBaseCurrentBatchInitialDigits
			, nUniversalProcessed, digitShow);
		ShowAdaptiveStr(host_AdaptiveBaseDigitSet,digitShow, strMnemoShow);
		printf("<FROM> * * * * * *\t %s </FROM> (%llu)\r\n", strMnemoShow, nUniversalProcessed+1);




		if (Stride->startDictionaryAttack(Config->cuda_grid, Config->cuda_block) != 0) {
			std::cerr << "Error START!!" << std::endl;
			return false;
		}
		tools::start_time();


		float delay;
		if (Stride->endDictionaryAttack() != 0) {
			std::cerr << "Error END!!" << std::endl;
			return false;
		}
		tools::stop_time_and_calc_sec(&delay);

		//if (bCfgSaveResultsIntoFile) {
		//	save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config->generate_path);
		//}



		if (hipSuccess != hipMemcpy(Data->host.nProcessedInstances, Data->dev.nProcessedInstances, 8, hipMemcpyDeviceToHost)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		nUniversalProcessed += *Data->host.nProcessedInstances;
		//printf("\t\t\t.\r\n\t\t\t.\r\n\t\t\t.\r\n");
		IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger, host_AdaptiveBaseCurrentBatchInitialDigits
			, nUniversalProcessed-1, digitShow);
		ShowAdaptiveStr(host_AdaptiveBaseDigitSet, digitShow, strMnemoShow);

		printf("<UPTO> * * * * * * \t %s </UPTO> (%llu)\r\n", strMnemoShow, nUniversalProcessed);

		printf("Checking results of %llu checkups.\r\n", *Data->host.nProcessedInstances);


		//std::cout << std::endl << "PROCESSED: at " << tools::formatPrefix((double)*Data->host.nProcessedInstances / delay) << " COMBO/Sec" << std::endl;


		std::cout << "Iteration " << *Data->host.nProcessedIterations
			<< " completed we have processed  " << *Data->host.nProcessedInstances << " COMBOs  at " << tools::formatPrefix((double)*Data->host.nProcessedInstances / delay) << " COMBO/Sec" << std::endl;


		if (hipSuccess != hipMemcpyFromSymbol(host_retEntropy, HIP_SYMBOL(dev_retEntropy), 16)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		if (host_retEntropy[0] != 0 || host_retEntropy[1] != 0) {
			printf("Entropy found: %llX-%llX\r\n", host_retEntropy[0] , host_retEntropy[1]);
			uint8_t disp[121];
			GetAllWords(host_retEntropy, disp);
			printf("------------------------------------------------------------------------------------------\r\n");
			printf("|\t %s \t |\r\n", disp);
			printf("------------------------------------------------------------------------------------------\r\n");
			playAlert();
			break;

		}

#if 0
		if (DictionaryCheckFound(Data->host.ret)) {
			tools::checkResult(Data->host.ret);
		}
#endif //old method
		++*Data->host.nProcessedIterations;
	} while (*Data->host.nProcessedIterations < nIterationsNeeded);//trunk

	return true;
}


bool InitalSync(ConfigClass* Config)
{
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] = 0;
	host_EntropyNextPrefix2[PTR_AVOIDER] = 0;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[0]) << 53;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[1]) << 42;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[2]) << 31;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[3]) << 20;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[4]) << 9;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[5]) >> 2;
	host_EntropyNextPrefix2[PTR_AVOIDER] = (uint64_t)(Config->words_indicies_mnemonic[5]) << 62; //two bits from main 6 words


	size_t copySize;
	hipError_t cudaResult;


	copySize = sizeof(uint8_t) * 20;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_uniqueTargetAddressBytes), host_uniqueTargetAddressBytes, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_uniqueTargetAddressBytes failed!: " << cudaResult << std::endl;
		return false;
	}


	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyAbsolutePrefix64), host_EntropyAbsolutePrefix64, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyAbsolutePrefix64 failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyNextPrefix2), host_EntropyNextPrefix2, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyBatchNext24 failed!: " << cudaResult << std::endl;
		return false;
	}


	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS * MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitSet), host_AdaptiveBaseDigitSet, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "dev_AdaptiveBaseCurrentBatchInitialDigits copying " << copySize << " bytes to dev_AdaptiveBaseDigitSet failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(host_AdaptiveBaseDigitCarryTrigger[0]) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitCarryTrigger), host_AdaptiveBaseDigitCarryTrigger, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseDigitCarryTrigger failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), host_AdaptiveBaseCurrentBatchInitialDigits, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
		return false;
	}

	return true;
}
