#include "hip/hip_runtime.h"
#include <stdafx.h>

#include <iostream>
#include <thread>

#include "hip/hip_runtime.h"

#include "DispatchDictionaryScan.cuh"
#include "DictionaryScanner.cuh"

#include "consts.h"
#include "AdaptiveBase.h"


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "Helper.h"
#include "EntropyTools.cuh"

#include <windows.h> //some beeping fancey
#include <mmsystem.h>

#define _USE_MATH_DEFINES
#include <cmath>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif


// 18 kB
const uint8_t arrBipWords[2048][9] = { "abandon", "ability", "able", "about", "above", "absent", "absorb", "abstract", "absurd","abuse", "access", "accident", "account", "accuse", "achieve", "acid", "acoustic", "acquire","across", "act", "action", "actor", "actress", "actual", "adapt", "add", "addict", "address","adjust", "admit", "adult", "advance", "advice", "aerobic", "affair", "afford", "afraid","again", "age", "agent", "agree", "ahead", "aim", "air", "airport", "aisle", "alarm", "album","alcohol", "alert", "alien", "all", "alley", "allow", "almost", "alone", "alpha", "already","also", "alter", "always", "amateur", "amazing", "among", "amount", "amused", "analyst","anchor", "ancient", "anger", "angle", "angry", "animal", "ankle", "announce", "annual","another", "answer", "antenna", "antique", "anxiety", "any", "apart", "apology", "appear","apple", "approve", "april", "arch", "arctic", "area", "arena", "argue", "arm", "armed","armor", "army", "around", "arrange", "arrest", "arrive", "arrow", "art", "artefact", "artist","artwork", "ask", "aspect", "assault", "asset", "assist", "assume", "asthma", "athlete","atom", "attack", "attend", "attitude", "attract", "auction", "audit", "august", "aunt","author", "auto", "autumn", "average", "avocado", "avoid", "awake", "aware", "away", "awesome","awful", "awkward", "axis", "baby", "bachelor", "bacon", "badge", "bag", "balance", "balcony","ball", "bamboo", "banana", "banner", "bar", "barely", "bargain", "barrel", "base", "basic","basket", "battle", "beach", "bean", "beauty", "because", "become", "beef", "before", "begin","behave", "behind", "believe", "below", "belt", "bench", "benefit", "best", "betray", "better","between", "beyond", "bicycle", "bid", "bike", "bind", "biology", "bird", "birth", "bitter","black", "blade", "blame", "blanket", "blast", "bleak", "bless", "blind", "blood", "blossom","blouse", "blue", "blur", "blush", "board", "boat", "body", "boil", "bomb", "bone", "bonus","book", "boost", "border", "boring", "borrow", "boss", "bottom", "bounce", "box", "boy","bracket", "brain", "brand", "brass", "brave", "bread", "breeze", "brick", "bridge", "brief","bright", "bring", "brisk", "broccoli", "broken", "bronze", "broom", "brother", "brown","brush", "bubble", "buddy", "budget", "buffalo", "build", "bulb", "bulk", "bullet", "bundle","bunker", "burden", "burger", "burst", "bus", "business", "busy", "butter", "buyer", "buzz","cabbage", "cabin", "cable", "cactus", "cage", "cake", "call", "calm", "camera", "camp", "can","canal", "cancel", "candy", "cannon", "canoe", "canvas", "canyon", "capable", "capital","captain", "car", "carbon", "card", "cargo", "carpet", "carry", "cart", "case", "cash","casino", "castle", "casual", "cat", "catalog", "catch", "category", "cattle", "caught","cause", "caution", "cave", "ceiling", "celery", "cement", "census", "century", "cereal","certain", "chair", "chalk", "champion", "change", "chaos", "chapter", "charge", "chase","chat", "cheap", "check", "cheese", "chef", "cherry", "chest", "chicken", "chief", "child","chimney", "choice", "choose", "chronic", "chuckle", "chunk", "churn", "cigar", "cinnamon","circle", "citizen", "city", "civil", "claim", "clap", "clarify", "claw", "clay", "clean","clerk", "clever", "click", "client", "cliff", "climb", "clinic", "clip", "clock", "clog","close", "cloth", "cloud", "clown", "club", "clump", "cluster", "clutch", "coach", "coast","coconut", "code", "coffee", "coil", "coin", "collect", "color", "column", "combine", "come","comfort", "comic", "common", "company", "concert", "conduct", "confirm", "congress","connect", "consider", "control", "convince", "cook", "cool", "copper", "copy", "coral","core", "corn", "correct", "cost", "cotton", "couch", "country", "couple", "course", "cousin","cover", "coyote", "crack", "cradle", "craft", "cram", "crane", "crash", "crater", "crawl","crazy", "cream", "credit", "creek", "crew", "cricket", "crime", "crisp", "critic", "crop","cross", "crouch", "crowd", "crucial", "cruel", "cruise", "crumble", "crunch", "crush", "cry","crystal", "cube", "culture", "cup", "cupboard", "curious", "current", "curtain", "curve","cushion", "custom", "cute", "cycle", "dad", "damage", "damp", "dance", "danger", "daring","dash", "daughter", "dawn", "day", "deal", "debate", "debris", "decade", "december", "decide","decline", "decorate", "decrease", "deer", "defense", "define", "defy", "degree", "delay","deliver", "demand", "demise", "denial", "dentist", "deny", "depart", "depend", "deposit","depth", "deputy", "derive", "describe", "desert", "design", "desk", "despair", "destroy","detail", "detect", "develop", "device", "devote", "diagram", "dial", "diamond", "diary","dice", "diesel", "diet", "differ", "digital", "dignity", "dilemma", "dinner", "dinosaur","direct", "dirt", "disagree", "discover", "disease", "dish", "dismiss", "disorder", "display","distance", "divert", "divide", "divorce", "dizzy", "doctor", "document", "dog", "doll","dolphin", "domain", "donate", "donkey", "donor", "door", "dose", "double", "dove", "draft","dragon", "drama", "drastic", "draw", "dream", "dress", "drift", "drill", "drink", "drip","drive", "drop", "drum", "dry", "duck", "dumb", "dune", "during", "dust", "dutch", "duty","dwarf", "dynamic", "eager", "eagle", "early", "earn", "earth", "easily", "east", "easy","echo", "ecology", "economy", "edge", "edit", "educate", "effort", "egg", "eight", "either","elbow", "elder", "electric", "elegant", "element", "elephant", "elevator", "elite", "else","embark", "embody", "embrace", "emerge", "emotion", "employ", "empower", "empty", "enable","enact", "end", "endless", "endorse", "enemy", "energy", "enforce", "engage", "engine","enhance", "enjoy", "enlist", "enough", "enrich", "enroll", "ensure", "enter", "entire","entry", "envelope", "episode", "equal", "equip", "era", "erase", "erode", "erosion", "error","erupt", "escape", "essay", "essence", "estate", "eternal", "ethics", "evidence", "evil","evoke", "evolve", "exact", "example", "excess", "exchange", "excite", "exclude", "excuse","execute", "exercise", "exhaust", "exhibit", "exile", "exist", "exit", "exotic", "expand","expect", "expire", "explain", "expose", "express", "extend", "extra", "eye", "eyebrow","fabric", "face", "faculty", "fade", "faint", "faith", "fall", "false", "fame", "family","famous", "fan", "fancy", "fantasy", "farm", "fashion", "fat", "fatal", "father", "fatigue","fault", "favorite", "feature", "february", "federal", "fee", "feed", "feel", "female","fence", "festival", "fetch", "fever", "few", "fiber", "fiction", "field", "figure", "file","film", "filter", "final", "find", "fine", "finger", "finish", "fire", "firm", "first","fiscal", "fish", "fit", "fitness", "fix", "flag", "flame", "flash", "flat", "flavor", "flee","flight", "flip", "float", "flock", "floor", "flower", "fluid", "flush", "fly", "foam","focus", "fog", "foil", "fold", "follow", "food", "foot", "force", "forest", "forget", "fork","fortune", "forum", "forward", "fossil", "foster", "found", "fox", "fragile", "frame","frequent", "fresh", "friend", "fringe", "frog", "front", "frost", "frown", "frozen", "fruit","fuel", "fun", "funny", "furnace", "fury", "future", "gadget", "gain", "galaxy", "gallery","game", "gap", "garage", "garbage", "garden", "garlic", "garment", "gas", "gasp", "gate","gather", "gauge", "gaze", "general", "genius", "genre", "gentle", "genuine", "gesture","ghost", "giant", "gift", "giggle", "ginger", "giraffe", "girl", "give", "glad", "glance","glare", "glass", "glide", "glimpse", "globe", "gloom", "glory", "glove", "glow", "glue","goat", "goddess", "gold", "good", "goose", "gorilla", "gospel", "gossip", "govern", "gown","grab", "grace", "grain", "grant", "grape", "grass", "gravity", "great", "green", "grid","grief", "grit", "grocery", "group", "grow", "grunt", "guard", "guess", "guide", "guilt","guitar", "gun", "gym", "habit", "hair", "half", "hammer", "hamster", "hand", "happy","harbor", "hard", "harsh", "harvest", "hat", "have", "hawk", "hazard", "head", "health","heart", "heavy", "hedgehog", "height", "hello", "helmet", "help", "hen", "hero", "hidden","high", "hill", "hint", "hip", "hire", "history", "hobby", "hockey", "hold", "hole", "holiday","hollow", "home", "honey", "hood", "hope", "horn", "horror", "horse", "hospital", "host","hotel", "hour", "hover", "hub", "huge", "human", "humble", "humor", "hundred", "hungry","hunt", "hurdle", "hurry", "hurt", "husband", "hybrid", "ice", "icon", "idea", "identify","idle", "ignore", "ill", "illegal", "illness", "image", "imitate", "immense", "immune","impact", "impose", "improve", "impulse", "inch", "include", "income", "increase", "index","indicate", "indoor", "industry", "infant", "inflict", "inform", "inhale", "inherit","initial", "inject", "injury", "inmate", "inner", "innocent", "input", "inquiry", "insane","insect", "inside", "inspire", "install", "intact", "interest", "into", "invest", "invite","involve", "iron", "island", "isolate", "issue", "item", "ivory", "jacket", "jaguar", "jar","jazz", "jealous", "jeans", "jelly", "jewel", "job", "join", "joke", "journey", "joy", "judge","juice", "jump", "jungle", "junior", "junk", "just", "kangaroo", "keen", "keep", "ketchup","key", "kick", "kid", "kidney", "kind", "kingdom", "kiss", "kit", "kitchen", "kite", "kitten","kiwi", "knee", "knife", "knock", "know", "lab", "label", "labor", "ladder", "lady", "lake","lamp", "language", "laptop", "large", "later", "latin", "laugh", "laundry", "lava", "law","lawn", "lawsuit", "layer", "lazy", "leader", "leaf", "learn", "leave", "lecture", "left","leg", "legal", "legend", "leisure", "lemon", "lend", "length", "lens", "leopard", "lesson","letter", "level", "liar", "liberty", "library", "license", "life", "lift", "light", "like","limb", "limit", "link", "lion", "liquid", "list", "little", "live", "lizard", "load", "loan","lobster", "local", "lock", "logic", "lonely", "long", "loop", "lottery", "loud", "lounge","love", "loyal", "lucky", "luggage", "lumber", "lunar", "lunch", "luxury", "lyrics", "machine","mad", "magic", "magnet", "maid", "mail", "main", "major", "make", "mammal", "man", "manage","mandate", "mango", "mansion", "manual", "maple", "marble", "march", "margin", "marine","market", "marriage", "mask", "mass", "master", "match", "material", "math", "matrix","matter", "maximum", "maze", "meadow", "mean", "measure", "meat", "mechanic", "medal", "media","melody", "melt", "member", "memory", "mention", "menu", "mercy", "merge", "merit", "merry","mesh", "message", "metal", "method", "middle", "midnight", "milk", "million", "mimic", "mind","minimum", "minor", "minute", "miracle", "mirror", "misery", "miss", "mistake", "mix", "mixed","mixture", "mobile", "model", "modify", "mom", "moment", "monitor", "monkey", "monster","month", "moon", "moral", "more", "morning", "mosquito", "mother", "motion", "motor","mountain", "mouse", "move", "movie", "much", "muffin", "mule", "multiply", "muscle", "museum","mushroom", "music", "must", "mutual", "myself", "mystery", "myth", "naive", "name", "napkin","narrow", "nasty", "nation", "nature", "near", "neck", "need", "negative", "neglect","neither", "nephew", "nerve", "nest", "net", "network", "neutral", "never", "news", "next","nice", "night", "noble", "noise", "nominee", "noodle", "normal", "north", "nose", "notable","note", "nothing", "notice", "novel", "now", "nuclear", "number", "nurse", "nut", "oak","obey", "object", "oblige", "obscure", "observe", "obtain", "obvious", "occur", "ocean","october", "odor", "off", "offer", "office", "often", "oil", "okay", "old", "olive", "olympic","omit", "once", "one", "onion", "online", "only", "open", "opera", "opinion", "oppose","option", "orange", "orbit", "orchard", "order", "ordinary", "organ", "orient", "original","orphan", "ostrich", "other", "outdoor", "outer", "output", "outside", "oval", "oven", "over","own", "owner", "oxygen", "oyster", "ozone", "pact", "paddle", "page", "pair", "palace","palm", "panda", "panel", "panic", "panther", "paper", "parade", "parent", "park", "parrot","party", "pass", "patch", "path", "patient", "patrol", "pattern", "pause", "pave", "payment","peace", "peanut", "pear", "peasant", "pelican", "pen", "penalty", "pencil", "people","pepper", "perfect", "permit", "person", "pet", "phone", "photo", "phrase", "physical","piano", "picnic", "picture", "piece", "pig", "pigeon", "pill", "pilot", "pink", "pioneer","pipe", "pistol", "pitch", "pizza", "place", "planet", "plastic", "plate", "play", "please","pledge", "pluck", "plug", "plunge", "poem", "poet", "point", "polar", "pole", "police","pond", "pony", "pool", "popular", "portion", "position", "possible", "post", "potato","pottery", "poverty", "powder", "power", "practice", "praise", "predict", "prefer", "prepare","present", "pretty", "prevent", "price", "pride", "primary", "print", "priority", "prison","private", "prize", "problem", "process", "produce", "profit", "program", "project", "promote","proof", "property", "prosper", "protect", "proud", "provide", "public", "pudding", "pull","pulp", "pulse", "pumpkin", "punch", "pupil", "puppy", "purchase", "purity", "purpose","purse", "push", "put", "puzzle", "pyramid", "quality", "quantum", "quarter", "question","quick", "quit", "quiz", "quote", "rabbit", "raccoon", "race", "rack", "radar", "radio","rail", "rain", "raise", "rally", "ramp", "ranch", "random", "range", "rapid", "rare", "rate","rather", "raven", "raw", "razor", "ready", "real", "reason", "rebel", "rebuild", "recall","receive", "recipe", "record", "recycle", "reduce", "reflect", "reform", "refuse", "region","regret", "regular", "reject", "relax", "release", "relief", "rely", "remain", "remember","remind", "remove", "render", "renew", "rent", "reopen", "repair", "repeat", "replace","report", "require", "rescue", "resemble", "resist", "resource", "response", "result","retire", "retreat", "return", "reunion", "reveal", "review", "reward", "rhythm", "rib","ribbon", "rice", "rich", "ride", "ridge", "rifle", "right", "rigid", "ring", "riot", "ripple","risk", "ritual", "rival", "river", "road", "roast", "robot", "robust", "rocket", "romance","roof", "rookie", "room", "rose", "rotate", "rough", "round", "route", "royal", "rubber","rude", "rug", "rule", "run", "runway", "rural", "sad", "saddle", "sadness", "safe", "sail","salad", "salmon", "salon", "salt", "salute", "same", "sample", "sand", "satisfy", "satoshi","sauce", "sausage", "save", "say", "scale", "scan", "scare", "scatter", "scene", "scheme","school", "science", "scissors", "scorpion", "scout", "scrap", "screen", "script", "scrub","sea", "search", "season", "seat", "second", "secret", "section", "security", "seed", "seek","segment", "select", "sell", "seminar", "senior", "sense", "sentence", "series", "service","session", "settle", "setup", "seven", "shadow", "shaft", "shallow", "share", "shed", "shell","sheriff", "shield", "shift", "shine", "ship", "shiver", "shock", "shoe", "shoot", "shop","short", "shoulder", "shove", "shrimp", "shrug", "shuffle", "shy", "sibling", "sick", "side","siege", "sight", "sign", "silent", "silk", "silly", "silver", "similar", "simple", "since","sing", "siren", "sister", "situate", "six", "size", "skate", "sketch", "ski", "skill", "skin","skirt", "skull", "slab", "slam", "sleep", "slender", "slice", "slide", "slight", "slim","slogan", "slot", "slow", "slush", "small", "smart", "smile", "smoke", "smooth", "snack","snake", "snap", "sniff", "snow", "soap", "soccer", "social", "sock", "soda", "soft", "solar","soldier", "solid", "solution", "solve", "someone", "song", "soon", "sorry", "sort", "soul","sound", "soup", "source", "south", "space", "spare", "spatial", "spawn", "speak", "special","speed", "spell", "spend", "sphere", "spice", "spider", "spike", "spin", "spirit", "split","spoil", "sponsor", "spoon", "sport", "spot", "spray", "spread", "spring", "spy", "square","squeeze", "squirrel", "stable", "stadium", "staff", "stage", "stairs", "stamp", "stand","start", "state", "stay", "steak", "steel", "stem", "step", "stereo", "stick", "still","sting", "stock", "stomach", "stone", "stool", "story", "stove", "strategy", "street","strike", "strong", "struggle", "student", "stuff", "stumble", "style", "subject", "submit","subway", "success", "such", "sudden", "suffer", "sugar", "suggest", "suit", "summer", "sun","sunny", "sunset", "super", "supply", "supreme", "sure", "surface", "surge", "surprise","surround", "survey", "suspect", "sustain", "swallow", "swamp", "swap", "swarm", "swear","sweet", "swift", "swim", "swing", "switch", "sword", "symbol", "symptom", "syrup", "system","table", "tackle", "tag", "tail", "talent", "talk", "tank", "tape", "target", "task", "taste","tattoo", "taxi", "teach", "team", "tell", "ten", "tenant", "tennis", "tent", "term", "test","text", "thank", "that", "theme", "then", "theory", "there", "they", "thing", "this","thought", "three", "thrive", "throw", "thumb", "thunder", "ticket", "tide", "tiger", "tilt","timber", "time", "tiny", "tip", "tired", "tissue", "title", "toast", "tobacco", "today","toddler", "toe", "together", "toilet", "token", "tomato", "tomorrow", "tone", "tongue","tonight", "tool", "tooth", "top", "topic", "topple", "torch", "tornado", "tortoise", "toss","total", "tourist", "toward", "tower", "town", "toy", "track", "trade", "traffic", "tragic","train", "transfer", "trap", "trash", "travel", "tray", "treat", "tree", "trend", "trial","tribe", "trick", "trigger", "trim", "trip", "trophy", "trouble", "truck", "true", "truly","trumpet", "trust", "truth", "try", "tube", "tuition", "tumble", "tuna", "tunnel", "turkey","turn", "turtle", "twelve", "twenty", "twice", "twin", "twist", "two", "type", "typical","ugly", "umbrella", "unable", "unaware", "uncle", "uncover", "under", "undo", "unfair","unfold", "unhappy", "uniform", "unique", "unit", "universe", "unknown", "unlock", "until","unusual", "unveil", "update", "upgrade", "uphold", "upon", "upper", "upset", "urban", "urge","usage", "use", "used", "useful", "useless", "usual", "utility", "vacant", "vacuum", "vague","valid", "valley", "valve", "van", "vanish", "vapor", "various", "vast", "vault", "vehicle","velvet", "vendor", "venture", "venue", "verb", "verify", "version", "very", "vessel","veteran", "viable", "vibrant", "vicious", "victory", "video", "view", "village", "vintage","violin", "virtual", "virus", "visa", "visit", "visual", "vital", "vivid", "vocal", "voice","void", "volcano", "volume", "vote", "voyage", "wage", "wagon", "wait", "walk", "wall","walnut", "want", "warfare", "warm", "warrior", "wash", "wasp", "waste", "water", "wave","way", "wealth", "weapon", "wear", "weasel", "weather", "web", "wedding", "weekend", "weird","welcome", "west", "wet", "whale", "what", "wheat", "wheel", "when", "where", "whip","whisper", "wide", "width", "wife", "wild", "will", "win", "window", "wine", "wing", "wink","winner", "winter", "wire", "wisdom", "wise", "wish", "witness", "wolf", "woman", "wonder","wood", "wool", "word", "work", "world", "worry", "worth", "wrap", "wreck", "wrestle", "wrist","write", "wrong", "yard", "year", "yellow", "you", "young", "youth", "zebra", "zero", "zone","zoo" };
const uint8_t arrBipWordsLengths[2048] = { 7,7,4,5,5,6,6,8,6,5,6,8,7,6,7,4,8,7,6,3,6,5,7,6,5,3,6,7,6,5,5,7,6,7,6,6,6,5,3,5,5,5,3,3,7,5,5,5,7,5,5,3,5,5,6,5,5,7,4,5,6,7,7,5,6,6,7,6,7,5,5,5,6,5,8,6,7,6,7,7,7,3,5,7,6,5,7,5,4,6,4,5,5,3,5,5,4,6,7,6,6,5,3,8,6,7,3,6,7,5,6,6,6,7,4,6,6,8,7,7,5,6,4,6,4,6,7,7,5,5,5,4,7,5,7,4,4,8,5,5,3,7,7,4,6,6,6,3,6,7,6,4,5,6,6,5,4,6,7,6,4,6,5,6,6,7,5,4,5,7,4,6,6,7,6,7,3,4,4,7,4,5,6,5,5,5,7,5,5,5,5,5,7,6,4,4,5,5,4,4,4,4,4,5,4,5,6,6,6,4,6,6,3,3,7,5,5,5,5,5,6,5,6,5,6,5,5,8,6,6,5,7,5,5,6,5,6,7,5,4,4,6,6,6,6,6,5,3,8,4,6,5,4,7,5,5,6,4,4,4,4,6,4,3,5,6,5,6,5,6,6,7,7,7,3,6,4,5,6,5,4,4,4,6,6,6,3,7,5,8,6,6,5,7,4,7,6,6,6,7,6,7,5,5,8,6,5,7,6,5,4,5,5,6,4,6,5,7,5,5,7,6,6,7,7,5,5,5,8,6,7,4,5,5,4,7,4,4,5,5,6,5,6,5,5,6,4,5,4,5,5,5,5,4,5,7,6,5,5,7,4,6,4,4,7,5,6,7,4,7,5,6,7,7,7,7,8,7,8,7,8,4,4,6,4,5,4,4,7,4,6,5,7,6,6,6,5,6,5,6,5,4,5,5,6,5,5,5,6,5,4,7,5,5,6,4,5,6,5,7,5,6,7,6,5,3,7,4,7,3,8,7,7,7,5,7,6,4,5,3,6,4,5,6,6,4,8,4,3,4,6,6,6,8,6,7,8,8,4,7,6,4,6,5,7,6,6,6,7,4,6,6,7,5,6,6,8,6,6,4,7,7,6,6,7,6,6,7,4,7,5,4,6,4,6,7,7,7,6,8,6,4,8,8,7,4,7,8,7,8,6,6,7,5,6,8,3,4,7,6,6,6,5,4,4,6,4,5,6,5,7,4,5,5,5,5,5,4,5,4,4,3,4,4,4,6,4,5,4,5,7,5,5,5,4,5,6,4,4,4,7,7,4,4,7,6,3,5,6,5,5,8,7,7,8,8,5,4,6,6,7,6,7,6,7,5,6,5,3,7,7,5,6,7,6,6,7,5,6,6,6,6,6,5,6,5,8,7,5,5,3,5,5,7,5,5,6,5,7,6,7,6,8,4,5,6,5,7,6,8,6,7,6,7,8,7,7,5,5,4,6,6,6,6,7,6,7,6,5,3,7,6,4,7,4,5,5,4,5,4,6,6,3,5,7,4,7,3,5,6,7,5,8,7,8,7,3,4,4,6,5,8,5,5,3,5,7,5,6,4,4,6,5,4,4,6,6,4,4,5,6,4,3,7,3,4,5,5,4,6,4,6,4,5,5,5,6,5,5,3,4,5,3,4,4,6,4,4,5,6,6,4,7,5,7,6,6,5,3,7,5,8,5,6,6,4,5,5,5,6,5,4,3,5,7,4,6,6,4,6,7,4,3,6,7,6,6,7,3,4,4,6,5,4,7,6,5,6,7,7,5,5,4,6,6,7,4,4,4,6,5,5,5,7,5,5,5,5,4,4,4,7,4,4,5,7,6,6,6,4,4,5,5,5,5,5,7,5,5,4,5,4,7,5,4,5,5,5,5,5,6,3,3,5,4,4,6,7,4,5,6,4,5,7,3,4,4,6,4,6,5,5,8,6,5,6,4,3,4,6,4,4,4,3,4,7,5,6,4,4,7,6,4,5,4,4,4,6,5,8,4,5,4,5,3,4,5,6,5,7,6,4,6,5,4,7,6,3,4,4,8,4,6,3,7,7,5,7,7,6,6,6,7,7,4,7,6,8,5,8,6,8,6,7,6,6,7,7,6,6,6,5,8,5,7,6,6,6,7,7,6,8,4,6,6,7,4,6,7,5,4,5,6,6,3,4,7,5,5,5,3,4,4,7,3,5,5,4,6,6,4,4,8,4,4,7,3,4,3,6,4,7,4,3,7,4,6,4,4,5,5,4,3,5,5,6,4,4,4,8,6,5,5,5,5,7,4,3,4,7,5,4,6,4,5,5,7,4,3,5,6,7,5,4,6,4,7,6,6,5,4,7,7,7,4,4,5,4,4,5,4,4,6,4,6,4,6,4,4,7,5,4,5,6,4,4,7,4,6,4,5,5,7,6,5,5,6,6,7,3,5,6,4,4,4,5,4,6,3,6,7,5,7,6,5,6,5,6,6,6,8,4,4,6,5,8,4,6,6,7,4,6,4,7,4,8,5,5,6,4,6,6,7,4,5,5,5,5,4,7,5,6,6,8,4,7,5,4,7,5,6,7,6,6,4,7,3,5,7,6,5,6,3,6,7,6,7,5,4,5,4,7,8,6,6,5,8,5,4,5,4,6,4,8,6,6,8,5,4,6,6,7,4,5,4,6,6,5,6,6,4,4,4,8,7,7,6,5,4,3,7,7,5,4,4,4,5,5,5,7,6,6,5,4,7,4,7,6,5,3,7,6,5,3,3,4,6,6,7,7,6,7,5,5,7,4,3,5,6,5,3,4,3,5,7,4,4,3,5,6,4,4,5,7,6,6,6,5,7,5,8,5,6,8,6,7,5,7,5,6,7,4,4,4,3,5,6,6,5,4,6,4,4,6,4,5,5,5,7,5,6,6,4,6,5,4,5,4,7,6,7,5,4,7,5,6,4,7,7,3,7,6,6,6,7,6,6,3,5,5,6,8,5,6,7,5,3,6,4,5,4,7,4,6,5,5,5,6,7,5,4,6,6,5,4,6,4,4,5,5,4,6,4,4,4,7,7,8,8,4,6,7,7,6,5,8,6,7,6,7,7,6,7,5,5,7,5,8,6,7,5,7,7,7,6,7,7,7,5,8,7,7,5,7,6,7,4,4,5,7,5,5,5,8,6,7,5,4,3,6,7,7,7,7,8,5,4,4,5,6,7,4,4,5,5,4,4,5,5,4,5,6,5,5,4,4,6,5,3,5,5,4,6,5,7,6,7,6,6,7,6,7,6,6,6,6,7,6,5,7,6,4,6,8,6,6,6,5,4,6,6,6,7,6,7,6,8,6,8,8,6,6,7,6,7,6,6,6,6,3,6,4,4,4,5,5,5,5,4,4,6,4,6,5,5,4,5,5,6,6,7,4,6,4,4,6,5,5,5,5,6,4,3,4,3,6,5,3,6,7,4,4,5,6,5,4,6,4,6,4,7,7,5,7,4,3,5,4,5,7,5,6,6,7,8,8,5,5,6,6,5,3,6,6,4,6,6,7,8,4,4,7,6,4,7,6,5,8,6,7,7,6,5,5,6,5,7,5,4,5,7,6,5,5,4,6,5,4,5,4,5,8,5,6,5,7,3,7,4,4,5,5,4,6,4,5,6,7,6,5,4,5,6,7,3,4,5,6,3,5,4,5,5,4,4,5,7,5,5,6,4,6,4,4,5,5,5,5,5,6,5,5,4,5,4,4,6,6,4,4,4,5,7,5,8,5,7,4,4,5,4,4,5,4,6,5,5,5,7,5,5,7,5,5,5,6,5,6,5,4,6,5,5,7,5,5,4,5,6,6,3,6,7,8,6,7,5,5,6,5,5,5,5,4,5,5,4,4,6,5,5,5,5,7,5,5,5,5,8,6,6,6,8,7,5,7,5,7,6,6,7,4,6,6,5,7,4,6,3,5,6,5,6,7,4,7,5,8,8,6,7,7,7,5,4,5,5,5,5,4,5,6,5,6,7,5,6,5,6,3,4,6,4,4,4,6,4,5,6,4,5,4,4,3,6,6,4,4,4,4,5,4,5,4,6,5,4,5,4,7,5,6,5,5,7,6,4,5,4,6,4,4,3,5,6,5,5,7,5,7,3,8,6,5,6,8,4,6,7,4,5,3,5,6,5,7,8,4,5,7,6,5,4,3,5,5,7,6,5,8,4,5,6,4,5,4,5,5,5,5,7,4,4,6,7,5,4,5,7,5,5,3,4,7,6,4,6,6,4,6,6,6,5,4,5,3,4,7,4,8,6,7,5,7,5,4,6,6,7,7,6,4,8,7,6,5,7,6,6,7,6,4,5,5,5,4,5,3,4,6,7,5,7,6,6,5,5,6,5,3,6,5,7,4,5,7,6,6,7,5,4,6,7,4,6,7,6,7,7,7,5,4,7,7,6,7,5,4,5,6,5,5,5,5,4,7,6,4,6,4,5,4,4,4,6,4,7,4,7,4,4,5,5,4,3,6,6,4,6,7,3,7,7,5,7,4,3,5,4,5,5,4,5,4,7,4,5,4,4,4,3,6,4,4,4,6,6,4,6,4,4,7,4,5,6,4,4,4,4,5,5,5,4,5,7,5,5,5,4,4,6,3,5,5,5,4,4,3 };

void ShowAdaptiveStr(int16_t digs[MAX_ADAPTIVE_BASE_POSITIONS], char *str) {
	int16_t bipVal;
	uint16_t offset = 0;
	for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++) {
		bipVal = host_AdaptiveBaseDigitSet[i][digs[i]];
		sprintf(str+offset, "%s ", arrBipWords [bipVal]);
		offset += arrBipWordsLengths[bipVal] + 1;
	}
}



//// A simple 1-second sine wave sound (440 Hz)
//const unsigned char soundDataSimple[] = {
//	0x52, 0x49, 0x46, 0x46, 0x24, 0x08, 0x00, 0x00, // RIFF header
//	0x57, 0x41, 0x56, 0x45, 0x66, 0x6d, 0x74, 0x20, // WAVE header
//	0x10, 0x00, 0x00, 0x00, 0x01, 0x00, 0x01, 0x00, // Format chunk
//	0x44, 0xac, 0x00, 0x00, 0x10, 0x00, 0x00, 0x00, // 44.1kHz, 16-bit
//	0x64, 0x61, 0x74, 0x61, 0x00, 0x08, 0x00, 0x00, // Data chunk header
//	// Actual audio data (440 Hz sine wave)
//	0x00, 0x00, 0x1d, 0x00, 0x38, 0x00, 0x4f, 0x00,
//	0x5e, 0x00, 0x64, 0x00, 0x68, 0x00, 0x6a, 0x00,
//	0x68, 0x00, 0x64, 0x00, 0x5e, 0x00, 0x4f, 0x00,
//	0x38, 0x00, 0x1d, 0x00, 0x00, 0x00, 0xe3, 0xff,
//	0xc8, 0xff, 0xa0, 0xff, 0x8f, 0xff, 0x7c, 0xff,
//	0x6f, 0xff, 0x68, 0xff, 0x68, 0xff, 0x70, 0xff,
//	0x7c, 0xff, 0x8f, 0xff, 0xa0, 0xff, 0xc8, 0xff,
//	0xe3, 0xff, 0x00, 0x00, 0x1d, 0x00, 0x38, 0x00,
//};

//void playWavFromMemory(const unsigned char* data, size_t size) {
//	HWAVEOUT hWaveOut;
//	WAVEFORMATEX wfx;
//
//	// Set up the WAVEFORMATEX structure
//	wfx.wFormatTag = WAVE_FORMAT_PCM;
//	wfx.nChannels = 1; // Mono
//	wfx.nSamplesPerSec = 44100; // Sample rate
//	wfx.wBitsPerSample = 16; // Bits per sample
//	wfx.nBlockAlign = (wfx.nChannels * wfx.wBitsPerSample) / 8;
//	wfx.nAvgBytesPerSec = wfx.nSamplesPerSec * wfx.nBlockAlign;
//
//	// Open the wave output device
//	waveOutOpen(&hWaveOut, WAVE_MAPPER, &wfx, 0, 0, CALLBACK_NULL);
//
//	// Prepare the wave header
//	WAVEHDR whdr;
//	whdr.lpData = (LPSTR)data; // Pointer to the data
//	whdr.dwBufferLength = (DWORD)size; // Size of the data
//	whdr.dwFlags = 0;
//
//	// Prepare and write the header
//	waveOutPrepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
//	waveOutWrite(hWaveOut, &whdr, sizeof(WAVEHDR));
//
//	// Wait for the sound to finish playing
//	while (!(whdr.dwFlags & WHDR_DONE)) {
//		Sleep(100);
//	}
//
//	// Clean up
//	waveOutUnprepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
//	waveOutClose(hWaveOut);
//}



const int SAMPLE_RATE = 44100;
const int DURATION = 1; // 1 second
const int FREQUENCY = 440; // Frequency of the sine wave (A4 note)

const int NUM_SAMPLES = SAMPLE_RATE * DURATION;
const int BYTE_RATE = SAMPLE_RATE * 2; // 16 bits = 2 bytes per sample

// Generating stereo sine wave data
unsigned char soundDataSine[NUM_SAMPLES * 4]; // 2 channels (stereo), 2 bytes per sample


void generateSineWave() {
	for (int i = 0; i < NUM_SAMPLES; i++) {
		// Calculate the sample value
		int16_t sample = static_cast<int16_t>(32767 * sin((2.0 * M_PI * FREQUENCY * i) / SAMPLE_RATE));

		// Fill left channel
		soundDataSine[i * 4] = (sample & 0xFF);          // Low byte
		soundDataSine[i * 4 + 1] = (sample >> 8) & 0xFF; // High byte

		// Fill right channel (same value for stereo effect)
		soundDataSine[i * 4 + 2] = (sample & 0xFF);      // Low byte
		soundDataSine[i * 4 + 3] = (sample >> 8) & 0xFF; // High byte
	}
}

void playSineWavFromMemory(const unsigned char* data, size_t size) {
	HWAVEOUT hWaveOut;
	WAVEFORMATEX wfx;

	// Set up the WAVEFORMATEX structure
	wfx.wFormatTag = WAVE_FORMAT_PCM;
	wfx.nChannels = 2; // Stereo
	wfx.nSamplesPerSec = SAMPLE_RATE; // Sample rate
	wfx.wBitsPerSample = 16; // Bits per sample
	wfx.nBlockAlign = (wfx.nChannels * wfx.wBitsPerSample) / 8;
	wfx.nAvgBytesPerSec = wfx.nSamplesPerSec * wfx.nBlockAlign;

	// Open the wave output device
	waveOutOpen(&hWaveOut, WAVE_MAPPER, &wfx, 0, 0, CALLBACK_NULL);

	// Prepare the wave header
	WAVEHDR whdr;
	whdr.lpData = (LPSTR)data; // Pointer to the data
	whdr.dwBufferLength = (DWORD)size; // Size of the data
	whdr.dwFlags = 0;

	// Prepare and write the header
	waveOutPrepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
	waveOutWrite(hWaveOut, &whdr, sizeof(WAVEHDR));

	// Wait for the sound to finish playing
	while (!(whdr.dwFlags & WHDR_DONE)) {
		Sleep(100);
	}

	// Clean up
	waveOutUnprepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
	waveOutClose(hWaveOut);
}


void playAlert() {
	//printf("Playing sound...\r\n");
	//Beep(2000, 1000);

	generateSineWave(); // Fill soundData with sine wave
	playSineWavFromMemory(soundDataSine, sizeof(soundDataSine));

	//playWavFromMemory(soundDataSimple, sizeof(soundDataSimple));
	//int frequencies[] = { 800, 1000, 1200, 1000 }; // Frequencies in Hz
	//int durations[] = { 300, 300, 300, 400 }; // Durations in milliseconds

	//for (int i = 0; i < sizeof(frequencies) / sizeof(frequencies[0]); ++i) {
	//	Beep(frequencies[i], durations[i]);
	//	Sleep(50);
	//}
}

bool  DispatchDictionaryScan(ConfigClass* Config, data_class* Data, stride_class* Stride) {

	if (InitalSync(Config) == false)
		return false;



	uint64_t nProblemPower =
		(uint64_t)host_AdaptiveBaseDigitCarryTrigger[0]
		* host_AdaptiveBaseDigitCarryTrigger[1]
		* host_AdaptiveBaseDigitCarryTrigger[2]
		* host_AdaptiveBaseDigitCarryTrigger[3]
		* host_AdaptiveBaseDigitCarryTrigger[4]
		* host_AdaptiveBaseDigitCarryTrigger[5];


	uint64_t nSolverThreads = Config->cuda_block * Config->cuda_grid;
	uint64_t nIterationPower = nSolverThreads * host_AdaptiveBaseDigitCarryTrigger[5];
	uint64_t nIterationsNeeded = nProblemPower / nIterationPower;

	if (nIterationsNeeded * nIterationPower < nProblemPower)
		nIterationsNeeded++;



	std::cout << "-- Starting Dictionary SCAN -- " << std::endl;

	std::cout << " Going to dispatch " << nProblemPower << " total COMBOs"
		<< " via " << nIterationsNeeded << " iterations "
		" (each able to process " << nIterationPower << " instances)." << std::endl;



	uint64_t nBatchMax = 1;

	int nBatch = 0;


	
	size_t copySize;
	hipError_t cudaResult;

	//uint64_t nMasterIteration = 0;
	*Data->host.nProcessedInstances = 0;
	*Data->host.nProcessedIterations = 0;

	if (hipSuccess != hipMemcpy(Data->dev.nProcessedInstances, Data->host.nProcessedInstances, 8, hipMemcpyHostToDevice)) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}
	const int nMnemoShowLen = MAX_ADAPTIVE_BASE_POSITIONS * 9 + MAX_ADAPTIVE_BASE_POSITIONS;
	char strMnemoShow[nMnemoShowLen] = {0};
	int16_t digitShow[MAX_ADAPTIVE_BASE_POSITIONS];
	uint64_t nUniversalProcessed = 0;

	do
	{
		//Set Master Iteration
		if (hipSuccess != hipMemcpy(Data->dev.nProcessedIterations, Data->host.nProcessedIterations, 8, hipMemcpyHostToDevice)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		//Zero Previous Count
		*Data->host.nProcessedInstances = 0;
		if (hipSuccess != hipMemcpy( Data->dev.nProcessedInstances, Data->host.nProcessedInstances, 8, hipMemcpyHostToDevice)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		printf("Iteration: %llu started.\r\n", *Data->host.nProcessedIterations + 1);
		IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger, host_AdaptiveBaseCurrentBatchInitialDigits
			, nUniversalProcessed, digitShow);
		ShowAdaptiveStr(digitShow, strMnemoShow);
		printf("<FROM> * * * * * *\t %s </FROM> (%llu)\r\n", strMnemoShow, nUniversalProcessed+1);




		if (Stride->startDictionaryAttack(Config->cuda_grid, Config->cuda_block) != 0) {
			std::cerr << "Error START!!" << std::endl;
			return false;
		}
		tools::start_time();


		float delay;
		if (Stride->endDictionaryAttack() != 0) {
			std::cerr << "Error END!!" << std::endl;
			return false;
		}
		tools::stop_time_and_calc_sec(&delay);

		//if (bCfgSaveResultsIntoFile) {
		//	save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config->generate_path);
		//}



		if (hipSuccess != hipMemcpy(Data->host.nProcessedInstances, Data->dev.nProcessedInstances, 8, hipMemcpyDeviceToHost)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		nUniversalProcessed += *Data->host.nProcessedInstances;
		//printf("\t\t\t.\r\n\t\t\t.\r\n\t\t\t.\r\n");
		IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger, host_AdaptiveBaseCurrentBatchInitialDigits
			, nUniversalProcessed-1, digitShow);
		ShowAdaptiveStr(digitShow, strMnemoShow);

		printf("<UPTO> * * * * * * \t %s </UPTO> (%llu)\r\n", strMnemoShow, nUniversalProcessed);

		printf("Checking results of %llu checkups.\r\n", *Data->host.nProcessedInstances);


		//std::cout << std::endl << "PROCESSED: at " << tools::formatPrefix((double)*Data->host.nProcessedInstances / delay) << " COMBO/Sec" << std::endl;


		std::cout << "Iteration " << *Data->host.nProcessedIterations
			<< " completed we have processed  " << *Data->host.nProcessedInstances << " COMBOs  at " << tools::formatPrefix((double)*Data->host.nProcessedInstances / delay) << " COMBO/Sec" << std::endl;

		if (DictionaryCheckFound(Data->host.ret)) {
			tools::checkResult(Data->host.ret);
			playAlert();
		}

		++*Data->host.nProcessedIterations;
	} while (*Data->host.nProcessedIterations < nIterationsNeeded);//trunk

	return true;
}


bool InitalSync(ConfigClass* Config)
{
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] = 0;
	host_EntropyNextPrefix2[PTR_AVOIDER] = 0;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[0]) << 53;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[1]) << 42;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[2]) << 31;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[3]) << 20;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[4]) << 9;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[5]) >> 2;
	host_EntropyNextPrefix2[PTR_AVOIDER] = (uint64_t)(Config->words_indicies_mnemonic[5]) << 62; //two bits from main 6 words


	size_t copySize;
	hipError_t cudaResult;

	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyAbsolutePrefix64), host_EntropyAbsolutePrefix64, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyAbsolutePrefix64 failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyNextPrefix2), host_EntropyNextPrefix2, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyBatchNext24 failed!: " << cudaResult << std::endl;
		return false;
	}


	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS * MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitSet), host_AdaptiveBaseDigitSet, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "dev_AdaptiveBaseCurrentBatchInitialDigits copying " << copySize << " bytes to dev_AdaptiveBaseDigitSet failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(host_AdaptiveBaseDigitCarryTrigger[0]) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitCarryTrigger), host_AdaptiveBaseDigitCarryTrigger, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseDigitCarryTrigger failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), host_AdaptiveBaseCurrentBatchInitialDigits, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
		return false;
	}

	return true;
}
