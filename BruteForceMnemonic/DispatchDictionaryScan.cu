#include "hip/hip_runtime.h"
#include <stdafx.h>

#include <iostream>
#include <thread>

#include "hip/hip_runtime.h"

#include "DispatchDictionaryScan.cuh"
#include "DictionaryScanner.cuh"

#include "consts.h"
#include "AdaptiveBase.h"


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "Helper.h"

#include <windows.h> //some beeping fancey
#include <mmsystem.h>

#define _USE_MATH_DEFINES
#include <cmath>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#include "EntropyTools.cuh"









//// A simple 1-second sine wave sound (440 Hz)
//const unsigned char soundDataSimple[] = {
//	0x52, 0x49, 0x46, 0x46, 0x24, 0x08, 0x00, 0x00, // RIFF header
//	0x57, 0x41, 0x56, 0x45, 0x66, 0x6d, 0x74, 0x20, // WAVE header
//	0x10, 0x00, 0x00, 0x00, 0x01, 0x00, 0x01, 0x00, // Format chunk
//	0x44, 0xac, 0x00, 0x00, 0x10, 0x00, 0x00, 0x00, // 44.1kHz, 16-bit
//	0x64, 0x61, 0x74, 0x61, 0x00, 0x08, 0x00, 0x00, // Data chunk header
//	// Actual audio data (440 Hz sine wave)
//	0x00, 0x00, 0x1d, 0x00, 0x38, 0x00, 0x4f, 0x00,
//	0x5e, 0x00, 0x64, 0x00, 0x68, 0x00, 0x6a, 0x00,
//	0x68, 0x00, 0x64, 0x00, 0x5e, 0x00, 0x4f, 0x00,
//	0x38, 0x00, 0x1d, 0x00, 0x00, 0x00, 0xe3, 0xff,
//	0xc8, 0xff, 0xa0, 0xff, 0x8f, 0xff, 0x7c, 0xff,
//	0x6f, 0xff, 0x68, 0xff, 0x68, 0xff, 0x70, 0xff,
//	0x7c, 0xff, 0x8f, 0xff, 0xa0, 0xff, 0xc8, 0xff,
//	0xe3, 0xff, 0x00, 0x00, 0x1d, 0x00, 0x38, 0x00,
//};

//void playWavFromMemory(const unsigned char* data, size_t size) {
//	HWAVEOUT hWaveOut;
//	WAVEFORMATEX wfx;
//
//	// Set up the WAVEFORMATEX structure
//	wfx.wFormatTag = WAVE_FORMAT_PCM;
//	wfx.nChannels = 1; // Mono
//	wfx.nSamplesPerSec = 44100; // Sample rate
//	wfx.wBitsPerSample = 16; // Bits per sample
//	wfx.nBlockAlign = (wfx.nChannels * wfx.wBitsPerSample) / 8;
//	wfx.nAvgBytesPerSec = wfx.nSamplesPerSec * wfx.nBlockAlign;
//
//	// Open the wave output device
//	waveOutOpen(&hWaveOut, WAVE_MAPPER, &wfx, 0, 0, CALLBACK_NULL);
//
//	// Prepare the wave header
//	WAVEHDR whdr;
//	whdr.lpData = (LPSTR)data; // Pointer to the data
//	whdr.dwBufferLength = (DWORD)size; // Size of the data
//	whdr.dwFlags = 0;
//
//	// Prepare and write the header
//	waveOutPrepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
//	waveOutWrite(hWaveOut, &whdr, sizeof(WAVEHDR));
//
//	// Wait for the sound to finish playing
//	while (!(whdr.dwFlags & WHDR_DONE)) {
//		Sleep(100);
//	}
//
//	// Clean up
//	waveOutUnprepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
//	waveOutClose(hWaveOut);
//}



const int SAMPLE_RATE = 44100;
const int DURATION = 1; // 1 second
const int FREQUENCY = 440; // Frequency of the sine wave (A4 note)

const int NUM_SAMPLES = SAMPLE_RATE * DURATION;
const int BYTE_RATE = SAMPLE_RATE * 2; // 16 bits = 2 bytes per sample

// Generating stereo sine wave data
unsigned char soundDataSine[NUM_SAMPLES * 4]; // 2 channels (stereo), 2 bytes per sample


void generateSineWave() {
	for (int i = 0; i < NUM_SAMPLES; i++) {
		// Calculate the sample value
		int16_t sample = static_cast<int16_t>(32767 * sin((2.0 * M_PI * FREQUENCY * i) / SAMPLE_RATE));

		// Fill left channel
		soundDataSine[i * 4] = (sample & 0xFF);          // Low byte
		soundDataSine[i * 4 + 1] = (sample >> 8) & 0xFF; // High byte

		// Fill right channel (same value for stereo effect)
		soundDataSine[i * 4 + 2] = (sample & 0xFF);      // Low byte
		soundDataSine[i * 4 + 3] = (sample >> 8) & 0xFF; // High byte
	}
}

void playSineWavFromMemory(const unsigned char* data, size_t size) {
	HWAVEOUT hWaveOut;
	WAVEFORMATEX wfx;

	// Set up the WAVEFORMATEX structure
	wfx.wFormatTag = WAVE_FORMAT_PCM;
	wfx.nChannels = 2; // Stereo
	wfx.nSamplesPerSec = SAMPLE_RATE; // Sample rate
	wfx.wBitsPerSample = 16; // Bits per sample
	wfx.nBlockAlign = (wfx.nChannels * wfx.wBitsPerSample) / 8;
	wfx.nAvgBytesPerSec = wfx.nSamplesPerSec * wfx.nBlockAlign;

	// Open the wave output device
	waveOutOpen(&hWaveOut, WAVE_MAPPER, &wfx, 0, 0, CALLBACK_NULL);

	// Prepare the wave header
	WAVEHDR whdr;
	whdr.lpData = (LPSTR)data; // Pointer to the data
	whdr.dwBufferLength = (DWORD)size; // Size of the data
	whdr.dwFlags = 0;

	// Prepare and write the header
	waveOutPrepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
	waveOutWrite(hWaveOut, &whdr, sizeof(WAVEHDR));

	// Wait for the sound to finish playing
	while (!(whdr.dwFlags & WHDR_DONE)) {
		Sleep(100);
	}

	// Clean up
	waveOutUnprepareHeader(hWaveOut, &whdr, sizeof(WAVEHDR));
	waveOutClose(hWaveOut);
}


void playAlert() {
	//printf("Playing sound...\r\n");
	//Beep(2000, 1000);

	generateSineWave(); // Fill soundData with sine wave
	playSineWavFromMemory(soundDataSine, sizeof(soundDataSine));

	//playWavFromMemory(soundDataSimple, sizeof(soundDataSimple));
	//int frequencies[] = { 800, 1000, 1200, 1000 }; // Frequencies in Hz
	//int durations[] = { 300, 300, 300, 400 }; // Durations in milliseconds

	//for (int i = 0; i < sizeof(frequencies) / sizeof(frequencies[0]); ++i) {
	//	Beep(frequencies[i], durations[i]);
	//	Sleep(50);
	//}
}

bool  DispatchDictionaryScan(ConfigClass* Config, data_class* Data, stride_class* Stride) {

	if (InitalSync(Config) == false)
		return false;



	uint64_t nProblemPower =
		(uint64_t)host_AdaptiveBaseDigitCarryTrigger[0]
		* host_AdaptiveBaseDigitCarryTrigger[1]
		* host_AdaptiveBaseDigitCarryTrigger[2]
		* host_AdaptiveBaseDigitCarryTrigger[3]
		* host_AdaptiveBaseDigitCarryTrigger[4]
		* host_AdaptiveBaseDigitCarryTrigger[5];


	uint64_t nSolverThreads = Config->cuda_block * Config->cuda_grid;
	uint64_t nThreadPower = host_AdaptiveBaseDigitCarryTrigger[4] * host_AdaptiveBaseDigitCarryTrigger[5];
	uint64_t nIterationPower = nSolverThreads * nThreadPower;
	uint64_t nIterationsNeeded = nProblemPower / nIterationPower;
	uint64_t nLastIterationRemainder = nProblemPower - nIterationsNeeded * nIterationPower;
	uint64_t nLastIterationMaxBlockIdx =  nLastIterationRemainder / nThreadPower / Config->cuda_block;
	if (nLastIterationRemainder > 0) {
		nIterationsNeeded++;
	}
	host_nManagedIterationsMaxCurrent[0] = nIterationsNeeded;
	host_nManagedIterationsMaxCurrent[1] = 0ui64;

	for (int i = 0; i < MAX_BLOCKS; i++) {
		nManagedIterationsPerBlock[i] = 0;
	}

	printf("Starting Dictionary Scan...\r\n");
	printf("Looking for Account Range %u to %u.\r\n",host_accntMinMax[0],host_accntMinMax[1]);
	printf("Looking for Children Address from %u to %u.\r\n ", host_childrenMinMax[0], host_childrenMinMax[1]);

	std::cout << " Going to dispatch " << nProblemPower << " total COMBOs" <<std::endl
		<< " {via " << nIterationsNeeded
		<<((nLastIterationRemainder > 0) ? "" : "Perfet")<< " iterations}" << " [Last one with:" <<
		nLastIterationRemainder<<" COMBOs]" <<std::endl
		<<" (each able to process " << nIterationPower << " instances)="<< Config->cuda_grid <<"x" << Config->cuda_block<< "x" << host_AdaptiveBaseDigitCarryTrigger[5] << std::endl;






	
	size_t copySize;
	hipError_t cudaResult;

	*Data->host.nProcessedInstances = 0;
	*Data->host.nProcessingIteration = 0;

#if 0

	dev_retEntropy[0] = 0ui64;
	dev_retEntropy[1] = 0ui64;

	dev_retAccntPath[0] = 0;
	dev_retAccntPath[1] = 0;


#else
	host_retEntropy[0] = 0ui64;
	host_retEntropy[1] = 0ui64;

	host_retAccntPath[0] = 0;
	host_retAccntPath[1] = 0;

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dev_retEntropy), host_retEntropy, 16)) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dev_retAccntPath), host_retAccntPath, 2)) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}

#endif
	//host_accntMinMax[0] = 0;
	//host_accntMinMax[1] = 5;
	//printf("Size[0]=%llu , Size_tot=%llu\r\n", sizeof(host_accntMinMax[0]), sizeof(host_accntMinMax));
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dev_accntMinMax), host_accntMinMax, sizeof (host_accntMinMax))) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dev_childrenMinMax), host_childrenMinMax, sizeof(host_childrenMinMax))) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dev_nManagedIterationsMaxCurrent), host_nManagedIterationsMaxCurrent, sizeof(host_nManagedIterationsMaxCurrent))) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}


	if (hipSuccess != hipMemcpy(Data->dev.nProcessedInstances, Data->host.nProcessedInstances, 8, hipMemcpyHostToDevice)) {
		std::cout << "Error-Line--" << __LINE__ << std::endl;
	}
	const int nMnemoShowLen = MAX_ADAPTIVE_BASE_POSITIONS * 9 + MAX_ADAPTIVE_BASE_POSITIONS;
	char strMnemoShow[nMnemoShowLen] = {0};
	int16_t digitShow[MAX_ADAPTIVE_BASE_POSITIONS];
	uint64_t nUniversalProcessed = 0;

		//Set Master Iteration
		if (hipSuccess != hipMemcpy(Data->dev.nProcessingIteration, Data->host.nProcessingIteration, 8, hipMemcpyHostToDevice)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		//Zero Previous Count
		*Data->host.nProcessedInstances = 0;
		if (hipSuccess != hipMemcpy( Data->dev.nProcessedInstances, Data->host.nProcessedInstances, 8, hipMemcpyHostToDevice)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		tools::start_time();
		if (Stride->startDictionaryAttack(Config->cuda_grid, Config->cuda_block) != 0) {
			std::cerr << "Error START!!" << std::endl;
			return false;
		}

		do
		{

		printf("Iteration: %llu started.\r\n", *Data->host.nProcessingIteration + 1);
		int16_t nDummyRet;

		IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger, host_AdaptiveBaseCurrentBatchInitialDigits
			, nUniversalProcessed, digitShow, &nDummyRet);

		ShowAdaptiveStr(host_AdaptiveBaseDigitSet,digitShow, arrBipWords,arrBipWordsLengths,strMnemoShow);
		printf("<FROM> * * * * * *\t %s </FROM> (%llu)\r\n", strMnemoShow, nUniversalProcessed+1);


#if 0
		{ //view iteration progress
			uint64_t nVisitCount = 0;
			uint64_t nBlocksToVisit = (nIterationsNeeded == *Data->host.nProcessingIteration + 1) ? nLastIterationMaxBlockIdx + 1 : MAX_BLOCKS;
			printf("\rWaiting for:%llu blocks to complete.", nBlocksToVisit);
			while (nVisitCount < nBlocksToVisit) {
				nVisitCount = 0;
				for (int gb = 0; gb < nBlocksToVisit; gb++) {
					printf("Checking completion....");

					if (nManagedIterationsPerBlock[gb] < *Data->host.nProcessingIteration) {
						printf("Not satisified.\r\n");
						continue;
					}

					nVisitCount++;

					double fPercent = 100.0 * nVisitCount / nBlocksToVisit;
					printf("\r Iteration Progress: %f", fPercent);
				}

				if (nVisitCount < nBlocksToVisit) {
					std:Sleep(1000);
				}
			}//while
		}

#endif
		float delay;
#if 1

		if (Stride->endDictionaryAttack() != 0) {
			std::cerr << "Error END!!" << std::endl;
			return false;
		}
#endif
		tools::stop_time_and_calc_sec(&delay);

#if 0
		if (hipSuccess != hipMemcpy(Data->host.nProcessedInstances, Data->dev.nProcessedInstances, 8, hipMemcpyDeviceToHost)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}
		
		nUniversalProcessed += *Data->host.nProcessedInstances;
		//printf("\t\t\t.\r\n\t\t\t.\r\n\t\t\t.\r\n");
		//int16_t nDummyRet;
		IncrementAdaptiveDigits(host_AdaptiveBaseDigitCarryTrigger, host_AdaptiveBaseCurrentBatchInitialDigits
			, nUniversalProcessed-1, digitShow, &nDummyRet);

		ShowAdaptiveStr(host_AdaptiveBaseDigitSet, digitShow, arrBipWords, arrBipWordsLengths, strMnemoShow);

		printf("<UPTO> * * * * * * \t %s </UPTO> (%llu)\r\n", strMnemoShow, nUniversalProcessed);

		printf("Checking results of %llu checkups.\r\n", *Data->host.nProcessedInstances);

		uint64_t nSkippedLast = (nIterationPower - *Data->host.nProcessedInstances);

#endif



		* Data->host.nProcessedInstances = nIterationPower;
		std::cout << "Iteration " << *Data->host.nProcessingIteration + 1
			<< " completed we have processed  " << *Data->host.nProcessedInstances << " COMBOs  at " << tools::formatPrefix((double)*Data->host.nProcessedInstances / delay) << " COMBO/Sec" 
			//<<" (Unused:"<<nSkippedLast<<") "
			<< std::endl;
			
#if 1
		if (hipSuccess != hipMemcpyFromSymbol(host_retEntropy, HIP_SYMBOL(dev_retEntropy), 16)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}

		if (hipSuccess != hipMemcpyFromSymbol(host_retAccntPath, HIP_SYMBOL(dev_retAccntPath), 2)) {
			std::cout << "Error-Line--" << __LINE__ << std::endl;
		}
#endif
		if (host_retEntropy[0] != 0 || host_retEntropy[1] != 0) {
			uint8_t disp[121];
			GetAllWords(host_retEntropy, disp);
			printf("|----------------------------------------------------------------------------------------\r\n");
			// Get the handle to the console
			HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);

			// Set the text color to green (Green text on black background)
			SetConsoleTextAttribute(hConsole, FOREGROUND_GREEN | FOREGROUND_INTENSITY);
			printf("|    %s \t \r\n", disp);
			// Reset the text color to default (usually white on black)
			SetConsoleTextAttribute(hConsole, FOREGROUND_RED | FOREGROUND_GREEN | FOREGROUND_BLUE);
			printf("|\t---------------------------------------------------------- \t\t\r\n");
			printf("|\t\t Account= %u \t Child= %u  \t\t\t\t\t\r\n", host_retAccntPath[0], host_retAccntPath[1]);
			printf("|\t---------------------------------------------------------- \t\t\r\n");
			printf("|\t\t Entropy : 0x%llX%llX\r\n", host_retEntropy[0], host_retEntropy[1]);
			printf("|----------------------------------------------------------------------------------------\r\n");
			playAlert();
			break;

		}
		else {
			printf("Total Batch Completed.\r\n");
		}

		++*Data->host.nProcessingIteration;
		break;
	} while (*Data->host.nProcessingIteration < nIterationsNeeded);//trunk

	return true;
}


bool InitalSync(ConfigClass* Config)
{
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] = 0;
	host_EntropyNextPrefix2[PTR_AVOIDER] = 0;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[0]) << 53;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[1]) << 42;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[2]) << 31;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[3]) << 20;
	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[4]) << 9;

	host_EntropyAbsolutePrefix64[PTR_AVOIDER] |= (uint64_t)(Config->words_indicies_mnemonic[5]) >> 2;
	host_EntropyNextPrefix2[PTR_AVOIDER] = (uint64_t)(Config->words_indicies_mnemonic[5]) << 62; //two bits from main 6 words


	size_t copySize;
	hipError_t cudaResult;


	copySize = sizeof(uint8_t) * 20;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_uniqueTargetAddressBytes), host_uniqueTargetAddressBytes, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_uniqueTargetAddressBytes failed!: " << cudaResult << std::endl;
		return false;
	}


	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyAbsolutePrefix64), host_EntropyAbsolutePrefix64, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyAbsolutePrefix64 failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(uint64_t);
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_EntropyNextPrefix2), host_EntropyNextPrefix2, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_EntropyBatchNext24 failed!: " << cudaResult << std::endl;
		return false;
	}


	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS * MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitSet), host_AdaptiveBaseDigitSet, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "dev_AdaptiveBaseCurrentBatchInitialDigits copying " << copySize << " bytes to dev_AdaptiveBaseDigitSet failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(host_AdaptiveBaseDigitCarryTrigger[0]) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseDigitCarryTrigger), host_AdaptiveBaseDigitCarryTrigger, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseDigitCarryTrigger failed!: " << cudaResult << std::endl;
		return false;
	}

	copySize = sizeof(int16_t) * MAX_ADAPTIVE_BASE_POSITIONS;
	cudaResult = hipMemcpyToSymbol(HIP_SYMBOL(dev_AdaptiveBaseCurrentBatchInitialDigits), host_AdaptiveBaseCurrentBatchInitialDigits, copySize, 0, hipMemcpyHostToDevice);
	if (cudaResult != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol copying " << copySize << " bytes to dev_AdaptiveBaseCurrentBatchInitialDigits failed!: " << cudaResult << std::endl;
		return false;
	}

	return true;
}
