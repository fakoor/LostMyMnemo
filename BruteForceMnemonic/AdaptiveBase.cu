#include "hip/hip_runtime.h"
#include <stdafx.h>
#include <stdio.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include <GPU.h>
#include "AdaptiveBase.h"



/*
* We onsider Mnemonics a Base - 2048 twelve-digit unsigned integer,
* if we have some information about any of digits, so that we can
* ommit 1 to 2047 digits (words) from a specific position, then that
* digit is adaptively based to Base-2047 down to Base-2 aka binary.
* The only mathematic operation we need for the purpose is increment,
* When we reach the carry for a VariableBase digit, uppon an increment,
* we will rewind that digit and increment the more significant digit
* DUE to less memory usage in constant-memory, we limit such mechanism
* to the 6 least significant mnemonic words in a 262 guessed space
* Hence the dictionary for positions 6 to 12 outght not to include
* more than 262 words. The other structure provides reverse lookup 
* for a value of a digit
*/



__device__ int dev_checkResult(retStruct* ret) {

	if (ret->f[0].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[0].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[1].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[1].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[2].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[2].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[0].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[0].count_found_bytes = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[1].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[1].count_found_bytes = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[2].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[2].count_found_bytes = MAX_FOUND_ADDRESSES;

	}

	if (ret->f[0].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[0].count_found; i++)
		{
			return 1;
		}
	}
	if (ret->f[1].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[1].count_found; i++)
		{
			return 1;
		}
	}
	if (ret->f[2].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[2].count_found; i++)
		{
			return 1;
		}
	}

	return 0;
}



__constant__ uint64_t dev_EntropyAbsolutePrefix64[1];
uint64_t host_EntropyAbsolutePrefix64[1];

__constant__ int16_t dev_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS];
 int16_t host_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS];

__constant__ int16_t dev_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION];
 int16_t host_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION];


__constant__ int16_t dev_AdaptiveBaseCurrentBatchInitialDigits[MAX_ADAPTIVE_BASE_POSITIONS];
 int16_t host_AdaptiveBaseCurrentBatchInitialDigits[MAX_ADAPTIVE_BASE_POSITIONS];


__constant__ uint64_t dev_EntropyNextPrefix2[1]; //Per-Batch Const
 uint64_t host_EntropyNextPrefix2[1]; //Per-Batch Const

 

__host__ /* __and__ */ __device__ bool IncrementAdaptiveDigits( int16_t * local_AdaptiveBaseDigitCarryTrigger, int16_t* inDigits, uint64_t howMuch, int16_t* outDigits) {
	uint64_t nYetToAdd = howMuch;
	uint64_t nCarryValue = 0;
	int16_t tmpResult [MAX_ADAPTIVE_BASE_POSITIONS];

	for (int i = MAX_ADAPTIVE_BASE_POSITIONS - 1; i >= 0; i--) {
		if (nYetToAdd == 0 && nCarryValue == 0) {
			tmpResult[i] = inDigits[i];
			continue;
		}

		int16_t beforeIncDigit = inDigits[i];
		int nCarryAt = local_AdaptiveBaseDigitCarryTrigger[i];

		int nThisIdeal = nYetToAdd + beforeIncDigit + nCarryValue;
		int nThisNewDigit = nThisIdeal % nCarryAt;


		tmpResult[i] = nThisNewDigit;
		nCarryValue = nThisIdeal / nCarryAt;
		nYetToAdd = 0; //all active in carry if any
	}
	if (nYetToAdd != 0 || nCarryValue != 0) {
		//ASSERT: We have carried out of our space, NOP anyway
		return false;
	}
	else {
		for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++) {
			outDigits[i] = tmpResult[i];
		}
	}
	return true;
}


__host__ /* __and__ */ __device__ void GetBipForAdaptiveDigit(
	  int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t* local_AdaptiveBaseDigitCarryTrigger
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
	, int16_t* inDigits, uint8_t pos, uint64_t* outBip) {
	int16_t curAdapriveDigit = inDigits[pos];
	*outBip = local_AdaptiveBaseDigitSet[pos][curAdapriveDigit];
}
__host__ /* __and__ */ __device__ void GetPaddedBipForAdaptiveDigit(
	int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t* local_AdaptiveBaseDigitCarryTrigger
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION], int16_t* inDigits, uint8_t pos, uint64_t* outPadBip) {
	uint64_t curBipForDigit;

	GetBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
		, local_AdaptiveBaseDigitCarryTrigger
		, local_AdaptiveBaseDigitSet, inDigits, pos, &curBipForDigit);

	uint8_t shiftCount;
	if (pos < MAX_ADAPTIVE_BASE_POSITIONS - 1) {
		shiftCount = (128 - 66 - pos * 11);
		*outPadBip = curBipForDigit << shiftCount;
	}
	else {
		shiftCount = 4;
		*outPadBip = curBipForDigit >> shiftCount;
	}
}
__host__ /* __and__ */ __device__ void AdaptiveDigitsToEntropy(
	int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t local_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS]
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION],
	uint64_t * local_EntropyAbsolutePrefix64,
	uint64_t * local_EntropyBatchNext24, 
	 int16_t* inDigits, uint64_t* outEntropy, uint8_t* checkSum) {

	outEntropy[0] = local_EntropyAbsolutePrefix64[0];
	outEntropy[1] = local_EntropyBatchNext24[0];

	uint64_t digitPaddedBip;
	for (int pos = 0; pos < MAX_ADAPTIVE_BASE_POSITIONS; pos++) {
		GetPaddedBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
			, local_AdaptiveBaseDigitCarryTrigger
			,local_AdaptiveBaseDigitSet, inDigits, pos, &digitPaddedBip);
		outEntropy[1] |= digitPaddedBip;
	}

	uint64_t lastWord;
	GetBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
		, local_AdaptiveBaseDigitCarryTrigger
		, local_AdaptiveBaseDigitSet, inDigits, MAX_ADAPTIVE_BASE_POSITIONS, &lastWord);
	*checkSum = lastWord & 0x000F;
}


__global__ void gl_DictionaryAttack(
	 uint64_t*  nBatchPlannedProc,
	 uint64_t*  nBatchMoreProc,
	const tableStruct* __restrict__ tables_legacy,
	const tableStruct* __restrict__ tables_segwit,
	const tableStruct* __restrict__ tables_native_segwit,
	retStruct* __restrict__ ret
)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int blockId = blockIdx.x;
	unsigned int threadId = threadIdx.x;

	uint16_t totalPlannedCount = blockDim.x * gridDim.x;

	__shared__ uint64_t ourBlockProcNormal;
	__shared__ uint64_t ourBlockProcExtra;
	__shared__ uint64_t ourBlockBadChkSum;
	__shared__ uint64_t ourBlockGoodChkSum;
	__shared__ int16_t myDigSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION];
	__shared__ uint64_t nMaxCloudAdd;
	__shared__ unsigned int nMoreIterated;
	int16_t local_static_word_index[12];

	// Initialize the shared variable
	if (threadIdx.x == 0) {
		ourBlockProcNormal = 0; // Only the first thread initializes it
		ourBlockProcExtra = 0;
		ourBlockBadChkSum = 0;
		ourBlockGoodChkSum = 0;
		for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++) {
			for (int j = 0; j < MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION; j++) {
				myDigSet[i][j] = dev_AdaptiveBaseDigitSet[i][j];
			}
		}
		nMaxCloudAdd = 0;
		nMoreIterated = 0;
	}
	__syncthreads(); // Synchronize to ensure the initialization is complete

	for (int i = 0; i < 6; i++) {
		local_static_word_index[i] = dev_static_words_indices[i];
	}

	unsigned int effective_idx = idx;



	//TODO: Each thread picks is load from Incremental Base!

	uint8_t reqChecksum=0;
	uint8_t achievedChecksum=1;
	bool bChkSumFailed=true;

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS] = {0,0,0,0,0,0};
	uint64_t curEntropy[2];
	curEntropy[0] = dev_EntropyAbsolutePrefix64[PTR_AVOIDER];
	curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];


	int nTried = 0;
	bool bCouldAdd = false;
	do {
		bCouldAdd = IncrementAdaptiveDigits(
			dev_AdaptiveBaseDigitCarryTrigger
			, dev_AdaptiveBaseCurrentBatchInitialDigits
			, effective_idx, curDigits);
		if (bCouldAdd == false ) {
			if (effective_idx == nMaxCloudAdd + 1) {
				printf("Can not add at %x", effective_idx);
			}

			break;
		}
		else {
			atomicMax(&nMaxCloudAdd, effective_idx);
		}

		AdaptiveUpdateMnemonicLow64(&curEntropy[1], myDigSet, curDigits);
		local_static_word_index[6] = myDigSet[0][curDigits[0]];
		local_static_word_index[7] = myDigSet[1][curDigits[1]];
		local_static_word_index[8] = myDigSet[2][curDigits[2]];
		local_static_word_index[9] = myDigSet[3][curDigits[3]];
		local_static_word_index[10] = myDigSet[4][curDigits[4]];
		local_static_word_index[11] = myDigSet[5][curDigits[5]];


			
		//Work with Current Entropy
		uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
		uint8_t* mnemonic = mnemonic_phrase;

		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);

		//printf("Begin - block %d - thread  %d - EffectiveId:%d - curDigits:%d-%d-%d-%d-%d-%d (%d) @ %s\r\n", blockId, threadId, effective_idx
		//	, curDigits[0], curDigits[1], curDigits[2], curDigits[3], curDigits[4], curDigits[5], local_static_word_index [11], mnemonic);

			

		int16_t chkPosIdx = MAX_ADAPTIVE_BASE_POSITIONS - 1;
		int16_t chkWordIdx = curDigits[chkPosIdx];
		uint16_t thisVal = (myDigSet[chkPosIdx][chkWordIdx]);
		uint8_t tmp = (uint8_t)(thisVal & 0x0F);
		reqChecksum = tmp;

		uint8_t entropy_hash[32];
		uint8_t bytes[16];
		uint64_t* entropy = curEntropy;

		bytes[15] = entropy[1] & 0xFF;
		bytes[14] = (entropy[1] >> 8) & 0xFF;
		bytes[13] = (entropy[1] >> 16) & 0xFF;
		bytes[12] = (entropy[1] >> 24) & 0xFF;
		bytes[11] = (entropy[1] >> 32) & 0xFF;
		bytes[10] = (entropy[1] >> 40) & 0xFF;
		bytes[9] = (entropy[1] >> 48) & 0xFF;
		bytes[8] = (entropy[1] >> 56) & 0xFF;

		bytes[7] = entropy[0] & 0xFF;
		bytes[6] = (entropy[0] >> 8) & 0xFF;
		bytes[5] = (entropy[0] >> 16) & 0xFF;
		bytes[4] = (entropy[0] >> 24) & 0xFF;
		bytes[3] = (entropy[0] >> 32) & 0xFF;
		bytes[2] = (entropy[0] >> 40) & 0xFF;
		bytes[1] = (entropy[0] >> 48) & 0xFF;
		bytes[0] = (entropy[0] >> 56) & 0xFF;
		
		sha256((uint32_t*)bytes, 16, (uint32_t*)entropy_hash);
		achievedChecksum = (entropy_hash[0] >> 4) & 0x0F;

		bChkSumFailed = (achievedChecksum != reqChecksum);

		nTried++;

		if (bChkSumFailed) {
			atomicAdd(&ourBlockProcExtra,1);
		}
		else {
			break;
		}
	} while (nTried < MAX_TRY_PER_THREAD); //do

	__syncthreads(); // Synchronize to and check if have a valid checksum to continue with
	if (bCouldAdd/*bChkSumFailed == false*/) { //scrutinize
		atomicAdd(&ourBlockGoodChkSum, 1);

		uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
		uint8_t* mnemonic = mnemonic_phrase;
		uint32_t ipad[256 / 4];
		uint32_t opad[256 / 4];
		uint32_t seed[64 / 4];


		//Work with Current Entropy
		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);

		//if (idx == 0) {
//			printf("nemo-%u  (retry.remain=%d/%d) = :%s \r\n\r\n", effective_idx,nTried,MAX_TRY_PER_THREAD, mnemonic);
		//}
		//entropy_to_mnemonic(entropy, mnemonic);
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)ipad + x) = 0x3636363636363636ULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)opad + x) = 0x5C5C5C5C5C5C5C5CULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x + 128 / 4] = *(uint32_t*)((uint32_t*)&salt_swap + x);
		}
		sha512_swap((uint64_t*)ipad, 140, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			seed[x] = ipad[128 / 4 + x];
		}
		for (int x = 1; x < 2048; x++) {
			sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				seed[x] = seed[x] ^ ipad[128 / 4 + x];
			}
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x] = 0x36363636 ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			opad[x] = 0x5C5C5C5C ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			ipad[x + 128 / 4] = seed[x];
		}
		//ipad[192 / 4] = 0;
		//opad[192 / 4] = 0;
		sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 128 / 8; x++) {
			*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x) = SWAP512(*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x));
		}

		//printf("END block %d - thread  %d - EffectiveId:%d - curDigits:%d-%d-%d-%d-%d-%d %s\r\n", blockId, threadId, effective_idx
		//	, curDigits[0], curDigits[1], curDigits[2], curDigits[3], curDigits[4], curDigits[5] , mnemonic);

		atomicAdd(&ourBlockProcNormal, 1);

		key_to_hash160((extended_private_key_t*)&ipad[128 / 4], tables_legacy, tables_segwit, tables_native_segwit, (uint32_t*)mnemonic, ret);
		//__syncthreads();
	}
	__syncthreads(); // Synchronize to ensure all data is loaded
	if (threadIdx.x == 0) {
		atomicAdd(nBatchPlannedProc , ourBlockProcNormal);
		//atomicAdd(nBatchMoreProc, ourBlockProcExtra);
		*nBatchMoreProc = 0;
	}

}//DICTIONARY ATTACK

__host__ /*and */ __device__
void AdaptiveUpdateMnemonicLow64(uint64_t* low64
	, int16_t digitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
	, int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS]
)

{
	uint64_t tmpHigh = *low64;
	uint64_t tmpAns = tmpHigh;

	tmpAns = tmpHigh >> 62;
	tmpAns = tmpAns << 2;

	for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS - 1; i++) {
		tmpAns = tmpAns << 11;
		tmpAns & 0xFFFFFFFFFFFFF7F;
		tmpAns |= (uint64_t)(digitSet[i][curDigits[i]]);
	}
	tmpAns = tmpAns << 7;
	tmpAns |= ((uint64_t)(digitSet[MAX_ADAPTIVE_BASE_POSITIONS - 1][curDigits[MAX_ADAPTIVE_BASE_POSITIONS - 1]]) >> 4);

	*low64 = tmpAns;
}


