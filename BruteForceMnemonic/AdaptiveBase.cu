#include <stdafx.h>
#include <stdio.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include <GPU.h>
#include "AdaptiveBase.h"
#include "EntropyTools.cuh"



/*
* We onsider Mnemonics a Base - 2048 twelve-digit unsigned integer,
* if we have some information about any of digits, so that we can
* ommit 1 to 2047 digits (words) from a specific position, then that
* digit is adaptively based to Base-2047 down to Base-2 aka binary.
* The only mathematic operation we need for the purpose is increment,
* When we reach the carry for a VariableBase digit, uppon an increment,
* we will rewind that digit and increment the more significant digit
* DUE to less memory usage in constant-memory, we limit such mechanism
* to the 6 least significant mnemonic words in a 262 guessed space
* Hence the dictionary for positions 6 to 12 outght not to include
* more than 262 words. The other structure provides reverse lookup 
* for a value of a digit
*/



 
 
 __host__ /* __and__ */ __device__ void GetBipForAdaptiveDigit(
	  int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t* local_AdaptiveBaseDigitCarryTrigger
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
	, int16_t* inDigits, uint8_t pos, uint64_t* outBip) {
	int16_t curAdapriveDigit = inDigits[pos];
	*outBip = local_AdaptiveBaseDigitSet[pos][curAdapriveDigit];
}
__host__ /* __and__ */ __device__ void GetPaddedBipForAdaptiveDigit(
	int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t* local_AdaptiveBaseDigitCarryTrigger
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION], int16_t* inDigits, uint8_t pos, uint64_t* outPadBip) {
	uint64_t curBipForDigit;

	GetBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
		, local_AdaptiveBaseDigitCarryTrigger
		, local_AdaptiveBaseDigitSet, inDigits, pos, &curBipForDigit);

	uint8_t shiftCount;
	if (pos < MAX_ADAPTIVE_BASE_POSITIONS - 1) {
		shiftCount = (128 - 66 - pos * 11);
		*outPadBip = curBipForDigit << shiftCount;
	}
	else {
		shiftCount = 4;
		*outPadBip = curBipForDigit >> shiftCount;
	}
}
__host__ /* __and__ */ __device__ void AdaptiveDigitsToEntropy(
	int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t local_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS]
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION],
	uint64_t * local_EntropyAbsolutePrefix64,
	uint64_t * local_EntropyBatchNext24, 
	 int16_t* inDigits, uint64_t* outEntropy, uint8_t* checkSum) {

	outEntropy[0] = local_EntropyAbsolutePrefix64[0];
	outEntropy[1] = local_EntropyBatchNext24[0];

	uint64_t digitPaddedBip;
	for (int pos = 0; pos < MAX_ADAPTIVE_BASE_POSITIONS; pos++) {
		GetPaddedBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
			, local_AdaptiveBaseDigitCarryTrigger
			,local_AdaptiveBaseDigitSet, inDigits, pos, &digitPaddedBip);
		outEntropy[1] |= digitPaddedBip;
	}

	uint64_t lastWord;
	GetBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
		, local_AdaptiveBaseDigitCarryTrigger
		, local_AdaptiveBaseDigitSet, inDigits, MAX_ADAPTIVE_BASE_POSITIONS, &lastWord);
	*checkSum = lastWord & 0x000F;
}





