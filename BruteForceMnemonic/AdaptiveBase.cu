#include "hip/hip_runtime.h"
#include <stdafx.h>

//#include ""
//#include <hip/device_functions.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <GPU.h>
#include "AdaptiveBase.h"



//#include <hip/hip_cooperative_groups.h>
//#include <sm_60_atomic_functions.h>
/*
* We onsider Mnemonics a Base - 2048 twelve-digit unsigned integer,
* if we have some information about any of digits, so that we can
* ommit 1 to 2047 digits (words) from a specific position, then that
* digit is adaptively based to Base-2047 down to Base-2 aka binary.
* The only mathematic operation we need for the purpose is increment,
* When we reach the carry for a VariableBase digit, uppon an increment,
* we will rewind that digit and increment the more significant digit
* DUE to less memory usage in constant-memory, we limit such mechanism
* to the 6 least significant mnemonic words in a 262 guessed space
* Hence the dictionary for positions 6 to 12 outght not to include
* more than 262 words. The other structure provides reverse lookup for a value of a digit
*/



__device__ int dev_checkResult(retStruct* ret) {

	if (ret->f[0].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[0].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[1].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[1].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[2].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[2].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[0].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[0].count_found_bytes = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[1].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[1].count_found_bytes = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[2].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[2].count_found_bytes = MAX_FOUND_ADDRESSES;

	}

	if (ret->f[0].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[0].count_found; i++)
		{
			//foundInfoStruct* info = &ret->f[0].found_info[i];
			//std::string mnemonic_str = (const char*)info->mnemonic;
			//std::string addr;
			//std::string path = getPath(info->path, info->child);
			//tools::encodeAddressBase58((const uint8_t*)info->hash160, addr);
			//tools::addFoundMnemonicInFile(path, mnemonic_str, addr);
			//std::cout << "!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n";
			//std::cout << "!!!FOUND ADDRESS (" << path << "): " << mnemonic_str << ", " << addr << std::endl;
			//std::cout << "!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n";
			return 1;
		}
	}
	if (ret->f[1].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[1].count_found; i++)
		{
			//foundInfoStruct* info = &ret->f[1].found_info[i];
			//std::string mnemonic_str = (const char*)info->mnemonic;
			//std::string addr;
			//std::string path = getPath(info->path, info->child);
			//tools::encodeAddressBIP49((const uint8_t*)info->hash160, addr);
			//tools::addFoundMnemonicInFile(path, mnemonic_str, addr);
			//std::cout << "!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n";
			//std::cout << "!!!FOUND ADDRESS (" << path << "): " << mnemonic_str << ", " << addr << std::endl;
			//std::cout << "!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n
			return 1;
		}
	}
	if (ret->f[2].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[2].count_found; i++)
		{
			//foundInfoStruct* info = &ret->f[2].found_info[i];
			//std::string mnemonic_str = (const char*)info->mnemonic;
			//std::string addr;
			//std::string path = getPath(info->path, info->child);
			//tools::encodeAddressBase32((const uint8_t*)info->hash160, addr);
			//tools::addFoundMnemonicInFile(path, mnemonic_str, addr);
			//std::cout << "!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n";
			//std::cout << "!!!FOUND ADDRESS (" << path << "): " << mnemonic_str << ", " << addr << std::endl;
			//std::cout << "!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n!!!FOUND!!!\n";
			return 1;
		}
	}

	//if (ret->f[0].count_found_bytes != 0)
	//{
	//	for (uint32_t i = 0; i < ret->f[0].count_found_bytes; i++)
	//	{
	//		foundBytesInfoStruct* info = &ret->f[0].found_bytes_info[i];
	//		int num_bytes = 0;
	//		for (int i = 0; i < 20; i++)
	//		{
	//			if (*(uint8_t*)((uint8_t*)info->hash160 + i) != *(uint8_t*)((uint8_t*)info->hash160_from_table + i)) break;
	//			num_bytes++;
	//		}

	//		std::string mnemonic_str = (const char*)info->mnemonic;
	//		std::string hash160 = tools::bytesToHexString((const uint8_t*)info->hash160, 20);
	//		std::string hash160_in_table = tools::bytesToHexString((const uint8_t*)info->hash160_from_table, 20);
	//		std::string addr;
	//		std::string addr_in_table;
	//		std::string path = getPath(info->path, info->child);
	//		tools::encodeAddressBase58((const uint8_t*)info->hash160, addr);
	//		tools::encodeAddressBase58((const uint8_t*)info->hash160_from_table, addr_in_table);
	//		std::cout << "\n!!!FOUND IN ADDRESS(HASH160) (" << path << ") EQUAL " << num_bytes << " BYTES: " << mnemonic_str << "," << addr << "," << addr_in_table << "," << hash160 << "," << hash160_in_table << " \n";
	//		tools::addInFileTest(num_bytes, path, mnemonic_str, hash160, hash160_in_table, addr, addr_in_table);
	//	}
	//}
	//if (ret->f[1].count_found_bytes != 0)
	//{
	//	for (uint32_t i = 0; i < ret->f[1].count_found_bytes; i++)
	//	{
	//		foundBytesInfoStruct* info = &ret->f[1].found_bytes_info[i];
	//		int num_bytes = 0;
	//		for (int i = 0; i < 20; i++)
	//		{
	//			if (*(uint8_t*)((uint8_t*)info->hash160 + i) != *(uint8_t*)((uint8_t*)info->hash160_from_table + i)) break;
	//			num_bytes++;
	//		}

	//		std::string mnemonic_str = (const char*)info->mnemonic;
	//		std::string hash160 = tools::bytesToHexString((const uint8_t*)info->hash160, 20);
	//		std::string hash160_in_table = tools::bytesToHexString((const uint8_t*)info->hash160_from_table, 20);
	//		std::string addr;
	//		std::string addr_in_table;
	//		std::string path = getPath(info->path, info->child);
	//		tools::encodeAddressBIP49((const uint8_t*)info->hash160, addr);
	//		tools::encodeAddressBIP49((const uint8_t*)info->hash160_from_table, addr_in_table);
	//		std::cout << "\n!!!FOUND IN ADDRESS(HASH160) (" << path << ") EQUAL " << num_bytes << " BYTES: " << mnemonic_str << "," << addr << "," << addr_in_table << "," << hash160 << "," << hash160_in_table << " \n";
	//		tools::addInFileTest(num_bytes, path, mnemonic_str, hash160, hash160_in_table, addr, addr_in_table);
	//	}
	//}
	//if (ret->f[2].count_found_bytes != 0)
	//{
	//	for (uint32_t i = 0; i < ret->f[2].count_found_bytes; i++)
	//	{
	//		foundBytesInfoStruct* info = &ret->f[2].found_bytes_info[i];
	//		int num_bytes = 0;
	//		for (int i = 0; i < 20; i++)
	//		{
	//			if (*(uint8_t*)((uint8_t*)info->hash160 + i) != *(uint8_t*)((uint8_t*)info->hash160_from_table + i)) break;
	//			num_bytes++;
	//		}

	//		std::string mnemonic_str = (const char*)info->mnemonic;
	//		std::string hash160 = tools::bytesToHexString((const uint8_t*)info->hash160, 20);
	//		std::string hash160_in_table = tools::bytesToHexString((const uint8_t*)info->hash160_from_table, 20);
	//		std::string addr;
	//		std::string addr_in_table;
	//		std::string path = getPath(info->path, info->child);
	//		tools::encodeAddressBase32((const uint8_t*)info->hash160, addr);
	//		tools::encodeAddressBase32((const uint8_t*)info->hash160_from_table, addr_in_table);
	//		std::cout << "\n!!!FOUND IN ADDRESS(HASH160) (" << path << ") EQUAL " << num_bytes << " BYTES: " << mnemonic_str << "," << addr << "," << addr_in_table << "," << hash160 << "," << hash160_in_table << " \n";
	//		tools::addInFileTest(num_bytes, path, mnemonic_str, hash160, hash160_in_table, addr, addr_in_table);
	//	}
	//}
	return 0;
}



__constant__ uint64_t dev_EntropyAbsolutePrefix64[1];
uint64_t host_EntropyAbsolutePrefix64[1];

__constant__ int16_t dev_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS];
 int16_t host_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS];

__constant__ int16_t dev_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION];
 int16_t host_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION];


__constant__ int16_t dev_AdaptiveBaseCurrentBatchInitialDigits[MAX_ADAPTIVE_BASE_POSITIONS];
 int16_t host_AdaptiveBaseCurrentBatchInitialDigits[MAX_ADAPTIVE_BASE_POSITIONS];


__constant__ uint64_t dev_EntropyBatchNext24[1]; //Per-Batch Const
 uint64_t host_EntropyBatchNext24[1]; //Per-Batch Const

 

__host__ /* __and__ */ __device__ void IncrementAdaptiveDigits( int16_t * local_AdaptiveBaseDigitCarryTrigger, int16_t* inDigits, uint64_t howMuch, int16_t* outDigits) {
	uint64_t nYetToAdd = howMuch;
	uint64_t nCarryValue = 0;

	for (int i = MAX_ADAPTIVE_BASE_POSITIONS - 1; i >= 0; i--) {
		if (nYetToAdd == 0 && nCarryValue == 0) {
			outDigits[i] = inDigits[i];
			continue;
		}

		int16_t beforeIncDigit = inDigits[i];
		int nCarryAt = local_AdaptiveBaseDigitCarryTrigger[i];

		int nThisIdeal = nYetToAdd + beforeIncDigit + nCarryValue;
		int nThisNewDigit = nThisIdeal % nCarryAt;


		outDigits[i] = nThisNewDigit;
		nCarryValue = nThisIdeal / nCarryAt;
		nYetToAdd = 0; //all active in carry if any
	}
	if (nYetToAdd != 0 || nCarryValue != 0) {
		//ASSERT: We have carried out of our space, NOP anyway
	}
}


__host__ /* __and__ */ __device__ void GetBipForAdaptiveDigit(
	  int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t* local_AdaptiveBaseDigitCarryTrigger
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
	, int16_t* inDigits, uint8_t pos, uint64_t* outBip) {
	int16_t curAdapriveDigit = inDigits[pos];
	*outBip = local_AdaptiveBaseDigitSet[pos][curAdapriveDigit];
}
__host__ /* __and__ */ __device__ void GetPaddedBipForAdaptiveDigit(
	int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t* local_AdaptiveBaseDigitCarryTrigger
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION], int16_t* inDigits, uint8_t pos, uint64_t* outPadBip) {
	uint64_t curBipForDigit;

	GetBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
		, local_AdaptiveBaseDigitCarryTrigger
		, local_AdaptiveBaseDigitSet, inDigits, pos, &curBipForDigit);

	uint8_t shiftCount;
	if (pos < MAX_ADAPTIVE_BASE_POSITIONS - 1) {
		shiftCount = (128 - 66 - pos * 11);
		*outPadBip = curBipForDigit << shiftCount;
	}
	else {
		shiftCount = 4;
		*outPadBip = curBipForDigit >> shiftCount;
	}
}
__host__ /* __and__ */ __device__ void AdaptiveDigitsToEntropy(
	int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t local_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS]
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION],
	uint64_t * local_EntropyAbsolutePrefix64,
	uint64_t * local_EntropyBatchNext24, 
	 int16_t* inDigits, uint64_t* outEntropy, uint8_t* checkSum) {

	outEntropy[0] = local_EntropyAbsolutePrefix64[0];
	outEntropy[1] = local_EntropyBatchNext24[0];

	uint64_t digitPaddedBip;
	for (int pos = 0; pos < MAX_ADAPTIVE_BASE_POSITIONS; pos++) {
		GetPaddedBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
			, local_AdaptiveBaseDigitCarryTrigger
			,local_AdaptiveBaseDigitSet, inDigits, pos, &digitPaddedBip);
		outEntropy[1] |= digitPaddedBip;
	}

	uint64_t lastWord;
	GetBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
		, local_AdaptiveBaseDigitCarryTrigger
		, local_AdaptiveBaseDigitSet, inDigits, MAX_ADAPTIVE_BASE_POSITIONS, &lastWord);
	*checkSum = lastWord & 0x000F;
}


__global__ void gl_DictionaryAttack(
	 uint64_t*  nBatchPlannedProc,
	 uint64_t*  nBatchMoreProc,
	const tableStruct* __restrict__ tables_legacy,
	const tableStruct* __restrict__ tables_segwit,
	const tableStruct* __restrict__ tables_native_segwit,
	retStruct* __restrict__ ret
)
{
	__shared__ uint64_t ourBlockProcNormal;
	__shared__ uint64_t ourBlockProcExtra;
	__shared__ uint64_t ourBlockBadChkSum;
	__shared__ uint64_t ourBlockGoodChkSum;
	__shared__ int16_t myDigSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION];


	// Initialize the shared variable
	if (threadIdx.x == 0) {
		ourBlockProcNormal = 0; // Only the first thread initializes it
		ourBlockProcExtra = 0;
		ourBlockBadChkSum = 0;
		ourBlockGoodChkSum = 0;
		
		for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS; i++) {
			for (int j = 0; j < MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION; j++) {
				myDigSet[i][j] = dev_AdaptiveBaseDigitSet[i][j];
			}
		}

	}
	__syncthreads(); // Synchronize to ensure the initialization is complete


	//TODO: Each thread picks is load from Incremental Base!
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	atomicAdd(&ourBlockProcNormal,1);

	uint8_t reqChecksum=0;
	uint8_t achievedChecksum=1;
	bool bChkSumFailed=true;

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS] = {0,0,0,0,0,0};
	uint64_t curEntropy[2];
	curEntropy[0] = dev_EntropyAbsolutePrefix64[PTR_AVOIDER];
	curEntropy[1] = dev_EntropyBatchNext24[PTR_AVOIDER];

	int nAlternateCandidateRemaining = MAX_ALTERNATE_CANDIDATE;
	while (nAlternateCandidateRemaining) {

		IncrementAdaptiveDigits(
			 dev_AdaptiveBaseDigitCarryTrigger
			, dev_AdaptiveBaseCurrentBatchInitialDigits
			, idx, curDigits);

		//AdaptiveDigitsToEntropy(curDigits
		//	, dev_AdaptiveBaseDigitCarryTrigger
		//	, dev_AdaptiveBaseDigitSet
		//	, dev_EntropyAbsolutePrefix64
		//	, dev_EntropyBatchNext24
		//	, curDigits, curEntropy, &reqChecksum);

		AdaptiveUpdateMnemonicLow64(&curEntropy[1], myDigSet, curDigits);

		if (idx == 0) {
			PrintNextMnemo(curEntropy, idx, dev_AdaptiveBaseDigitCarryTrigger, curDigits, myDigSet);
		}

		int16_t chkPosIdx = MAX_ADAPTIVE_BASE_POSITIONS - 1;
		int16_t chkWordIdx = curDigits[chkPosIdx];
		uint16_t thisVal = (myDigSet[chkPosIdx][chkWordIdx]);
		uint8_t tmp = (uint8_t)(thisVal & 0x0F);
		reqChecksum = tmp;

		uint8_t entropy_hash[32];
		uint8_t bytes[16];
		uint64_t* entropy = curEntropy;

		bytes[15] = entropy[1] & 0xFF;
		bytes[14] = (entropy[1] >> 8) & 0xFF;
		bytes[13] = (entropy[1] >> 16) & 0xFF;
		bytes[12] = (entropy[1] >> 24) & 0xFF;
		bytes[11] = (entropy[1] >> 32) & 0xFF;
		bytes[10] = (entropy[1] >> 40) & 0xFF;
		bytes[9] = (entropy[1] >> 48) & 0xFF;
		bytes[8] = (entropy[1] >> 56) & 0xFF;

		bytes[7] = entropy[0] & 0xFF;
		bytes[6] = (entropy[0] >> 8) & 0xFF;
		bytes[5] = (entropy[0] >> 16) & 0xFF;
		bytes[4] = (entropy[0] >> 24) & 0xFF;
		bytes[3] = (entropy[0] >> 32) & 0xFF;
		bytes[2] = (entropy[0] >> 40) & 0xFF;
		bytes[1] = (entropy[0] >> 48) & 0xFF;
		bytes[0] = (entropy[0] >> 56) & 0xFF;
		
		sha256((uint32_t*)bytes, 16, (uint32_t*)entropy_hash);
		achievedChecksum = (entropy_hash[0] >> 4) & 0x0F;

		bChkSumFailed = (achievedChecksum != reqChecksum);

		if (bChkSumFailed) {
			atomicAdd(&ourBlockProcExtra,1);
			nAlternateCandidateRemaining--;
		}
		else {
			nAlternateCandidateRemaining = 0; //no need to process more in this thread-batch
		}
	} //do

	__syncthreads(); // Synchronize to and check if have a valid checksum to continue with
	if (bChkSumFailed == false) { //scrutinize
		atomicAdd(&ourBlockGoodChkSum, 1);

		uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
		uint8_t* mnemonic = mnemonic_phrase;
		uint32_t ipad[256 / 4];
		uint32_t opad[256 / 4];
		uint32_t seed[64 / 4];


		//Work with Current Entropy
		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0);
		//entropy_to_mnemonic(entropy, mnemonic);
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)ipad + x) = 0x3636363636363636ULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)opad + x) = 0x5C5C5C5C5C5C5C5CULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x + 128 / 4] = *(uint32_t*)((uint32_t*)&salt_swap + x);
		}
		sha512_swap((uint64_t*)ipad, 140, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			seed[x] = ipad[128 / 4 + x];
		}
		for (int x = 1; x < 2048; x++) {
			sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				seed[x] = seed[x] ^ ipad[128 / 4 + x];
			}
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x] = 0x36363636 ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			opad[x] = 0x5C5C5C5C ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			ipad[x + 128 / 4] = seed[x];
		}
		//ipad[192 / 4] = 0;
		//opad[192 / 4] = 0;
		sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 128 / 8; x++) {
			*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x) = SWAP512(*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x));
		}
		key_to_hash160((extended_private_key_t*)&ipad[128 / 4], tables_legacy, tables_segwit, tables_native_segwit, (uint32_t*)mnemonic, ret);
		//__syncthreads();
	}
	__syncthreads(); // Synchronize to ensure all data is loaded
	if (threadIdx.x == 0) {
		atomicAdd(nBatchPlannedProc , ourBlockProcNormal);
		atomicAdd(nBatchMoreProc, ourBlockProcExtra);
	}

}//DICTIONARY ATTACK

__host__ /*and */ __device__
void AdaptiveUpdateMnemonicLow64(uint64_t* low64
	, int16_t digitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
	, int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS]
)

{
	uint64_t tmpHigh = *low64;
	uint64_t tmpAns = tmpHigh;

	tmpAns = tmpHigh >> 62;
	tmpAns = tmpAns << 2;

	for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS - 1; i++) {
		tmpAns = tmpAns << 11;
		tmpAns & 0xFFFFFFFFFFFFF7F;
		tmpAns |= (uint64_t)(digitSet[i][curDigits[i]]);
	}
	tmpAns = tmpAns << 7;
	tmpAns |= ((uint64_t)(digitSet[MAX_ADAPTIVE_BASE_POSITIONS - 1][curDigits[MAX_ADAPTIVE_BASE_POSITIONS - 1]]) >> 4);

	*low64 = tmpAns;
}


