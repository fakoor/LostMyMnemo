#include "hip/hip_runtime.h"
#include <stdafx.h>

#include "hip/hip_runtime.h"
#include ""
#include "AdaptiveBase.h"
#include <GPU.h>
#include <hip/hip_runtime.h>


/*
* We onsider Mnemonics a Base - 2048 twelve-digit unsigned integer,
* if we have some information about any of digits, so that we can
* ommit 1 to 2047 digits (words) from a specific position, then that
* digit is adaptively based to Base-2047 down to Base-2 aka binary.
* The only mathematic operation we need for the purpose is increment,
* When we reach the carry for a VariableBase digit, uppon an increment,
* we will rewind that digit and increment the more significant digit
* DUE to less memory usage in constant-memory, we limit such mechanism
* to the 6 least significant mnemonic words in a 262 guessed space
* Hence the dictionary for positions 6 to 12 outght not to include
* more than 262 words. The other structure provides reverse lookup for a value of a digit
*/




__constant__ AdaptiveStructConstType dev_adaptiveConsts;
__device__ AdaptiveStructVarType dev_adaptiveVars;

__host__ /* __and__ */ __device__ void IncrementAdaptiveDigits(int16_t* inDigits, uint64_t howMuch, int16_t* outDigits) {
	uint64_t nYetToAdd = howMuch;
	uint64_t nCarryValue = 0;

	for (int i = MAX_ADAPTIVE_BASE_POSITIONS - 1; i >= 0; i--) {
		if (nYetToAdd == 0 && nCarryValue == 0) {
			outDigits[i] = inDigits[i];
			continue;
		}

		int16_t beforeIncDigit = dev_adaptiveConsts.dev_AdaptiveBaseCurrentBatchInitialDigits[i];
		int nCarryAt = dev_adaptiveConsts.dev_AdaptiveBaseDigitCarryTrigger[i];

		int nThisIdeal = nYetToAdd + beforeIncDigit + nCarryValue;
		int nThisNewDigit = nThisIdeal % nCarryAt;


		outDigits[i] = nThisNewDigit;
		nCarryValue = nThisIdeal / nCarryAt;
		nYetToAdd = 0; //all active in carry if any
	}
	if (nYetToAdd != 0 || nCarryValue != 0) {
		//ASSERT: We have carried out of our space, NOP anyway
	}
}


__host__ /* __and__ */ __device__ void GetBipForAdaptiveDigit(int16_t* inDigits, uint8_t pos, uint64_t* outBip) {
	int16_t curAdapriveDigit = inDigits[pos];
	*outBip = dev_adaptiveConsts.dev_AdaptiveBaseDigitSet[pos][curAdapriveDigit];
}
__host__ /* __and__ */ __device__ void GetPaddedBipForAdaptiveDigit(int16_t* inDigits, uint8_t pos, uint64_t* outPadBip) {
	uint64_t curBipForDigit;
	GetBipForAdaptiveDigit(inDigits, pos, &curBipForDigit);
	uint8_t shiftCount;
	if (pos < MAX_ADAPTIVE_BASE_POSITIONS - 1) {
		shiftCount = (128 - 66 - pos * 11);
		*outPadBip = curBipForDigit << shiftCount;
	}
	else {
		shiftCount = 4;
		*outPadBip = curBipForDigit >> shiftCount;
	}
}
__host__ /* __and__ */ __device__ void AdaptiveDigitsToEntropy(int16_t* inDigits, uint64_t* outEntropy, uint8_t* checkSum) {

	outEntropy[0] = dev_adaptiveConsts.dev_EntropyAbsolutePrefix64;
	outEntropy[1] = dev_adaptiveConsts.dev_EntropyBatchNext24;

	uint64_t digitPaddedBip;
	for (int pos = 0; pos < MAX_ADAPTIVE_BASE_POSITIONS; pos++) {
		GetPaddedBipForAdaptiveDigit(inDigits, pos, &digitPaddedBip);
		outEntropy[1] |= digitPaddedBip;
	}

	uint64_t lastWord;
	GetBipForAdaptiveDigit(inDigits, MAX_ADAPTIVE_BASE_POSITIONS, &lastWord);
	*checkSum = lastWord & 0x000F;
}


__global__ void gl_DictionaryAttack(
	const uint64_t* __restrict__ entropy,
	const tableStruct* __restrict__ tables_legacy,
	const tableStruct* __restrict__ tables_segwit,
	const tableStruct* __restrict__ tables_native_segwit,
	retStruct* __restrict__ ret
)
{

	//TODO: Each thread picks is load from Incremental Base!
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS];
	uint64_t curEntropy[2];
	uint8_t reqChecksum;

	IncrementAdaptiveDigits(dev_adaptiveConsts.dev_AdaptiveBaseCurrentBatchInitialDigits, idx, curDigits);
	AdaptiveDigitsToEntropy(curDigits, curEntropy, &reqChecksum);

	uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
	uint8_t* mnemonic = mnemonic_phrase;
	uint32_t ipad[256 / 4];
	uint32_t opad[256 / 4];
	uint32_t seed[64 / 4];

	entropy_to_mnemonic(entropy, mnemonic);
#pragma unroll
	for (int x = 0; x < 120 / 8; x++) {
		*(uint64_t*)((uint64_t*)ipad + x) = 0x3636363636363636ULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
	}
#pragma unroll
	for (int x = 0; x < 120 / 8; x++) {
		*(uint64_t*)((uint64_t*)opad + x) = 0x5C5C5C5C5C5C5C5CULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
	}
#pragma unroll
	for (int x = 120 / 4; x < 128 / 4; x++) {
		ipad[x] = 0x36363636;
	}
#pragma unroll
	for (int x = 120 / 4; x < 128 / 4; x++) {
		opad[x] = 0x5C5C5C5C;
	}
#pragma unroll
	for (int x = 0; x < 16 / 4; x++) {
		ipad[x + 128 / 4] = *(uint32_t*)((uint32_t*)&salt_swap + x);
	}
	sha512_swap((uint64_t*)ipad, 140, (uint64_t*)&opad[128 / 4]);
	sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
	for (int x = 0; x < 64 / 4; x++) {
		seed[x] = ipad[128 / 4 + x];
	}
	for (int x = 1; x < 2048; x++) {
		sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			seed[x] = seed[x] ^ ipad[128 / 4 + x];
		}
	}
#pragma unroll
	for (int x = 0; x < 16 / 4; x++) {
		ipad[x] = 0x36363636 ^ *(uint32_t*)((uint32_t*)&key_swap + x);
	}
#pragma unroll
	for (int x = 0; x < 16 / 4; x++) {
		opad[x] = 0x5C5C5C5C ^ *(uint32_t*)((uint32_t*)&key_swap + x);
	}
#pragma unroll
	for (int x = 16 / 4; x < 128 / 4; x++) {
		ipad[x] = 0x36363636;
	}
#pragma unroll
	for (int x = 16 / 4; x < 128 / 4; x++) {
		opad[x] = 0x5C5C5C5C;
	}
#pragma unroll
	for (int x = 0; x < 64 / 4; x++) {
		ipad[x + 128 / 4] = seed[x];
	}
	//ipad[192 / 4] = 0;
	//opad[192 / 4] = 0;
	sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
	sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
	for (int x = 0; x < 128 / 8; x++) {
		*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x) = SWAP512(*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x));
	}
	key_to_hash160((extended_private_key_t*)&ipad[128 / 4], tables_legacy, tables_segwit, tables_native_segwit, (uint32_t*)mnemonic, ret);
	//__syncthreads();
}

