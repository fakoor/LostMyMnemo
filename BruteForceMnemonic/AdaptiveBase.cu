#include <stdafx.h>
#include <stdio.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include <GPU.h>
#include "AdaptiveBase.h"



/*
* We onsider Mnemonics a Base - 2048 twelve-digit unsigned integer,
* if we have some information about any of digits, so that we can
* ommit 1 to 2047 digits (words) from a specific position, then that
* digit is adaptively based to Base-2047 down to Base-2 aka binary.
* The only mathematic operation we need for the purpose is increment,
* When we reach the carry for a VariableBase digit, uppon an increment,
* we will rewind that digit and increment the more significant digit
* DUE to less memory usage in constant-memory, we limit such mechanism
* to the 6 least significant mnemonic words in a 262 guessed space
* Hence the dictionary for positions 6 to 12 outght not to include
* more than 262 words. The other structure provides reverse lookup 
* for a value of a digit
*/



__device__ int dev_checkResult(retStruct* ret) {

	if (ret->f[0].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[0].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[1].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[1].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[2].count_found >= MAX_FOUND_ADDRESSES)
	{
		ret->f[2].count_found = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[0].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[0].count_found_bytes = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[1].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[1].count_found_bytes = MAX_FOUND_ADDRESSES;
	}
	if (ret->f[2].count_found_bytes >= MAX_FOUND_ADDRESSES)
	{
		ret->f[2].count_found_bytes = MAX_FOUND_ADDRESSES;

	}

	if (ret->f[0].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[0].count_found; i++)
		{
			return 1;
		}
	}
	if (ret->f[1].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[1].count_found; i++)
		{
			return 1;
		}
	}
	if (ret->f[2].count_found != 0)
	{
		for (uint32_t i = 0; i < ret->f[2].count_found; i++)
		{
			return 1;
		}
	}

	return 0;
}



__constant__ uint64_t dev_EntropyAbsolutePrefix64[1];
uint64_t host_EntropyAbsolutePrefix64[1];

__constant__ int16_t dev_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS];
 int16_t host_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS];

__constant__ int16_t dev_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION];
 int16_t host_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION];


__constant__ int16_t dev_AdaptiveBaseCurrentBatchInitialDigits[MAX_ADAPTIVE_BASE_POSITIONS];
 int16_t host_AdaptiveBaseCurrentBatchInitialDigits[MAX_ADAPTIVE_BASE_POSITIONS];


__constant__ uint64_t dev_EntropyNextPrefix2[1]; //Per-Batch Const
 uint64_t host_EntropyNextPrefix2[1]; //Per-Batch Const

 
 
 __host__ /* __and__ */ __device__ void GetBipForAdaptiveDigit(
	  int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t* local_AdaptiveBaseDigitCarryTrigger
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
	, int16_t* inDigits, uint8_t pos, uint64_t* outBip) {
	int16_t curAdapriveDigit = inDigits[pos];
	*outBip = local_AdaptiveBaseDigitSet[pos][curAdapriveDigit];
}
__host__ /* __and__ */ __device__ void GetPaddedBipForAdaptiveDigit(
	int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t* local_AdaptiveBaseDigitCarryTrigger
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION], int16_t* inDigits, uint8_t pos, uint64_t* outPadBip) {
	uint64_t curBipForDigit;

	GetBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
		, local_AdaptiveBaseDigitCarryTrigger
		, local_AdaptiveBaseDigitSet, inDigits, pos, &curBipForDigit);

	uint8_t shiftCount;
	if (pos < MAX_ADAPTIVE_BASE_POSITIONS - 1) {
		shiftCount = (128 - 66 - pos * 11);
		*outPadBip = curBipForDigit << shiftCount;
	}
	else {
		shiftCount = 4;
		*outPadBip = curBipForDigit >> shiftCount;
	}
}
__host__ /* __and__ */ __device__ void AdaptiveDigitsToEntropy(
	int16_t* local_AdaptiveBaseCurrentBatchInitialDigits
	, int16_t local_AdaptiveBaseDigitCarryTrigger[MAX_ADAPTIVE_BASE_POSITIONS]
	, int16_t local_AdaptiveBaseDigitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION],
	uint64_t * local_EntropyAbsolutePrefix64,
	uint64_t * local_EntropyBatchNext24, 
	 int16_t* inDigits, uint64_t* outEntropy, uint8_t* checkSum) {

	outEntropy[0] = local_EntropyAbsolutePrefix64[0];
	outEntropy[1] = local_EntropyBatchNext24[0];

	uint64_t digitPaddedBip;
	for (int pos = 0; pos < MAX_ADAPTIVE_BASE_POSITIONS; pos++) {
		GetPaddedBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
			, local_AdaptiveBaseDigitCarryTrigger
			,local_AdaptiveBaseDigitSet, inDigits, pos, &digitPaddedBip);
		outEntropy[1] |= digitPaddedBip;
	}

	uint64_t lastWord;
	GetBipForAdaptiveDigit(local_AdaptiveBaseCurrentBatchInitialDigits
		, local_AdaptiveBaseDigitCarryTrigger
		, local_AdaptiveBaseDigitSet, inDigits, MAX_ADAPTIVE_BASE_POSITIONS, &lastWord);
	*checkSum = lastWord & 0x000F;
}



__host__ /*and */ __device__
void AdaptiveUpdateMnemonicLow64(uint64_t* low64
	, int16_t digitSet[MAX_ADAPTIVE_BASE_POSITIONS][MAX_ADAPTIVE_BASE_VARIANTS_PER_POSITION]
	, int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS]
)

{
	uint64_t tmpHigh = *low64;
	uint64_t tmpAns = tmpHigh;

	tmpAns = tmpHigh >> 62;
	tmpAns = tmpAns << 2;

	for (int i = 0; i < MAX_ADAPTIVE_BASE_POSITIONS - 1; i++) {
		tmpAns = tmpAns << 11;
		tmpAns & 0xFFFFFFFFFFFFF7F;
		tmpAns |= (uint64_t)(digitSet[i][curDigits[i]]);
	}
	tmpAns = tmpAns << 7;
	tmpAns |= ((uint64_t)(digitSet[MAX_ADAPTIVE_BASE_POSITIONS - 1][curDigits[MAX_ADAPTIVE_BASE_POSITIONS - 1]]) >> 4);

	*low64 = tmpAns;
}


