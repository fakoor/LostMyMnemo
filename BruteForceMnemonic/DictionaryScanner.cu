#include "hip/hip_runtime.h"
#include <stdafx.h>
#include <stdio.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "GPU.h"
#include "AdaptiveBase.h"

#include "EntropyTools.cuh"
#include "DictionaryScanner.cuh"


__device__
int LookupHash(const uint32_t* hash, uint32_t* hash_from_table, const uint32_t* mnemonic, foundStruct* fnd_ret, uint32_t path, uint32_t child)
{
	int found = 0;
	bool search_state = true;
	uint32_t line_cnt = 1;
	uint32_t point = 0;
	uint32_t point_last = 0;
	uint32_t interval = line_cnt / 3;
	//uint32_t* hash_from_table;
	while (point < line_cnt) {
		point_last = point;
		if (interval == 0) {
			search_state = false;
		}
		if (search_state) {
			point += interval;

			if (point >= line_cnt) {
				point = point_last;
				interval = (line_cnt - point) / 2;
				continue;
			}
			//hash_from_table = &table.table[point * (20 / 4)];
		}
		else {
			//hash_from_table = &table.table[point * (20 / 4)];
			point += 1;
		}

		int cmp = 0;
		if (hash[0] < hash_from_table[0])
		{
			cmp = -1;
		}
		else if (hash[0] > hash_from_table[0])
		{
			cmp = 1;
		}
		else if (hash[1] < hash_from_table[1])
		{
			cmp = -2;
		}
		else if (hash[1] > hash_from_table[1])
		{
			cmp = 2;
		}
		else if (hash[2] < hash_from_table[2])
		{
			cmp = -3;
		}
		else if (hash[2] > hash_from_table[2])
		{
			cmp = 3;
		}
		else if (hash[3] < hash_from_table[3])
		{
			cmp = -4;
		}
		else if (hash[3] > hash_from_table[3])
		{
			cmp = 4;
		}
		else if (hash[4] < hash_from_table[4])
		{
			cmp = -5;
		}
		else if (hash[4] > hash_from_table[4])
		{
			cmp = 5;
		}

		if (search_state) {
			if (cmp < 0) {
				if (interval < 20) {
					search_state = false;
				}
				else
				{
					interval = interval / 2;
				}
				point = point_last;
				continue;
			}
			else if (cmp == 0) {
				search_state = false;
			}
			else {
				continue;
			}
		}

		if (cmp <= 0) {
			if (cmp == 0)
			{
				found = 1;
				uint32_t cnt = fnd_ret->count_found;
				fnd_ret->count_found++;
				if (cnt < MAX_FOUND_ADDRESSES)
				{
					for (int i = 0; i < 5; i++) fnd_ret->found_info[cnt].hash160[i] = hash[i];
					for (int i = 0; i < SIZE32_MNEMONIC_FRAME; i++) fnd_ret->found_info[cnt].mnemonic[i] = mnemonic[i];
					fnd_ret->found_info[cnt].path = path;
					fnd_ret->found_info[cnt].child = child;
				}
			}
			break;
		}

		if (cmp > 1) {
			if (dev_num_bytes_find[0] == 8) {
				if (hash[1] == hash_from_table[1]) found = 2;
			}
#ifdef TEST_MODE
			else if (dev_num_bytes_find[0] == 7) {
				if ((hash[1] & 0x00FFFFFF) == (hash_from_table[1] & 0x00FFFFFF)) found = 2;
			}
			else if (dev_num_bytes_find[0] == 6) {
				if ((hash[1] & 0x0000FFFF) == (hash_from_table[1] & 0x0000FFFF)) found = 2;
			}
			else if (dev_num_bytes_find[0] == 5) {
				if ((hash[1] & 0x000000FF) == (hash_from_table[1] & 0x000000FF)) found = 2;
			}
#endif //TEST_MODE
		}


		if (found == 2) {
			uint32_t cnt = fnd_ret->count_found_bytes;
			fnd_ret->count_found_bytes++;
			if (cnt < MAX_FOUND_ADDRESSES)
			{
				for (int i = 0; i < 5; i++)
				{
					fnd_ret->found_bytes_info[cnt].hash160_from_table[i] = hash_from_table[i];
					fnd_ret->found_bytes_info[cnt].hash160[i] = hash[i];
				}
				for (int i = 0; i < SIZE32_MNEMONIC_FRAME; i++) fnd_ret->found_bytes_info[cnt].mnemonic[i] = mnemonic[i];
				fnd_ret->found_bytes_info[cnt].path = path;
				fnd_ret->found_bytes_info[cnt].child = child;
			}
			break;
		}

	}

	return found;
}


__global__ void gl_DictionaryScanner(
	const uint64_t* __restrict__ nProcessedIterations,
	uint64_t* nProcessedInstances,
	const tableStruct* __restrict__ tables_legacy,
	const tableStruct* __restrict__ tables_segwit,
	const tableStruct* __restrict__ tables_native_segwit,
	retStruct* __restrict__ ret
)
{
	unsigned int effective_idx = blockIdx.x * blockDim.x + threadIdx.x;

	uint32_t nTotalThreads = blockDim.x * gridDim.x;

	__shared__ uint64_t ourBlockProcNormal;
	__shared__ unsigned int nMoreIterated;
	__shared__ uint64_t nGridJobCap;
	__shared__ int bDone;

	int16_t local_static_word_index[12];

	// Initialize the shared variable
	if (threadIdx.x == 0) {
		ourBlockProcNormal = 0; // Only the first thread initializes it

		nMoreIterated = 0;
		nGridJobCap = ULLONG_MAX;//0xFFFFFFFFFFFFFFFFull;
		bDone = 0;
	}
	__syncthreads(); // Synchronize to ensure the initialization is complete

	for (int i = 0; i < 6; i++) {
		local_static_word_index[i] = dev_static_words_indices[i];
	}


	uint64_t curEntropy[2];
	curEntropy[0] = dev_EntropyAbsolutePrefix64[PTR_AVOIDER];
	curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];


	uint8_t reqChecksum = 0;
	uint8_t achievedChecksum = 1;
	bool bChkMatched = false;

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS] = { 
		 dev_AdaptiveBaseCurrentBatchInitialDigits[0]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[1]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[2]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[3]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[4]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[5] 
	};

	//TODO block: prefix is based on  words 9 and 10 while the last word 11 is iterated inside the thread

	int nTried = 0;
	bool bCouldAdd = false;
	const int16_t lastPos_adaptive = MAX_ADAPTIVE_BASE_POSITIONS - 1;
	const int16_t lastPosCarryTrig = dev_AdaptiveBaseDigitCarryTrigger[lastPos_adaptive];

	//instead of effective_idx, increment so that bypass and leave last word iteration completely to this thread

	SyncBipIndexFromAdaptiveDigits(local_static_word_index , dev_AdaptiveBaseDigitSet , curDigits);

	//Work with Current Entropy
	uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
	uint8_t* mnemonic = mnemonic_phrase;
	uint64_t nLoopMasterOffset = effective_idx * lastPosCarryTrig + *nProcessedIterations * nTotalThreads;
	for (int16_t nWordElevenOffset = 0; nWordElevenOffset < lastPosCarryTrig; nWordElevenOffset++) {
		//break on nTried < MAX_TRY_PER_THREAD
		uint64_t nInstanceOffset = nLoopMasterOffset + nWordElevenOffset;

		if (nInstanceOffset > nGridJobCap) {
			//if (blockIdx.x == 0) {
			//	printf("\r\nBlock Job done at:%llu\r\n", nInstanceOffset);
			//}
			break;
		}

		bCouldAdd = IncrementAdaptiveDigits(
			dev_AdaptiveBaseDigitCarryTrigger
			, dev_AdaptiveBaseCurrentBatchInitialDigits
			, nInstanceOffset, curDigits);

		if (bDone)
			break;

		if (bCouldAdd == false /*&& lastPosCarryTrig == nWordElevenOffset*/) {
			//if (effective_idx == nMaxCloudAdd + 1) {
			//	printf("Can not add bulk at %x", lastPosCarryTrig);
			//}

			atomicMin(&nGridJobCap, nInstanceOffset);
			break;
		}
		atomicAdd(&ourBlockProcNormal, 1);
		//else {
		//	atomicMax(&nMaxCloudAdd, effective_idx);
		//}

		SyncBipIndexFromAdaptiveDigits(local_static_word_index, dev_AdaptiveBaseDigitSet, curDigits);

		//int16_t word_11_BIP = dev_AdaptiveBaseDigitSet[lastPos_adaptive][nWordElevenOffset];
		//curDigits[lastPos_adaptive] = nWordElevenOffset;
		curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];
		AdaptiveUpdateMnemonicLow64(&curEntropy[1], dev_AdaptiveBaseDigitSet, curDigits);
		//local_static_word_index[11] = word_11_BIP;
		int16_t wordElevenBipVal = local_static_word_index[11];


#if 0 //not required for checksum comparison here
		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);
#endif
		reqChecksum = wordElevenBipVal & 0x000F;

		uint8_t entropy_hash[32];
		uint8_t bytes[16];
		uint64_t* entropy = curEntropy;

		bytes[15] = entropy[1] & 0xFF;
		bytes[14] = (entropy[1] >> 8) & 0xFF;
		bytes[13] = (entropy[1] >> 16) & 0xFF;
		bytes[12] = (entropy[1] >> 24) & 0xFF;
		bytes[11] = (entropy[1] >> 32) & 0xFF;
		bytes[10] = (entropy[1] >> 40) & 0xFF;
		bytes[9] = (entropy[1] >> 48) & 0xFF;
		bytes[8] = (entropy[1] >> 56) & 0xFF;

		bytes[7] = entropy[0] & 0xFF;
		bytes[6] = (entropy[0] >> 8) & 0xFF;
		bytes[5] = (entropy[0] >> 16) & 0xFF;
		bytes[4] = (entropy[0] >> 24) & 0xFF;
		bytes[3] = (entropy[0] >> 32) & 0xFF;
		bytes[2] = (entropy[0] >> 40) & 0xFF;
		bytes[1] = (entropy[0] >> 48) & 0xFF;
		bytes[0] = (entropy[0] >> 56) & 0xFF;

		sha256((uint32_t*)bytes, 16, (uint32_t*)entropy_hash);
		achievedChecksum = (entropy_hash[0] >> 4) & 0x0F;

		bChkMatched = (achievedChecksum == reqChecksum);

		nTried++;
#if 0
		if (effective_idx <= 2 || (effective_idx <= 242 && effective_idx >= 240)) {
			uint8_t word_11_text[10];
			GetWordFromBipIndex(wordElevenBipVal, word_11_text);
			//entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);
			printf("idx:%u [+ %llu ] @%s (%d : %d) on [%s]  CHK: %s req=%u ach=%u \r\n"
				, effective_idx
				, nInstanceOffset
				, word_11_text
				, nWordElevenOffset
				, wordElevenBipVal
				, mnemonic
				, (bChkMatched) ?  "OK" : "Bad"
				, reqChecksum
				, achievedChecksum
			);
		}
#endif
#if 1
		if (!bChkMatched) {
			continue;
		}
#endif
		//NOTE : If we reach here the checksum is already matching, just need to check the address
		//__syncthreads(); // Synchronize to and check if have a valid checksum to continue with
		//if (bChkMatched) { //scrutinize : bCouldAdd

		uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
		uint8_t* mnemonic = mnemonic_phrase;
		uint32_t ipad[256 / 4];
		uint32_t opad[256 / 4];
		uint32_t seed[64 / 4];


		//Work with Current Entropy
		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);

		//if (idx == 0) {
//			printf("nemo-%u  (retry.remain=%d/%d) = :%s \r\n\r\n", effective_idx,nTried,MAX_TRY_PER_THREAD, mnemonic);
		//}
		//entropy_to_mnemonic(entropy, mnemonic);
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)ipad + x) = 0x3636363636363636ULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)opad + x) = 0x5C5C5C5C5C5C5C5CULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x + 128 / 4] = *(uint32_t*)((uint32_t*)&salt_swap + x);
		}
		sha512_swap((uint64_t*)ipad, 140, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			seed[x] = ipad[128 / 4 + x];
		}
		for (int x = 1; x < 2048; x++) {
			sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				seed[x] = seed[x] ^ ipad[128 / 4 + x];
			}
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x] = 0x36363636 ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			opad[x] = 0x5C5C5C5C ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			ipad[x + 128 / 4] = seed[x];
		}
		//ipad[192 / 4] = 0;
		//opad[192 / 4] = 0;
		sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 128 / 8; x++) {
			*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x) = SWAP512(*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x));
		}

		//printf("END block %d - thread  %d - EffectiveId:%d - curDigits:%d-%d-%d-%d-%d-%d %s\r\n", blockId, threadId, effective_idx
		//	, curDigits[0], curDigits[1], curDigits[2], curDigits[3], curDigits[4], curDigits[5] , mnemonic);

		if (bDone)
			break;
		//dev_uniqueTargetAddressBytes;
		{
			const extended_private_key_t* master_private = (extended_private_key_t*)&ipad[128 / 4];

			uint32_t hash[(20 / 4)];
			extended_private_key_t target_key;
			extended_private_key_t target_key_fo_pub;
			extended_private_key_t master_private_fo_extint;
			extended_public_key_t target_public_key;

			hardened_private_child_from_private(master_private, &target_key, 44);
			hardened_private_child_from_private(&target_key, &target_key, 0);
			hardened_private_child_from_private(&target_key, &master_private_fo_extint, 0);

			normal_private_child_from_private(&master_private_fo_extint, &target_key, 0);
			//m/44'/0'/0'/0/x
			normal_private_child_from_private(&target_key, &target_key_fo_pub, 0);
			calc_public(&target_key_fo_pub, &target_public_key);
			calc_hash160(&target_public_key, hash);

			//find_hash_in_table(hash, tables_legacy[(uint8_t)hash[0]], (uint32_t*) mnemonic, &ret->f[0], 4, 0);
			LookupHash(hash, (uint32_t*) dev_uniqueTargetAddressBytes, (uint32_t*)mnemonic, &ret->f[0], 4, 0);


		}

#if 0
		key_to_hash160((extended_private_key_t*)&ipad[128 / 4], tables_legacy, tables_segwit, tables_native_segwit, (uint32_t*)mnemonic, ret);
#endif

		atomicMax(&bDone, DictionaryCheckFound(ret));
		if (bDone ) {
			atomicMin(&nGridJobCap, nInstanceOffset);
			//if (nInstanceOffset == nGridJobCap) {
			//	printf("\r\n\r\n\tBreaking operation at %llu Since Match is Found!\r\n\r\n", nInstanceOffset);
			//}

			break;
		}
	}//for 

	__syncthreads(); // Synchronize to ensure all data is loaded
	if (threadIdx.x == 0) {
		atomicAdd(nProcessedInstances, ourBlockProcNormal);
	}

}//DICTIONARY ATTACK
