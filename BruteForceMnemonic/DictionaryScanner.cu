#include "hip/hip_runtime.h"
#include <stdafx.h>
#include <stdio.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "GPU.h"
#include "AdaptiveBase.h"

#include "EntropyTools.cuh"
#include "DictionaryScanner.cuh"
#include "Bip39Tools.cuh"



static inline __device__ int device_hashcmp(const  uint32_t* p1, const uint32_t* p2) {
#pragma unroll
	for (auto i = 0; i < 20/4; ++i) {
		if (p1[i] != p2[i]) {
			return p1[i] < p2[i] ? -1 : 1; // Return -1 if p1 < p2, 1 if p1 > p2
		}
	}
	return 0; // Memory regions are equal
}


__global__ void gl_DictionaryScanner(
	const uint64_t* __restrict__ nProcessingIteration,
	uint64_t* nProcessedInstances
)
{
	unsigned int effective_idx = blockIdx.x * blockDim.x + threadIdx.x;

	uint32_t nTotalThreads = blockDim.x * gridDim.x;

	__shared__ uint64_t ourBlockProcNormal;
	__shared__ uint64_t nGridJobCap;
	__shared__ uint8_t bDone;

	int16_t local_static_word_index[12];

	// Initialize the shared variable (first thread of each block)
	if (threadIdx.x == 0) {
		ourBlockProcNormal = 0;
		nGridJobCap = ULLONG_MAX;
		bDone = 0;
	}
	__syncthreads(); // Synchronize to ensure the initialization is complete

	for (int i = 0; i < 6; i++) {
		local_static_word_index[i] = dev_static_words_indices[i];
	}


	uint64_t curEntropy[2];
	curEntropy[0] = dev_EntropyAbsolutePrefix64[PTR_AVOIDER];
	curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];


	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS] = {
		 dev_AdaptiveBaseCurrentBatchInitialDigits[0]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[1]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[2]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[3]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[4]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[5]
	};

	//TODO block: prefix is based on  words 9 and 10 while the last word 11 is iterated inside the thread

	const int16_t nPosElevenAdaptiveIdx = MAX_ADAPTIVE_BASE_POSITIONS - 1;
	const int16_t nPosElevenCarryTrig = dev_AdaptiveBaseDigitCarryTrigger[nPosElevenAdaptiveIdx];
	const int16_t nPostTenAdaptiveIdx = nPosElevenAdaptiveIdx - 1;
	const int16_t nPosTenCarryTrig = dev_AdaptiveBaseDigitCarryTrigger[nPostTenAdaptiveIdx];

	//instead of effective_idx, increment so that bypass and leave last word iteration completely to this thread

	SyncBipIndexFromAdaptiveDigits(local_static_word_index, dev_AdaptiveBaseDigitSet, curDigits);

	//Work with Current Entropy
	uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
	uint8_t* mnemonic = mnemonic_phrase;
	uint64_t nLoopMasterOffset = effective_idx * nPosElevenCarryTrig * nPosTenCarryTrig 
		+ *nProcessingIteration * (nTotalThreads * nPosElevenCarryTrig * nPosTenCarryTrig);

	uint8_t checkSumInputBlock[16];//with constant portion initalization
	checkSumInputBlock[7] = dev_EntropyAbsolutePrefix64[0] & 0xFF;
	checkSumInputBlock[6] = (dev_EntropyAbsolutePrefix64[0] >> 8) & 0xFF;
	checkSumInputBlock[5] = (dev_EntropyAbsolutePrefix64[0] >> 16) & 0xFF;
	checkSumInputBlock[4] = (dev_EntropyAbsolutePrefix64[0] >> 24) & 0xFF;
	checkSumInputBlock[3] = (dev_EntropyAbsolutePrefix64[0] >> 32) & 0xFF;
	checkSumInputBlock[2] = (dev_EntropyAbsolutePrefix64[0] >> 40) & 0xFF;
	checkSumInputBlock[1] = (dev_EntropyAbsolutePrefix64[0] >> 48) & 0xFF;
	checkSumInputBlock[0] = (dev_EntropyAbsolutePrefix64[0] >> 56) & 0xFF;


	//#pragma unroll
	for (int16_t nWordTenOffset = 0; nWordTenOffset < nPosTenCarryTrig; nWordTenOffset++) {
		for (int16_t nWordElevenOffset = 0; nWordElevenOffset < nPosElevenCarryTrig; nWordElevenOffset++) {
			uint64_t nInstanceOffset = 
				  nLoopMasterOffset 
				+ nWordTenOffset * nPosElevenCarryTrig
				+ nWordElevenOffset
			;
			if (nInstanceOffset > nGridJobCap) {
				break;
			}

			if (false == IncrementAdaptiveDigits(
				dev_AdaptiveBaseDigitCarryTrigger
				, dev_AdaptiveBaseCurrentBatchInitialDigits
				, nInstanceOffset, curDigits)) {

				atomicMin(&nGridJobCap, nInstanceOffset);
				break;
			}
			atomicAdd(&ourBlockProcNormal, 1);


			SyncBipIndexFromAdaptiveDigits(local_static_word_index, dev_AdaptiveBaseDigitSet, curDigits);

			curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];
			AdaptiveUpdateMnemonicLow64(&curEntropy[1], dev_AdaptiveBaseDigitSet, curDigits);
			int16_t wordElevenBipVal = local_static_word_index[11];


			uint8_t reqChecksum = wordElevenBipVal & 0x000F;
			bool bChkMatched = CheckSumValidate(checkSumInputBlock, curEntropy, reqChecksum);


			if (!bChkMatched) {
				continue;
			}
			//NOTE : If we reach here the checksum is already matching, just need to check the address
			//__syncthreads(); // Synchronize to and check if have a valid checksum to continue with
			//if (bChkMatched) { //scrutinize : bCouldAdd

			uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
			uint8_t* mnemonic = mnemonic_phrase;
			uint32_t ipad[256 / 4];
			uint32_t opad[256 / 4];
			uint32_t seed[64 / 4];


			//Work with Current Entropy
			//entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);
			IndicesToMnemonic(local_static_word_index, (uint8_t*)mnemonic, words, word_lengths);

#pragma unroll
			for (int x = 0; x < 120 / 8; x++) {
				*(uint64_t*)((uint64_t*)ipad + x) = 0x3636363636363636ULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
			}
#pragma unroll
			for (int x = 0; x < 120 / 8; x++) {
				*(uint64_t*)((uint64_t*)opad + x) = 0x5C5C5C5C5C5C5C5CULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
			}
#pragma unroll
			for (int x = 120 / 4; x < 128 / 4; x++) {
				ipad[x] = 0x36363636;
			}
#pragma unroll
			for (int x = 120 / 4; x < 128 / 4; x++) {
				opad[x] = 0x5C5C5C5C;
			}
#pragma unroll
			for (int x = 0; x < 16 / 4; x++) {
				ipad[x + 128 / 4] = *(uint32_t*)((uint32_t*)&salt_swap + x);
			}
			sha512_swap((uint64_t*)ipad, 140, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				seed[x] = ipad[128 / 4 + x];
			}
			for (int x = 1; x < 2048; x++) {
				sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
				sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
				for (int x = 0; x < 64 / 4; x++) {
					seed[x] = seed[x] ^ ipad[128 / 4 + x];
				}
			}
#pragma unroll
			for (int x = 0; x < 16 / 4; x++) {
				ipad[x] = 0x36363636 ^ *(uint32_t*)((uint32_t*)&key_swap + x);
			}
#pragma unroll
			for (int x = 0; x < 16 / 4; x++) {
				opad[x] = 0x5C5C5C5C ^ *(uint32_t*)((uint32_t*)&key_swap + x);
			}
#pragma unroll
			for (int x = 16 / 4; x < 128 / 4; x++) {
				ipad[x] = 0x36363636;
			}
#pragma unroll
			for (int x = 16 / 4; x < 128 / 4; x++) {
				opad[x] = 0x5C5C5C5C;
			}
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				ipad[x + 128 / 4] = seed[x];
			}
			//ipad[192 / 4] = 0;
			//opad[192 / 4] = 0;
			sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 128 / 8; x++) {
				*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x) = SWAP512(*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x));
			}

			{
				const extended_private_key_t* master_private = (extended_private_key_t*)&ipad[128 / 4];

				uint32_t hash[(20 / 4)];
				extended_private_key_t target_key;
				extended_private_key_t target_key_fo_pub;
				extended_private_key_t master_private_fo_extint;
				extended_public_key_t target_public_key;

				for (uint8_t accNo = dev_accntMinMax[0]; accNo <= dev_accntMinMax[1]; accNo++) {
					hardened_private_child_from_private(master_private, &target_key, 44);
					hardened_private_child_from_private(&target_key, &target_key, 0);
					hardened_private_child_from_private(&target_key, &master_private_fo_extint, accNo); //acount-number
					normal_private_child_from_private(&master_private_fo_extint, &target_key, 0); //extension-0-internal-external
					//m/44'/0'/0'/0/x
					for (int x = dev_childrenMinMax[0]; x <= dev_childrenMinMax[1]; x++) {

						normal_private_child_from_private(&target_key, &target_key_fo_pub, x); //child x
						calc_public(&target_key_fo_pub, &target_public_key);
						calc_hash160(&target_public_key, hash);


						if (device_hashcmp((uint32_t*)hash, (uint32_t*)dev_uniqueTargetAddressBytes) == 0) {
							dev_retEntropy[0] = curEntropy[0];
							dev_retEntropy[1] = curEntropy[1];
							dev_retAccntPath[0] = accNo;
							dev_retAccntPath[1] = x;
							bDone = 1;
							break;
						}
						if (bDone != 0)
							break;
					}
					if (bDone != 0)
						break;
				}//accNo
				if (bDone != 0)
					break;
			}
		}//for word 11 
	}//word 10
	__syncthreads(); // Synchronize to ensure all data is loaded
	if (threadIdx.x == 0) {
		atomicAdd(nProcessedInstances, ourBlockProcNormal);
	}
}//DICTIONARY ATTACK
