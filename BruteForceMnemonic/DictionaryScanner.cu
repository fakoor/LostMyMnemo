#include "hip/hip_runtime.h"
#include <stdafx.h>
#include <stdio.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "GPU.h"
#include "AdaptiveBase.h"

#include "EntropyTools.cuh"
#include "DictionaryScanner.cuh"


static inline __device__ int device_hashcmp(const  uint32_t* p1, const uint32_t* p2) {
#pragma unroll
	for (auto i = 0; i < 20/4; ++i) {
		if (p1[i] != p2[i]) {
			return p1[i] < p2[i] ? -1 : 1; // Return -1 if p1 < p2, 1 if p1 > p2
		}
	}
	return 0; // Memory regions are equal
}


__device__
bool CheckSumValidate(uint8_t checkSumInputBlock[16], uint64_t entropy[2], uint8_t reqChecksum) {
	uint8_t entropy_hash[32];
	//uint64_t* entropy = curEntropy;

	checkSumInputBlock[15] = entropy[1] & 0xFF;
	checkSumInputBlock[14] = (entropy[1] >> 8) & 0xFF;
	checkSumInputBlock[13] = (entropy[1] >> 16) & 0xFF;
	checkSumInputBlock[12] = (entropy[1] >> 24) & 0xFF;
	checkSumInputBlock[11] = (entropy[1] >> 32) & 0xFF;
	checkSumInputBlock[10] = (entropy[1] >> 40) & 0xFF;
	checkSumInputBlock[9] = (entropy[1] >> 48) & 0xFF;
	checkSumInputBlock[8] = (entropy[1] >> 56) & 0xFF;


	sha256((uint32_t*)checkSumInputBlock, 16, (uint32_t*)entropy_hash);
	uint8_t achievedChecksum = (entropy_hash[0] >> 4) & 0x0F;

	bool bChkMatched = (achievedChecksum == reqChecksum);
	return bChkMatched;
}

__device__
int LookupHash(const uint32_t* hash, uint32_t* hash_from_table, const uint32_t* mnemonic, foundStruct* fnd_ret, uint32_t path, uint32_t child)
{
	int found = 0;
	bool search_state = true;
	uint32_t line_cnt = 1;
	uint32_t point = 0;
	uint32_t point_last = 0;
	uint32_t interval = line_cnt / 3;
	//uint32_t* hash_from_table;
	while (point < line_cnt) {
		point_last = point;
		if (interval == 0) {
			search_state = false;
		}
		if (search_state) {
			point += interval;

			if (point >= line_cnt) {
				point = point_last;
				interval = (line_cnt - point) / 2;
				continue;
			}
			//hash_from_table = &table.table[point * (20 / 4)];
		}
		else {
			//hash_from_table = &table.table[point * (20 / 4)];
			point += 1;
		}

		int cmp = 0;
		if (hash[0] < hash_from_table[0])
		{
			cmp = -1;
		}
		else if (hash[0] > hash_from_table[0])
		{
			cmp = 1;
		}
		else if (hash[1] < hash_from_table[1])
		{
			cmp = -2;
		}
		else if (hash[1] > hash_from_table[1])
		{
			cmp = 2;
		}
		else if (hash[2] < hash_from_table[2])
		{
			cmp = -3;
		}
		else if (hash[2] > hash_from_table[2])
		{
			cmp = 3;
		}
		else if (hash[3] < hash_from_table[3])
		{
			cmp = -4;
		}
		else if (hash[3] > hash_from_table[3])
		{
			cmp = 4;
		}
		else if (hash[4] < hash_from_table[4])
		{
			cmp = -5;
		}
		else if (hash[4] > hash_from_table[4])
		{
			cmp = 5;
		}

		if (search_state) {
			if (cmp < 0) {
				if (interval < 20) {
					search_state = false;
				}
				else
				{
					interval = interval / 2;
				}
				point = point_last;
				continue;
			}
			else if (cmp == 0) {
				search_state = false;
			}
			else {
				continue;
			}
		}

		if (cmp <= 0) {
			if (cmp == 0)
			{
				found = 1;
				uint32_t cnt = fnd_ret->count_found;
				fnd_ret->count_found++;
				if (cnt < MAX_FOUND_ADDRESSES)
				{
					for (int i = 0; i < 5; i++) fnd_ret->found_info[cnt].hash160[i] = hash[i];
					for (int i = 0; i < SIZE32_MNEMONIC_FRAME; i++) fnd_ret->found_info[cnt].mnemonic[i] = mnemonic[i];
					fnd_ret->found_info[cnt].path = path;
					fnd_ret->found_info[cnt].child = child;
				}
			}
			break;
		}

		if (cmp > 1) {
			if (dev_num_bytes_find[0] == 8) {
				if (hash[1] == hash_from_table[1]) found = 2;
			}
#ifdef TEST_MODE
			else if (dev_num_bytes_find[0] == 7) {
				if ((hash[1] & 0x00FFFFFF) == (hash_from_table[1] & 0x00FFFFFF)) found = 2;
			}
			else if (dev_num_bytes_find[0] == 6) {
				if ((hash[1] & 0x0000FFFF) == (hash_from_table[1] & 0x0000FFFF)) found = 2;
			}
			else if (dev_num_bytes_find[0] == 5) {
				if ((hash[1] & 0x000000FF) == (hash_from_table[1] & 0x000000FF)) found = 2;
			}
#endif //TEST_MODE
		}


		if (found == 2) {
			uint32_t cnt = fnd_ret->count_found_bytes;
			fnd_ret->count_found_bytes++;
			if (cnt < MAX_FOUND_ADDRESSES)
			{
				for (int i = 0; i < 5; i++)
				{
					fnd_ret->found_bytes_info[cnt].hash160_from_table[i] = hash_from_table[i];
					fnd_ret->found_bytes_info[cnt].hash160[i] = hash[i];
				}
				for (int i = 0; i < SIZE32_MNEMONIC_FRAME; i++) fnd_ret->found_bytes_info[cnt].mnemonic[i] = mnemonic[i];
				fnd_ret->found_bytes_info[cnt].path = path;
				fnd_ret->found_bytes_info[cnt].child = child;
			}
			break;
		}

	}

	return found;
}


__global__ void gl_DictionaryScanner(
	const uint64_t* __restrict__ nProcessingIteration,
	uint64_t* nProcessedInstances
)
{
	unsigned int effective_idx = blockIdx.x * blockDim.x + threadIdx.x;

	uint32_t nTotalThreads = blockDim.x * gridDim.x;

	__shared__ uint64_t ourBlockProcNormal;
	__shared__ uint64_t nGridJobCap;
	__shared__ uint8_t bDone;

	int16_t local_static_word_index[12];

	// Initialize the shared variable
	if (threadIdx.x == 0) {
		ourBlockProcNormal = 0; // Only the first thread initializes it

		nGridJobCap = ULLONG_MAX;//0xFFFFFFFFFFFFFFFFull;
		bDone = 0;
	}
	__syncthreads(); // Synchronize to ensure the initialization is complete

	for (int i = 0; i < 6; i++) {
		local_static_word_index[i] = dev_static_words_indices[i];
	}


	uint64_t curEntropy[2];
	curEntropy[0] = dev_EntropyAbsolutePrefix64[PTR_AVOIDER];
	curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];


	uint8_t reqChecksum = 0;

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS] = { 
		 dev_AdaptiveBaseCurrentBatchInitialDigits[0]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[1]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[2]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[3]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[4]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[5] 
	};

	//TODO block: prefix is based on  words 9 and 10 while the last word 11 is iterated inside the thread

	const int16_t lastPos_adaptive = MAX_ADAPTIVE_BASE_POSITIONS - 1;
	const int16_t lastPosCarryTrig = dev_AdaptiveBaseDigitCarryTrigger[lastPos_adaptive];

	//instead of effective_idx, increment so that bypass and leave last word iteration completely to this thread

	SyncBipIndexFromAdaptiveDigits(local_static_word_index , dev_AdaptiveBaseDigitSet , curDigits);

	//Work with Current Entropy
	uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
	uint8_t* mnemonic = mnemonic_phrase;
	uint64_t nLoopMasterOffset = effective_idx * lastPosCarryTrig + *nProcessingIteration * (nTotalThreads * lastPosCarryTrig);

	uint8_t checkSumInputBlock[16];//with constant portion initalization
	checkSumInputBlock[7] = dev_EntropyAbsolutePrefix64[0] & 0xFF;
	checkSumInputBlock[6] = (dev_EntropyAbsolutePrefix64[0] >> 8) & 0xFF;
	checkSumInputBlock[5] = (dev_EntropyAbsolutePrefix64[0] >> 16) & 0xFF;
	checkSumInputBlock[4] = (dev_EntropyAbsolutePrefix64[0] >> 24) & 0xFF;
	checkSumInputBlock[3] = (dev_EntropyAbsolutePrefix64[0] >> 32) & 0xFF;
	checkSumInputBlock[2] = (dev_EntropyAbsolutePrefix64[0] >> 40) & 0xFF;
	checkSumInputBlock[1] = (dev_EntropyAbsolutePrefix64[0] >> 48) & 0xFF;
	checkSumInputBlock[0] = (dev_EntropyAbsolutePrefix64[0] >> 56) & 0xFF;


#pragma unroll
	for (int16_t nWordElevenOffset = 0; nWordElevenOffset < lastPosCarryTrig; nWordElevenOffset++) {
		//break on nTried < MAX_TRY_PER_THREAD
		uint64_t nInstanceOffset = nLoopMasterOffset + nWordElevenOffset;

		if (nInstanceOffset > nGridJobCap) {
			break;
		}

		if (false== IncrementAdaptiveDigits(
			dev_AdaptiveBaseDigitCarryTrigger
			, dev_AdaptiveBaseCurrentBatchInitialDigits
			, nInstanceOffset, curDigits)){

			atomicMin(&nGridJobCap, nInstanceOffset);
			break;
		}
		atomicAdd(&ourBlockProcNormal, 1);

		if (threadIdx.x == 0) {

		}
		//else {
		//	atomicMax(&nMaxCloudAdd, effective_idx);
		//}

		SyncBipIndexFromAdaptiveDigits(local_static_word_index, dev_AdaptiveBaseDigitSet, curDigits);

		curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];
		AdaptiveUpdateMnemonicLow64(&curEntropy[1], dev_AdaptiveBaseDigitSet, curDigits);
		int16_t wordElevenBipVal = local_static_word_index[11];


#if 0 //not required for checksum comparison here
		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);
#endif
		reqChecksum = wordElevenBipVal & 0x000F;
		bool bChkMatched = CheckSumValidate (checkSumInputBlock, curEntropy, reqChecksum);


		if (!bChkMatched) {
			continue;
		}
		//NOTE : If we reach here the checksum is already matching, just need to check the address
		//__syncthreads(); // Synchronize to and check if have a valid checksum to continue with
		//if (bChkMatched) { //scrutinize : bCouldAdd

		uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
		uint8_t* mnemonic = mnemonic_phrase;
		uint32_t ipad[256 / 4];
		uint32_t opad[256 / 4];
		uint32_t seed[64 / 4];


		//Work with Current Entropy
		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);

		//if (idx == 0) {
//			printf("nemo-%u  (retry.remain=%d/%d) = :%s \r\n\r\n", effective_idx,nTried,MAX_TRY_PER_THREAD, mnemonic);
		//}
		//entropy_to_mnemonic(entropy, mnemonic);
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)ipad + x) = 0x3636363636363636ULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)opad + x) = 0x5C5C5C5C5C5C5C5CULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x + 128 / 4] = *(uint32_t*)((uint32_t*)&salt_swap + x);
		}
		sha512_swap((uint64_t*)ipad, 140, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			seed[x] = ipad[128 / 4 + x];
		}
		for (int x = 1; x < 2048; x++) {
			sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				seed[x] = seed[x] ^ ipad[128 / 4 + x];
			}
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x] = 0x36363636 ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			opad[x] = 0x5C5C5C5C ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			ipad[x + 128 / 4] = seed[x];
		}
		//ipad[192 / 4] = 0;
		//opad[192 / 4] = 0;
		sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 128 / 8; x++) {
			*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x) = SWAP512(*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x));
		}

		//printf("END block %d - thread  %d - EffectiveId:%d - curDigits:%d-%d-%d-%d-%d-%d %s\r\n", blockId, threadId, effective_idx
		//	, curDigits[0], curDigits[1], curDigits[2], curDigits[3], curDigits[4], curDigits[5] , mnemonic);

		//dev_uniqueTargetAddressBytes;
		{
			const extended_private_key_t* master_private = (extended_private_key_t*)&ipad[128 / 4];

			uint32_t hash[(20 / 4)];
			extended_private_key_t target_key;
			extended_private_key_t target_key_fo_pub;
			extended_private_key_t master_private_fo_extint;
			extended_public_key_t target_public_key;

			for (uint8_t accNo = 0; accNo < 3; accNo++) {
				hardened_private_child_from_private(master_private, &target_key, 44);
				hardened_private_child_from_private(&target_key, &target_key, 0);

				hardened_private_child_from_private(&target_key, &master_private_fo_extint, accNo); //acount-number

				//for (uint8_t h33 = 0; h33 < 3; h33++) {
					normal_private_child_from_private(&master_private_fo_extint, &target_key, 0); //extension-0-internal-external
					//m/44'/0'/0'/0/x
					for (int x = 0; x < dev_num_childs[0]; x++) {

						normal_private_child_from_private(&target_key, &target_key_fo_pub, x); //child x
						calc_public(&target_key_fo_pub, &target_public_key);
						calc_hash160(&target_public_key, hash);

						//find_hash_in_table(hash, tables_legacy[(uint8_t)hash[0]], (uint32_t*) mnemonic, &ret->f[0], 4, 0);
						//LookupHash(hash, (uint32_t*) dev_uniqueTargetAddressBytes, (uint32_t*)mnemonic, &ret->f[0], 4, 0);

						if (device_hashcmp((uint32_t*)hash, (uint32_t*)dev_uniqueTargetAddressBytes) == 0) {
#if 1
							dev_retEntropy[0] = curEntropy[0];
							dev_retEntropy[1] = curEntropy[1];
							dev_retAccntPath[0] = accNo;
							dev_retAccntPath[1] = x;
							bDone = 1;
							break;
#endif
						}
						if (bDone != 0)
							break;

					}
					if (bDone != 0)
						break;

				//}//h33
			}//h34
			if (bDone != 0)
				break;

		}
#if 0
		key_to_hash160((extended_private_key_t*)&ipad[128 / 4], tables_legacy, tables_segwit, tables_native_segwit, (uint32_t*)mnemonic, ret);

		atomicMax(&bDone, DictionaryCheckFound(ret));
		if (bDone ) {
			atomicMin(&nGridJobCap, nInstanceOffset);
			//if (nInstanceOffset == nGridJobCap) {
			//	printf("\r\n\r\n\tBreaking operation at %llu Since Match is Found!\r\n\r\n", nInstanceOffset);
			//}

			break;
		}
#endif
	}//for 

	__syncthreads(); // Synchronize to ensure all data is loaded
	if (threadIdx.x == 0) {
		atomicAdd(nProcessedInstances, ourBlockProcNormal);
	}

}//DICTIONARY ATTACK
