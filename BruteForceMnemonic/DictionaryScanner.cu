#include "hip/hip_runtime.h"
#include <stdafx.h>
#include <stdio.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "GPU.h"
#include "AdaptiveBase.h"

#include "EntropyTools.cuh"

__global__ void gl_DictionaryScanner(
	const uint64_t* __restrict__ nProcessedIterations,
	uint64_t* nProcessedInstances,
	const tableStruct* __restrict__ tables_legacy,
	const tableStruct* __restrict__ tables_segwit,
	const tableStruct* __restrict__ tables_native_segwit,
	retStruct* __restrict__ ret
)
{
	unsigned int effective_idx = blockIdx.x * blockDim.x + threadIdx.x;

	uint32_t nTotalThreads = blockDim.x * gridDim.x;

	__shared__ uint64_t ourBlockProcNormal;
	__shared__ unsigned int nMoreIterated;
	__shared__ uint64_t bBulkJobeDoneAt;

	int16_t local_static_word_index[12];

	// Initialize the shared variable
	if (threadIdx.x == 0) {
		ourBlockProcNormal = 0; // Only the first thread initializes it

		nMoreIterated = 0;
		bBulkJobeDoneAt = 0xFFFFFFFFFFFFFFFFull;
	}
	__syncthreads(); // Synchronize to ensure the initialization is complete

	for (int i = 0; i < 6; i++) {
		local_static_word_index[i] = dev_static_words_indices[i];
	}


	uint64_t curEntropy[2];
	curEntropy[0] = dev_EntropyAbsolutePrefix64[PTR_AVOIDER];
	curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];


	uint8_t reqChecksum = 0;
	uint8_t achievedChecksum = 1;
	bool bChkMatched = false;

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS] = { 
		 dev_AdaptiveBaseCurrentBatchInitialDigits[0]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[1]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[2]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[3]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[4]
		,dev_AdaptiveBaseCurrentBatchInitialDigits[5] 
	};

	//TODO block: prefix is based on  words 9 and 10 while the last word 11 is iterated inside the thread

	int nTried = 0;
	bool bCouldAdd = false;
	const int16_t lastPos_adaptive = MAX_ADAPTIVE_BASE_POSITIONS - 1;
	const int16_t lastPosCarryTrig = dev_AdaptiveBaseDigitCarryTrigger[lastPos_adaptive];

	//instead of effective_idx, increment so that bypass and leave last word iteration completely to this thread

	SyncBipIndexFromAdaptiveDigits(local_static_word_index , dev_AdaptiveBaseDigitSet , curDigits);

	//Work with Current Entropy
	uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
	uint8_t* mnemonic = mnemonic_phrase;
	uint64_t nLoopMasterOffset = effective_idx * lastPosCarryTrig + *nProcessedIterations * nTotalThreads;
	for (int16_t nWordElevenOffset = 0; nWordElevenOffset < lastPosCarryTrig; nWordElevenOffset++) {
		//break on nTried < MAX_TRY_PER_THREAD
		uint64_t nInstanceOffset = nLoopMasterOffset + nWordElevenOffset;

		if (nInstanceOffset > bBulkJobeDoneAt) {
			//if (blockIdx.x == 0) {
			//	printf("\r\nBlock Job done at:%llu\r\n", nInstanceOffset);
			//}
			break;
		}

		bCouldAdd = IncrementAdaptiveDigits(
			dev_AdaptiveBaseDigitCarryTrigger
			, dev_AdaptiveBaseCurrentBatchInitialDigits
			, nInstanceOffset, curDigits);

		if (bCouldAdd == false /*&& lastPosCarryTrig == nWordElevenOffset*/) {
			//if (effective_idx == nMaxCloudAdd + 1) {
			//	printf("Can not add bulk at %x", lastPosCarryTrig);
			//}

			atomicExch(&bBulkJobeDoneAt, nInstanceOffset);
			break;
		}
		atomicAdd(&ourBlockProcNormal, 1);
		//else {
		//	atomicMax(&nMaxCloudAdd, effective_idx);
		//}

		SyncBipIndexFromAdaptiveDigits(local_static_word_index, dev_AdaptiveBaseDigitSet, curDigits);

		//int16_t word_11_BIP = dev_AdaptiveBaseDigitSet[lastPos_adaptive][nWordElevenOffset];
		//curDigits[lastPos_adaptive] = nWordElevenOffset;
		curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];
		AdaptiveUpdateMnemonicLow64(&curEntropy[1], dev_AdaptiveBaseDigitSet, curDigits);
		//local_static_word_index[11] = word_11_BIP;
		int16_t wordElevenBipVal = local_static_word_index[11];


#if 0 //not required for checksum comparison here
		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);
#endif
		reqChecksum = wordElevenBipVal & 0x000F;

		uint8_t entropy_hash[32];
		uint8_t bytes[16];
		uint64_t* entropy = curEntropy;

		bytes[15] = entropy[1] & 0xFF;
		bytes[14] = (entropy[1] >> 8) & 0xFF;
		bytes[13] = (entropy[1] >> 16) & 0xFF;
		bytes[12] = (entropy[1] >> 24) & 0xFF;
		bytes[11] = (entropy[1] >> 32) & 0xFF;
		bytes[10] = (entropy[1] >> 40) & 0xFF;
		bytes[9] = (entropy[1] >> 48) & 0xFF;
		bytes[8] = (entropy[1] >> 56) & 0xFF;

		bytes[7] = entropy[0] & 0xFF;
		bytes[6] = (entropy[0] >> 8) & 0xFF;
		bytes[5] = (entropy[0] >> 16) & 0xFF;
		bytes[4] = (entropy[0] >> 24) & 0xFF;
		bytes[3] = (entropy[0] >> 32) & 0xFF;
		bytes[2] = (entropy[0] >> 40) & 0xFF;
		bytes[1] = (entropy[0] >> 48) & 0xFF;
		bytes[0] = (entropy[0] >> 56) & 0xFF;

		sha256((uint32_t*)bytes, 16, (uint32_t*)entropy_hash);
		achievedChecksum = (entropy_hash[0] >> 4) & 0x0F;

		bChkMatched = (achievedChecksum == reqChecksum);

		nTried++;
#if 0
		if (effective_idx <= 2 || (effective_idx <= 242 && effective_idx >= 240)) {
			uint8_t word_11_text[10];
			GetWordFromBipIndex(wordElevenBipVal, word_11_text);
			//entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);
			printf("idx:%u [+ %llu ] @%s (%d : %d) on [%s]  CHK: %s req=%u ach=%u \r\n"
				, effective_idx
				, nInstanceOffset
				, word_11_text
				, nWordElevenOffset
				, wordElevenBipVal
				, mnemonic
				, (bChkMatched) ?  "OK" : "Bad"
				, reqChecksum
				, achievedChecksum
			);
		}
#endif
#if 1
		if (!bChkMatched) {
			continue;
		}
#endif
		//__syncthreads(); // Synchronize to and check if have a valid checksum to continue with
		if (bChkMatched) { //scrutinize : bCouldAdd

			uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
			uint8_t* mnemonic = mnemonic_phrase;
			uint32_t ipad[256 / 4];
			uint32_t opad[256 / 4];
			uint32_t seed[64 / 4];


			//Work with Current Entropy
			entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);

			//if (idx == 0) {
	//			printf("nemo-%u  (retry.remain=%d/%d) = :%s \r\n\r\n", effective_idx,nTried,MAX_TRY_PER_THREAD, mnemonic);
			//}
			//entropy_to_mnemonic(entropy, mnemonic);
#pragma unroll
			for (int x = 0; x < 120 / 8; x++) {
				*(uint64_t*)((uint64_t*)ipad + x) = 0x3636363636363636ULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
			}
#pragma unroll
			for (int x = 0; x < 120 / 8; x++) {
				*(uint64_t*)((uint64_t*)opad + x) = 0x5C5C5C5C5C5C5C5CULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
			}
#pragma unroll
			for (int x = 120 / 4; x < 128 / 4; x++) {
				ipad[x] = 0x36363636;
			}
#pragma unroll
			for (int x = 120 / 4; x < 128 / 4; x++) {
				opad[x] = 0x5C5C5C5C;
			}
#pragma unroll
			for (int x = 0; x < 16 / 4; x++) {
				ipad[x + 128 / 4] = *(uint32_t*)((uint32_t*)&salt_swap + x);
			}
			sha512_swap((uint64_t*)ipad, 140, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				seed[x] = ipad[128 / 4 + x];
			}
			for (int x = 1; x < 2048; x++) {
				sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
				sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
				for (int x = 0; x < 64 / 4; x++) {
					seed[x] = seed[x] ^ ipad[128 / 4 + x];
				}
			}
#pragma unroll
			for (int x = 0; x < 16 / 4; x++) {
				ipad[x] = 0x36363636 ^ *(uint32_t*)((uint32_t*)&key_swap + x);
			}
#pragma unroll
			for (int x = 0; x < 16 / 4; x++) {
				opad[x] = 0x5C5C5C5C ^ *(uint32_t*)((uint32_t*)&key_swap + x);
			}
#pragma unroll
			for (int x = 16 / 4; x < 128 / 4; x++) {
				ipad[x] = 0x36363636;
			}
#pragma unroll
			for (int x = 16 / 4; x < 128 / 4; x++) {
				opad[x] = 0x5C5C5C5C;
			}
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				ipad[x + 128 / 4] = seed[x];
			}
			//ipad[192 / 4] = 0;
			//opad[192 / 4] = 0;
			sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 128 / 8; x++) {
				*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x) = SWAP512(*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x));
			}

			//printf("END block %d - thread  %d - EffectiveId:%d - curDigits:%d-%d-%d-%d-%d-%d %s\r\n", blockId, threadId, effective_idx
			//	, curDigits[0], curDigits[1], curDigits[2], curDigits[3], curDigits[4], curDigits[5] , mnemonic);


			key_to_hash160((extended_private_key_t*)&ipad[128 / 4], tables_legacy, tables_segwit, tables_native_segwit, (uint32_t*)mnemonic, ret);
			
		}
	} 

	__syncthreads(); // Synchronize to ensure all data is loaded
	if (threadIdx.x == 0) {
		atomicAdd(nProcessedInstances, ourBlockProcNormal);
	}

}//DICTIONARY ATTACK
