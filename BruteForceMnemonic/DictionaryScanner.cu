#include "hip/hip_runtime.h"
#include <stdafx.h>
#include <stdio.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include <GPU.h>
#include "AdaptiveBase.h"

#include "EntropyTools.cuh"

__global__ void gl_DictionaryScanner(
	uint64_t* nBatchPlannedProc,
	uint64_t* nBatchMoreProc,
	const tableStruct* __restrict__ tables_legacy,
	const tableStruct* __restrict__ tables_segwit,
	const tableStruct* __restrict__ tables_native_segwit,
	retStruct* __restrict__ ret
)
{
	unsigned int effective_idx = blockIdx.x * blockDim.x + threadIdx.x;

	uint16_t totalPlannedCount = blockDim.x * gridDim.x;

	__shared__ uint64_t ourBlockProcNormal;
	__shared__ uint64_t ourBlockProcExtra;
	__shared__ uint64_t ourBlockBadChkSum;
	__shared__ uint64_t ourBlockGoodChkSum;
	__shared__ uint64_t nMaxCloudAdd;
	__shared__ unsigned int nMoreIterated;
	int16_t local_static_word_index[12];

	// Initialize the shared variable
	if (threadIdx.x == 0) {
		ourBlockProcNormal = 0; // Only the first thread initializes it
		ourBlockProcExtra = 0;
		ourBlockBadChkSum = 0;
		ourBlockGoodChkSum = 0;

		nMaxCloudAdd = 0;
		nMoreIterated = 0;
	}
	__syncthreads(); // Synchronize to ensure the initialization is complete

	for (int i = 0; i < 6; i++) {
		local_static_word_index[i] = dev_static_words_indices[i];
	}


	uint64_t curEntropy[2];
	curEntropy[0] = dev_EntropyAbsolutePrefix64[PTR_AVOIDER];
	curEntropy[1] = dev_EntropyNextPrefix2[PTR_AVOIDER];


	uint8_t reqChecksum = 0;
	uint8_t achievedChecksum = 1;
	bool bChkSumFailed = true;

	int16_t curDigits[MAX_ADAPTIVE_BASE_POSITIONS] = { 0,0,0,0,0,0 };

	//TODO block: prefix is based on  words 9 and 10 while the last word 11 is iterated inside the thread

	int nTried = 0;
	bool bCouldAdd = false;
	do {
		bCouldAdd = IncrementAdaptiveDigits(
			dev_AdaptiveBaseDigitCarryTrigger
			, dev_AdaptiveBaseCurrentBatchInitialDigits
			, effective_idx, curDigits);
		if (bCouldAdd == false) {
			if (effective_idx == nMaxCloudAdd + 1) {
				printf("Can not add at %x", effective_idx);
			}

			break;
		}
		else {
			atomicMax(&nMaxCloudAdd, effective_idx);
		}

		AdaptiveUpdateMnemonicLow64(&curEntropy[1], dev_AdaptiveBaseDigitSet, curDigits);
		local_static_word_index[6] = dev_AdaptiveBaseDigitSet[0][curDigits[0]];
		local_static_word_index[7] = dev_AdaptiveBaseDigitSet[1][curDigits[1]];
		local_static_word_index[8] = dev_AdaptiveBaseDigitSet[2][curDigits[2]];
		local_static_word_index[9] = dev_AdaptiveBaseDigitSet[3][curDigits[3]];
		local_static_word_index[10] = dev_AdaptiveBaseDigitSet[4][curDigits[4]];
		local_static_word_index[11] = dev_AdaptiveBaseDigitSet[5][curDigits[5]];



		//Work with Current Entropy
		uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
		uint8_t* mnemonic = mnemonic_phrase;

		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);



		int16_t chkPosIdx = MAX_ADAPTIVE_BASE_POSITIONS - 1;
		int16_t chkWordIdx = curDigits[chkPosIdx];
		uint16_t thisVal = (dev_AdaptiveBaseDigitSet[chkPosIdx][chkWordIdx]);
		uint8_t tmp = (uint8_t)(thisVal & 0x0F);
		reqChecksum = tmp;

		uint8_t entropy_hash[32];
		uint8_t bytes[16];
		uint64_t* entropy = curEntropy;

		bytes[15] = entropy[1] & 0xFF;
		bytes[14] = (entropy[1] >> 8) & 0xFF;
		bytes[13] = (entropy[1] >> 16) & 0xFF;
		bytes[12] = (entropy[1] >> 24) & 0xFF;
		bytes[11] = (entropy[1] >> 32) & 0xFF;
		bytes[10] = (entropy[1] >> 40) & 0xFF;
		bytes[9] = (entropy[1] >> 48) & 0xFF;
		bytes[8] = (entropy[1] >> 56) & 0xFF;

		bytes[7] = entropy[0] & 0xFF;
		bytes[6] = (entropy[0] >> 8) & 0xFF;
		bytes[5] = (entropy[0] >> 16) & 0xFF;
		bytes[4] = (entropy[0] >> 24) & 0xFF;
		bytes[3] = (entropy[0] >> 32) & 0xFF;
		bytes[2] = (entropy[0] >> 40) & 0xFF;
		bytes[1] = (entropy[0] >> 48) & 0xFF;
		bytes[0] = (entropy[0] >> 56) & 0xFF;

		sha256((uint32_t*)bytes, 16, (uint32_t*)entropy_hash);
		achievedChecksum = (entropy_hash[0] >> 4) & 0x0F;

		bChkSumFailed = (achievedChecksum != reqChecksum);

		nTried++;

		if (bChkSumFailed) {
			atomicAdd(&ourBlockProcExtra, 1);
		}
		else {
			break;
		}
	} while (nTried < MAX_TRY_PER_THREAD); //do

	__syncthreads(); // Synchronize to and check if have a valid checksum to continue with
	if (bCouldAdd/*bChkSumFailed == false*/) { //scrutinize
		atomicAdd(&ourBlockGoodChkSum, 1);

		uint8_t mnemonic_phrase[SIZE_MNEMONIC_FRAME] = { 0 };
		uint8_t* mnemonic = mnemonic_phrase;
		uint32_t ipad[256 / 4];
		uint32_t opad[256 / 4];
		uint32_t seed[64 / 4];


		//Work with Current Entropy
		entropy_to_mnemonic_with_offset(curEntropy, mnemonic, 0, local_static_word_index);

		//if (idx == 0) {
//			printf("nemo-%u  (retry.remain=%d/%d) = :%s \r\n\r\n", effective_idx,nTried,MAX_TRY_PER_THREAD, mnemonic);
		//}
		//entropy_to_mnemonic(entropy, mnemonic);
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)ipad + x) = 0x3636363636363636ULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 0; x < 120 / 8; x++) {
			*(uint64_t*)((uint64_t*)opad + x) = 0x5C5C5C5C5C5C5C5CULL ^ SWAP512(*(uint64_t*)((uint64_t*)mnemonic + x));
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 120 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x + 128 / 4] = *(uint32_t*)((uint32_t*)&salt_swap + x);
		}
		sha512_swap((uint64_t*)ipad, 140, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			seed[x] = ipad[128 / 4 + x];
		}
		for (int x = 1; x < 2048; x++) {
			sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
			sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
			for (int x = 0; x < 64 / 4; x++) {
				seed[x] = seed[x] ^ ipad[128 / 4 + x];
			}
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			ipad[x] = 0x36363636 ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 0; x < 16 / 4; x++) {
			opad[x] = 0x5C5C5C5C ^ *(uint32_t*)((uint32_t*)&key_swap + x);
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			ipad[x] = 0x36363636;
		}
#pragma unroll
		for (int x = 16 / 4; x < 128 / 4; x++) {
			opad[x] = 0x5C5C5C5C;
		}
#pragma unroll
		for (int x = 0; x < 64 / 4; x++) {
			ipad[x + 128 / 4] = seed[x];
		}
		//ipad[192 / 4] = 0;
		//opad[192 / 4] = 0;
		sha512_swap((uint64_t*)ipad, 192, (uint64_t*)&opad[128 / 4]);
		sha512_swap((uint64_t*)opad, 192, (uint64_t*)&ipad[128 / 4]);
#pragma unroll
		for (int x = 0; x < 128 / 8; x++) {
			*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x) = SWAP512(*(uint64_t*)((uint64_t*)&ipad[128 / 4] + x));
		}

		//printf("END block %d - thread  %d - EffectiveId:%d - curDigits:%d-%d-%d-%d-%d-%d %s\r\n", blockId, threadId, effective_idx
		//	, curDigits[0], curDigits[1], curDigits[2], curDigits[3], curDigits[4], curDigits[5] , mnemonic);

		atomicAdd(&ourBlockProcNormal, 1);

		key_to_hash160((extended_private_key_t*)&ipad[128 / 4], tables_legacy, tables_segwit, tables_native_segwit, (uint32_t*)mnemonic, ret);
		//__syncthreads();
	}
	__syncthreads(); // Synchronize to ensure all data is loaded
	if (threadIdx.x == 0) {
		atomicAdd(nBatchPlannedProc, ourBlockProcNormal);
		//atomicAdd(nBatchMoreProc, ourBlockProcExtra);
		*nBatchMoreProc = 0;
	}

}//DICTIONARY ATTACK
