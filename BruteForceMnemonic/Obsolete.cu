#include "stdafx.h"
#include "hip/hip_runtime.h"
#include "GPU.h"

__device__
int LookupHash(const uint32_t* hash, uint32_t* hash_from_table, const uint32_t* mnemonic, foundStruct* fnd_ret, uint32_t path, uint32_t child)
{
	int found = 0;
	bool search_state = true;
	uint32_t line_cnt = 1;
	uint32_t point = 0;
	uint32_t point_last = 0;
	uint32_t interval = line_cnt / 3;
	//uint32_t* hash_from_table;
	while (point < line_cnt) {
		point_last = point;
		if (interval == 0) {
			search_state = false;
		}
		if (search_state) {
			point += interval;

			if (point >= line_cnt) {
				point = point_last;
				interval = (line_cnt - point) / 2;
				continue;
			}
			//hash_from_table = &table.table[point * (20 / 4)];
		}
		else {
			//hash_from_table = &table.table[point * (20 / 4)];
			point += 1;
		}

		int cmp = 0;
		if (hash[0] < hash_from_table[0])
		{
			cmp = -1;
		}
		else if (hash[0] > hash_from_table[0])
		{
			cmp = 1;
		}
		else if (hash[1] < hash_from_table[1])
		{
			cmp = -2;
		}
		else if (hash[1] > hash_from_table[1])
		{
			cmp = 2;
		}
		else if (hash[2] < hash_from_table[2])
		{
			cmp = -3;
		}
		else if (hash[2] > hash_from_table[2])
		{
			cmp = 3;
		}
		else if (hash[3] < hash_from_table[3])
		{
			cmp = -4;
		}
		else if (hash[3] > hash_from_table[3])
		{
			cmp = 4;
		}
		else if (hash[4] < hash_from_table[4])
		{
			cmp = -5;
		}
		else if (hash[4] > hash_from_table[4])
		{
			cmp = 5;
		}

		if (search_state) {
			if (cmp < 0) {
				if (interval < 20) {
					search_state = false;
				}
				else
				{
					interval = interval / 2;
				}
				point = point_last;
				continue;
			}
			else if (cmp == 0) {
				search_state = false;
			}
			else {
				continue;
			}
		}

		if (cmp <= 0) {
			if (cmp == 0)
			{
				found = 1;
				uint32_t cnt = fnd_ret->count_found;
				fnd_ret->count_found++;
				if (cnt < MAX_FOUND_ADDRESSES)
				{
					for (int i = 0; i < 5; i++) fnd_ret->found_info[cnt].hash160[i] = hash[i];
					for (int i = 0; i < SIZE32_MNEMONIC_FRAME; i++) fnd_ret->found_info[cnt].mnemonic[i] = mnemonic[i];
					fnd_ret->found_info[cnt].path = path;
					fnd_ret->found_info[cnt].child = child;
				}
			}
			break;
		}

		if (cmp > 1) {
			if (dev_num_bytes_find[0] == 8) {
				if (hash[1] == hash_from_table[1]) found = 2;
			}
#ifdef TEST_MODE
			else if (dev_num_bytes_find[0] == 7) {
				if ((hash[1] & 0x00FFFFFF) == (hash_from_table[1] & 0x00FFFFFF)) found = 2;
			}
			else if (dev_num_bytes_find[0] == 6) {
				if ((hash[1] & 0x0000FFFF) == (hash_from_table[1] & 0x0000FFFF)) found = 2;
			}
			else if (dev_num_bytes_find[0] == 5) {
				if ((hash[1] & 0x000000FF) == (hash_from_table[1] & 0x000000FF)) found = 2;
			}
#endif //TEST_MODE
		}


		if (found == 2) {
			uint32_t cnt = fnd_ret->count_found_bytes;
			fnd_ret->count_found_bytes++;
			if (cnt < MAX_FOUND_ADDRESSES)
			{
				for (int i = 0; i < 5; i++)
				{
					fnd_ret->found_bytes_info[cnt].hash160_from_table[i] = hash_from_table[i];
					fnd_ret->found_bytes_info[cnt].hash160[i] = hash[i];
				}
				for (int i = 0; i < SIZE32_MNEMONIC_FRAME; i++) fnd_ret->found_bytes_info[cnt].mnemonic[i] = mnemonic[i];
				fnd_ret->found_bytes_info[cnt].path = path;
				fnd_ret->found_bytes_info[cnt].child = child;
			}
			break;
		}

	}

	return found;
}
